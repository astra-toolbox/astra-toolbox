#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/arith3d.h"

#include <cassert>

namespace astraCUDA3d {

struct opAddScaled {
	__device__ void operator()(float& out, const float in, const float inp) {
		out += in * inp;
	}
};
struct opScaleAndAdd {
	__device__ void operator()(float& out, const float in, const float inp) {
		out = in + out * inp;
	}
};
struct opAddMulScaled {
	__device__ void operator()(float& out, const float in1, const float in2, const float inp) {
		out += in1 * in2 * inp;
	}
};
struct opAddMul {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 * in2;
	}
};
struct opAdd {
	__device__ void operator()(float& out, const float in) {
		out += in;
	}
};
struct opMul {
	__device__ void operator()(float& out, const float in) {
		out *= in;
	}
};
struct opMul2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out *= in1 * in2;
	}
};
struct opDividedBy {
	__device__ void operator()(float& out, const float in) {
		if (out > 0.000001f) // out is assumed to be positive
			out = in / out;
		else
			out = 0.0f;
	}
};
struct opInvert {
	__device__ void operator()(float& out) {
		if (out > 0.000001f) // out is assumed to be positive
			out = 1 / out;
		else
			out = 0.0f;
	}
};
struct opSet {
	__device__ void operator()(float& out, const float inp) {
		out = inp;
	}
};
struct opClampMin {
	__device__ void operator()(float& out, const float inp) {
		if (out < inp)
			out = inp;
	}
};
struct opClampMax {
	__device__ void operator()(float& out, const float inp) {
		if (out > inp)
			out = inp;
	}
};




template<class op, unsigned int repeat>
__global__ void devtoD(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devFtoD(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], fParam);
		off += pitch;
		y++;
	}
}


template<class op, unsigned int repeat>
__global__ void devDtoD(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDFtoD(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDtoD(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDFtoD(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off], fParam);
		off += pitch;
		y++;
	}
}









template<typename op>
void processVol(hipDeviceptr_t* out, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+511)/512);

	float *pfOut = (float*)out;

	devtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol(hipDeviceptr_t* out, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	float *pfOut = (float*)out;

	devFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol(hipDeviceptr_t* out, const hipDeviceptr_t* in, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	float *pfOut = (float*)out;
	const float *pfIn = (const float*)in;

	devDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol(hipDeviceptr_t* out, const hipDeviceptr_t* in, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	float *pfOut = (float*)out;
	const float *pfIn = (const float*)in;

	devDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol(hipDeviceptr_t* out, const hipDeviceptr_t* in1, const hipDeviceptr_t* in2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	float *pfOut = (float*)out;
	const float *pfIn1 = (const float*)in1;
	const float *pfIn2 = (const float*)in2;

	devDDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol(hipDeviceptr_t* out, const hipDeviceptr_t* in1, const hipDeviceptr_t* in2, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	float *pfOut = (float*)out;
	const float *pfIn1 = (const float*)in1;
	const float *pfIn2 = (const float*)in2;

	devDDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

















template<typename op>
void processVol3D(hipPitchedPtr& out, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devtoD<op, 32><<<gridSize, blockSize>>>(pfOut, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}













template<typename op>
void processSino3D(hipPitchedPtr& out, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devtoD<op, 32><<<gridSize, blockSize>>>(pfOut, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processSino3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}


















#define INST_DFtoD(name) \
  template void processVol<name>(hipDeviceptr_t* out, const hipDeviceptr_t* in, float fParam, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims);

#define INST_DtoD(name) \
  template void processVol<name>(hipDeviceptr_t* out, const hipDeviceptr_t* in, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims);

#define INST_DDtoD(name) \
  template void processVol<name>(hipDeviceptr_t* out, const hipDeviceptr_t* in1, const hipDeviceptr_t* in2, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims);

#define INST_DDFtoD(name) \
  template void processVol<name>(hipDeviceptr_t* out, const hipDeviceptr_t* in1, const hipDeviceptr_t* in2, float fParam, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims);


#define INST_toD(name) \
  template void processVol<name>(hipDeviceptr_t* out, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const SDimensions3D& dims);

#define INST_FtoD(name) \
  template void processVol<name>(hipDeviceptr_t* out, float fParam, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, float fParam, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, float fParam, const SDimensions3D& dims);



INST_DFtoD(opAddScaled)
INST_DFtoD(opScaleAndAdd)
INST_DDFtoD(opAddMulScaled)
INST_DDtoD(opAddMul)
INST_DDtoD(opMul2)
INST_DtoD(opMul)
INST_DtoD(opAdd)
INST_DtoD(opDividedBy)
INST_toD(opInvert)
INST_FtoD(opMul)
INST_FtoD(opSet)
INST_FtoD(opClampMin)
INST_FtoD(opClampMax)


}
