#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2021, imec Vision Lab, University of Antwerp
           2014-2021, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneProjTexture;

namespace astraCUDA3d {

#define ZSIZE 6
static const unsigned int g_volBlockZ = ZSIZE;

static const unsigned int g_anglesPerBlock = 32;
static const unsigned int g_volBlockX = 16;
static const unsigned int g_volBlockY = 32;

static const unsigned g_MaxAngles = 1024;

struct DevConeParams {
	float4 fNumU;
	float4 fNumV;
	float4 fDen;
};

__constant__ DevConeParams gC_C[g_MaxAngles];

bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneProjTexture.filterMode = hipFilterModeLinear;
	gT_coneProjTexture.normalized = false;

	hipBindTextureToArray(gT_coneProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


//__launch_bounds__(32*16, 4)
template<bool FDKWEIGHT>
__global__ void dev_cone_BP(void* D_volData, unsigned int volPitch, int startAngle,
                            int angleOffset, int startZ, const astraCUDA3d::SDimensions3D dims,
                            float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
	//            y = z



	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	startZ += blockIdx.y * g_volBlockZ;
	if (startZ >= dims.iVolZ)
		return;

	const float fX = X - 0.5f*dims.iVolX + 0.5f;
	const float fY = Y - 0.5f*dims.iVolY + 0.5f;
	const float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	float Z[ZSIZE];
	for(int i=0; i < ZSIZE; i++)
		Z[i] = 0.0f;


	{
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu  = gC_C[angle].fNumU;
			float4 fCv  = gC_C[angle].fNumV;
			float4 fCd  = gC_C[angle].fDen;

			float fUNum = fCu.w + fX * fCu.x + fY * fCu.y + fZ * fCu.z;
			float fVNum = fCv.w + fX * fCv.x + fY * fCv.y + fZ * fCv.z;
			float fDen  = (FDKWEIGHT ? 1.0f : fCd.w) + fX * fCd.x + fY * fCd.y + fZ * fCd.z;

			float fU,fV, fr;

			for (int idx = 0; idx < ZSIZE; idx++)
			{
				fr = __fdividef(1.0f, fDen);
				fU = fUNum * fr;
				fV = fVNum * fr;
				float fVal = tex3D(gT_coneProjTexture, fU, fAngle, fV);
				Z[idx] += fr*fr*fVal;

				fUNum += fCu.z;
				fVNum += fCv.z;
				fDen  += fCd.z;
			}
		}
	}

	int endZ = ZSIZE;
	if (endZ > (int)dims.iVolZ - startZ)
		endZ = (int)dims.iVolZ - startZ;

	for(int i=0; i < endZ; i++)
		volData[((startZ+i)*dims.iVolY+Y)*volPitch+X] += Z[i] * fOutputScale;
} //End kernel



// supersampling version
__global__ void dev_cone_BP_SS(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, int startZ, const SDimensions3D dims, int iRaysPerVoxelDim, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	startZ += blockIdx.y * g_volBlockZ;
	if (startZ >= dims.iVolZ)
		return;

	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	const float fSubStep = 1.0f/iRaysPerVoxelDim;

	fOutputScale /= (iRaysPerVoxelDim*iRaysPerVoxelDim*iRaysPerVoxelDim);


	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu  = gC_C[angle].fNumU;
			float4 fCv  = gC_C[angle].fNumV;
			float4 fCd  = gC_C[angle].fDen;

			float fXs = fX;
			for (int iSubX = 0; iSubX < iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < iRaysPerVoxelDim; ++iSubZ) {

				const float fUNum = fCu.w + fXs * fCu.x + fYs * fCu.y + fZs * fCu.z;
				const float fVNum = fCv.w + fXs * fCv.x + fYs * fCv.y + fZs * fCv.z;
				const float fDen  = fCd.w + fXs * fCd.x + fYs * fCd.y + fZs * fCd.z;

				const float fr = __fdividef(1.0f, fDen);
				const float fU = fUNum * fr;
				const float fV = fVNum * fr;

				fVal += tex3D(gT_coneProjTexture, fU, fAngle, fV) * fr * fr;

				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal * fOutputScale;
	}
}


bool transferConstants(const SConeProjection* angles, unsigned int iProjAngles, const SProjectorParams3D& params)
{
	DevConeParams *p = new DevConeParams[iProjAngles];

	// We need three things in the kernel:
	// projected coordinates of pixels on the detector:

	// u: || (x-s) v (s-d) || / || u v (s-x) ||
	// v: -|| u (x-s) (s-d) || / || u v (s-x) ||

	// ray density weighting factor for the adjoint
	// || u v (s-d) ||^2 / ( |cross(u,v)| * || u v (s-x) ||^2 )

	// FDK weighting factor
	// ( || u v s || / || u v (s-x) || ) ^ 2

	// Since u and v are ratios with the same denominator, we have
	// a degree of freedom to scale the denominator. We use that to make
	// the square of the denominator equal to the relevant weighting factor.


	for (unsigned int i = 0; i < iProjAngles; ++i) {
		Vec3 u(angles[i].fDetUX, angles[i].fDetUY, angles[i].fDetUZ);
		Vec3 v(angles[i].fDetVX, angles[i].fDetVY, angles[i].fDetVZ);
		Vec3 s(angles[i].fSrcX, angles[i].fSrcY, angles[i].fSrcZ);
		Vec3 d(angles[i].fDetSX, angles[i].fDetSY, angles[i].fDetSZ);



		double fScale;
		if (!params.bFDKWeighting) {
			// goal: 1/fDen^2 = || u v (s-d) ||^2 / ( |cross(u,v)| * || u v (s-x) ||^2 )
			// fDen = ( sqrt(|cross(u,v)|) * || u v (s-x) || ) / || u v (s-d) || 
			// i.e. scale = sqrt(|cross(u,v)|) * / || u v (s-d) ||


			// NB: for cross(u,v) we invert the volume scaling (for the voxel
			// size normalization) to get the proper dimensions for
			// the scaling of the adjoint

			fScale = sqrt(scaled_cross3(u,v,Vec3(params.fVolScaleX,params.fVolScaleY,params.fVolScaleZ)).norm()) / det3(u, v, s-d);
		} else {
			// goal: 1/fDen = || u v s || / || u v (s-x) ||
			// fDen = || u v (s-x) || / || u v s ||
			// i.e., scale = 1 / || u v s ||

			fScale = 1.0 / det3(u, v, s);
		}

		p[i].fNumU.w = fScale * det3(s,v,d);
		p[i].fNumU.x = fScale * det3x(v,s-d);
		p[i].fNumU.y = fScale * det3y(v,s-d);
		p[i].fNumU.z = fScale * det3z(v,s-d);
		p[i].fNumV.w = -fScale * det3(s,u,d);
		p[i].fNumV.x = -fScale * det3x(u,s-d);
		p[i].fNumV.y = -fScale * det3y(u,s-d);
		p[i].fNumV.z = -fScale * det3z(u,s-d);
		p[i].fDen.w = fScale * det3(u, v, s); // == 1.0 for FDK
		p[i].fDen.x = -fScale * det3x(u, v);
		p[i].fDen.y = -fScale * det3y(u, v);
		p[i].fDen.z = -fScale * det3z(u, v);
	}

	// TODO: Check for errors
	hipMemcpyToSymbol(HIP_SYMBOL(gC_C), p, iProjAngles*sizeof(DevConeParams), 0, hipMemcpyHostToDevice);

	delete[] p;

	return true;
}


bool ConeBP_Array(hipPitchedPtr D_volumeData,
                  hipArray *D_projArray,
                  const SDimensions3D& dims, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	bindProjDataTexture(D_projArray);

	float fOutputScale;
	if (params.bFDKWeighting) {
		// NB: assuming cube voxels here
		fOutputScale = params.fOutputScale / (params.fVolScaleX);
	} else {
		fOutputScale = params.fOutputScale * (params.fVolScaleX * params.fVolScaleY * params.fVolScaleZ);
	}

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		bool ok = transferConstants(angles, angleCount, params);
		if (!ok)
			return false;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX/1+g_volBlockX-1)/(g_volBlockX))*((dims.iVolY/1+1*g_volBlockY-1)/(1*g_volBlockY)), 8);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
		for (unsigned int startZ = 0; startZ < dims.iVolZ; startZ += 8*g_volBlockZ) {

		// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (params.bFDKWeighting)
				dev_cone_BP<true><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, startZ, dims, fOutputScale);
			else if (params.iRaysPerVoxelDim == 1)
				dev_cone_BP<false><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, startZ, dims, fOutputScale);
			else
				dev_cone_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, startZ, dims, params.iRaysPerVoxelDim, fOutputScale);
		}
		}

		cudaTextForceKernelsCompletion();

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}


	return true;
}

bool ConeBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = ConeBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}


}
