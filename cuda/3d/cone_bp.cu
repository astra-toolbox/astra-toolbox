#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "cone_fp.h"
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneProjTexture;

namespace astraCUDA3d {

#define ZSIZE 6
static const unsigned int g_volBlockZ = ZSIZE;

static const unsigned int g_anglesPerBlock = 32;
static const unsigned int g_volBlockX = 16;
static const unsigned int g_volBlockY = 32;

static const unsigned g_MaxAngles = 1024;

__constant__ float gC_C[12*g_MaxAngles];

bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneProjTexture.filterMode = hipFilterModeLinear;
	gT_coneProjTexture.normalized = false;

	hipBindTextureToArray(gT_coneProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


//__launch_bounds__(32*16, 4)
template<bool FDKWEIGHT>
__global__ void dev_cone_BP(void* D_volData, unsigned int volPitch, int startAngle,
                            int angleOffset, const astraCUDA3d::SDimensions3D dims,
                            float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
	//            y = z



	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	const float fX = X - 0.5f*dims.iVolX + 0.5f;
	const float fY = Y - 0.5f*dims.iVolY + 0.5f;
	const float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	float Z[ZSIZE];
	for(int i=0; i < ZSIZE; i++)
		Z[i] = 0.0f;


	{
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu  = make_float4(gC_C[12*angle+0], gC_C[12*angle+1], gC_C[12*angle+2], gC_C[12*angle+3]);
			float4 fCv  = make_float4(gC_C[12*angle+4], gC_C[12*angle+5], gC_C[12*angle+6], gC_C[12*angle+7]);
			float4 fCd  = make_float4(gC_C[12*angle+8], gC_C[12*angle+9], gC_C[12*angle+10], gC_C[12*angle+11]);

			float fUNum = fCu.w + fX * fCu.x + fY * fCu.y + fZ * fCu.z;
			float fVNum = fCv.w + fX * fCv.x + fY * fCv.y + fZ * fCv.z;
			float fDen  = fCd.w + fX * fCd.x + fY * fCd.y + fZ * fCd.z;

			float fU,fV, fr;

			for (int idx = 0; idx < ZSIZE; idx++)
			{
				fr = __fdividef(1.0f, fDen);
				fU = fUNum * fr;
				fV = fVNum * fr;
				float fVal = tex3D(gT_coneProjTexture, fU, fAngle, fV);
				if (FDKWEIGHT)
					Z[idx] += fr*fr*fVal;
				else
					Z[idx] += fVal;

				fUNum += fCu.z;
				fVNum += fCv.z;
				fDen  += fCd.z;
			}
		}
	}

	int endZ = ZSIZE;
	if (endZ > dims.iVolZ - startZ)
		endZ = dims.iVolZ - startZ;

	for(int i=0; i < endZ; i++)
		volData[((startZ+i)*dims.iVolY+Y)*volPitch+X] += Z[i] * fOutputScale;
} //End kernel



// supersampling version
__global__ void dev_cone_BP_SS(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, const SDimensions3D dims, int iRaysPerVoxelDim, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	const float fSubStep = 1.0f/iRaysPerVoxelDim;

	fOutputScale /= (iRaysPerVoxelDim*iRaysPerVoxelDim*iRaysPerVoxelDim);


	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			const float fCux = gC_C[12*angle+0];
			const float fCuy = gC_C[12*angle+1];
			const float fCuz = gC_C[12*angle+2];
			const float fCuc = gC_C[12*angle+3];
			const float fCvx = gC_C[12*angle+4];
			const float fCvy = gC_C[12*angle+5];
			const float fCvz = gC_C[12*angle+6];
			const float fCvc = gC_C[12*angle+7];
			const float fCdx = gC_C[12*angle+8];
			const float fCdy = gC_C[12*angle+9];
			const float fCdz = gC_C[12*angle+10];
			const float fCdc = gC_C[12*angle+11];

			float fXs = fX;
			for (int iSubX = 0; iSubX < iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < iRaysPerVoxelDim; ++iSubZ) {

				const float fUNum = fCuc + fXs * fCux + fYs * fCuy + fZs * fCuz;
				const float fVNum = fCvc + fXs * fCvx + fYs * fCvy + fZs * fCvz;
				const float fDen = fCdc + fXs * fCdx + fYs * fCdy + fZs * fCdz;

				const float fU = fUNum / fDen;
				const float fV = fVNum / fDen;

				fVal += tex3D(gT_coneProjTexture, fU, fV, fAngle);

				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal * fOutputScale;
	}
}


bool ConeBP_Array(hipPitchedPtr D_volumeData,
                  hipArray *D_projArray,
                  const SDimensions3D& dims, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	bindProjDataTexture(D_projArray);

	float fOutputScale = params.fOutputScale * params.fVolScaleX * params.fVolScaleY * params.fVolScaleZ;

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		// transfer angles to constant memory
		float* tmp = new float[12*angleCount];


		// NB: We increment angles at the end of the loop body.


#define TRANSFER_TO_CONSTANT(expr,name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[12*i+name] = (expr) ; } while (0)

		TRANSFER_TO_CONSTANT( (angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetVY - (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetVZ , 0 );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetVZ -(angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetVX , 1 );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetVX - (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetVY , 2 );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSY*angles[i].fDetVZ - angles[i].fDetSZ*angles[i].fDetVY)*angles[i].fSrcX - (angles[i].fDetSX*angles[i].fDetVZ - angles[i].fDetSZ*angles[i].fDetVX)*angles[i].fSrcY + (angles[i].fDetSX*angles[i].fDetVY - angles[i].fDetSY*angles[i].fDetVX)*angles[i].fSrcZ , 3 );

		TRANSFER_TO_CONSTANT( (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetUZ-(angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetUY, 4 );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetUX - (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetUZ , 5 );
		TRANSFER_TO_CONSTANT((angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetUY-(angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetUX , 6 );
		TRANSFER_TO_CONSTANT( -(angles[i].fDetSY*angles[i].fDetUZ - angles[i].fDetSZ*angles[i].fDetUY)*angles[i].fSrcX + (angles[i].fDetSX*angles[i].fDetUZ - angles[i].fDetSZ*angles[i].fDetUX)*angles[i].fSrcY - (angles[i].fDetSX*angles[i].fDetUY - angles[i].fDetSY*angles[i].fDetUX)*angles[i].fSrcZ , 7 );

		TRANSFER_TO_CONSTANT( angles[i].fDetUY*angles[i].fDetVZ - angles[i].fDetUZ*angles[i].fDetVY , 8 );
		TRANSFER_TO_CONSTANT( angles[i].fDetUZ*angles[i].fDetVX - angles[i].fDetUX*angles[i].fDetVZ , 9 );
		TRANSFER_TO_CONSTANT( angles[i].fDetUX*angles[i].fDetVY - angles[i].fDetUY*angles[i].fDetVX , 10 );
		TRANSFER_TO_CONSTANT( -angles[i].fSrcX * (angles[i].fDetUY*angles[i].fDetVZ - angles[i].fDetUZ*angles[i].fDetVY) - angles[i].fSrcY * (angles[i].fDetUZ*angles[i].fDetVX - angles[i].fDetUX*angles[i].fDetVZ) - angles[i].fSrcZ * (angles[i].fDetUX*angles[i].fDetVY - angles[i].fDetUY*angles[i].fDetVX) , 11 );

#undef TRANSFER_TO_CONSTANT
		hipMemcpyToSymbol(HIP_SYMBOL(gC_C), tmp, angleCount*12*sizeof(float), 0, hipMemcpyHostToDevice); 

		delete[] tmp;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX/1+g_volBlockX-1)/(g_volBlockX))*((dims.iVolY/1+1*g_volBlockY-1)/(1*g_volBlockY)), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
		// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (params.bFDKWeighting)
				dev_cone_BP<true><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims, fOutputScale);
			else if (params.iRaysPerVoxelDim == 1)
				dev_cone_BP<false><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims, fOutputScale);
			else
				dev_cone_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims, params.iRaysPerVoxelDim, fOutputScale);
		}

		cudaTextForceKernelsCompletion();

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}


	return true;
}

bool ConeBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = ConeBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}


}

#ifdef STANDALONE
int main()
{
	astraCUDA3d::SDimensions3D dims;
	dims.iVolX = 512;
	dims.iVolY = 512;
	dims.iVolZ = 512;
	dims.iProjAngles = 496;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDetDim = 1;
	dims.iRaysPerVoxelDim = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

#if 0
	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
#if 0
		if (i == 128) {
			for (unsigned int j = 0; j < 256*256; ++j)
				slice[j] = 0.0f;
		}
#endif 
	}
#endif


	astraCUDA3d::SConeProjection angle[512];
	angle[0].fSrcX = -5120;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 512;
	angle[0].fDetSY = -256;
	angle[0].fDetSZ = -256;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = 1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 512; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/512);
		ROTATE0(DetS, i, i*2*M_PI/512);
		ROTATE0(DetU, i, i*2*M_PI/512);
		ROTATE0(DetV, i, i*2*M_PI/512);
	}
#undef ROTATE0

#if 0
	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);
#endif
#if 0
	float* bufs = new float[180*512];

	for (int i = 0; i < 512; ++i) {
		hipMemcpy(bufs, ((float*)projData.ptr)+180*512*i, 180*512*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

		char fname[20];
		sprintf(fname, "sino%03d.png", i);
		saveImage(fname, 180, 512, bufs);
	}

	float* bufp = new float[512*512];

	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "proj%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif		
#if 0
	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 0.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
	}
#endif

	astraCUDA3d::ConeBP(volData, projData, dims, angle, 1.0f);
#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)volData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", volData.pitch, volData.xsize, volData.ysize);

		char fname[20];
		sprintf(fname, "vol%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

}
#endif
