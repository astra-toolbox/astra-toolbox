#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>

namespace astraCUDA3d {

static const unsigned int g_volBlockZ = 6;

static const unsigned int g_anglesPerBlock = 32;
static const unsigned int g_volBlockX = 16;
static const unsigned int g_volBlockY = 32;

static const unsigned g_MaxAngles = 1024;

struct DevConeParams {
	float4 fNumU;
	float4 fNumV;
	float4 fDen;
};

__constant__ DevConeParams gC_C[g_MaxAngles];

//__launch_bounds__(32*16, 4)
template<bool FDKWEIGHT, unsigned int ZSIZE>
__global__ void dev_cone_BP(void* D_volData, unsigned int volPitch,
                            hipTextureObject_t tex,
                            int startAngle, int angleOffset,
                            const astraCUDA3d::SDimensions3D dims,
                            float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
	//            y = z



	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	const float fX = X - 0.5f*dims.iVolX + 0.5f;
	const float fY = Y - 0.5f*dims.iVolY + 0.5f;
	const float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	float Z[ZSIZE];
	for(int i=0; i < ZSIZE; i++)
		Z[i] = 0.0f;


	{
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu  = gC_C[angle].fNumU;
			float4 fCv  = gC_C[angle].fNumV;
			float4 fCd  = gC_C[angle].fDen;

			float fUNum = fCu.w + fX * fCu.x + fY * fCu.y + fZ * fCu.z;
			float fVNum = fCv.w + fX * fCv.x + fY * fCv.y + fZ * fCv.z;
			float fDen  = (FDKWEIGHT ? 1.0f : fCd.w) + fX * fCd.x + fY * fCd.y + fZ * fCd.z;

			float fU,fV, fr;

			for (int idx = 0; idx < ZSIZE; idx++)
			{
				fr = __fdividef(1.0f, fDen);
				fU = fUNum * fr;
				fV = fVNum * fr;
				float fVal = tex3D<float>(tex, fU, fAngle, fV);
				Z[idx] += fr*fr*fVal;

				fUNum += fCu.z;
				fVNum += fCv.z;
				fDen  += fCd.z;
			}
		}
	}

	int endZ = ZSIZE;
	if (endZ > dims.iVolZ - startZ)
		endZ = dims.iVolZ - startZ;

	for(int i=0; i < endZ; i++)
		volData[((startZ+i)*dims.iVolY+Y)*volPitch+X] += Z[i] * fOutputScale;
} //End kernel



// supersampling version
__global__ void dev_cone_BP_SS(void* D_volData, unsigned int volPitch, hipTextureObject_t tex, int startAngle, int angleOffset, const SDimensions3D dims, int iRaysPerVoxelDim, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	const float fSubStep = 1.0f/iRaysPerVoxelDim;

	fOutputScale /= (iRaysPerVoxelDim*iRaysPerVoxelDim*iRaysPerVoxelDim);


	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu  = gC_C[angle].fNumU;
			float4 fCv  = gC_C[angle].fNumV;
			float4 fCd  = gC_C[angle].fDen;

			float fXs = fX;
			for (int iSubX = 0; iSubX < iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < iRaysPerVoxelDim; ++iSubZ) {

				const float fUNum = fCu.w + fXs * fCu.x + fYs * fCu.y + fZs * fCu.z;
				const float fVNum = fCv.w + fXs * fCv.x + fYs * fCv.y + fZs * fCv.z;
				const float fDen  = fCd.w + fXs * fCd.x + fYs * fCd.y + fZs * fCd.z;

				const float fr = __fdividef(1.0f, fDen);
				const float fU = fUNum * fr;
				const float fV = fVNum * fr;

				fVal += tex3D<float>(tex, fU, fAngle, fV) * fr * fr;

				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal * fOutputScale;
	}
}


bool transferConstants(const SConeProjection* angles, unsigned int iProjAngles, const SProjectorParams3D& params)
{
	DevConeParams *p = new DevConeParams[iProjAngles];

	// We need three things in the kernel:
	// projected coordinates of pixels on the detector:

	// u: || (x-s) v (s-d) || / || u v (s-x) ||
	// v: -|| u (x-s) (s-d) || / || u v (s-x) ||

	// ray density weighting factor for the adjoint
	// || u v (s-d) ||^2 / ( |cross(u,v)| * || u v (s-x) ||^2 )

	// FDK weighting factor
	// ( || u v s || / || u v (s-x) || ) ^ 2

	// Since u and v are ratios with the same denominator, we have
	// a degree of freedom to scale the denominator. We use that to make
	// the square of the denominator equal to the relevant weighting factor.


	for (unsigned int i = 0; i < iProjAngles; ++i) {
		Vec3 u(angles[i].fDetUX, angles[i].fDetUY, angles[i].fDetUZ);
		Vec3 v(angles[i].fDetVX, angles[i].fDetVY, angles[i].fDetVZ);
		Vec3 s(angles[i].fSrcX, angles[i].fSrcY, angles[i].fSrcZ);
		Vec3 d(angles[i].fDetSX, angles[i].fDetSY, angles[i].fDetSZ);



		double fScale;
		if (!params.bFDKWeighting) {
			// goal: 1/fDen^2 = || u v (s-d) ||^2 / ( |cross(u,v)| * || u v (s-x) ||^2 )
			// fDen = ( sqrt(|cross(u,v)|) * || u v (s-x) || ) / || u v (s-d) || 
			// i.e. scale = sqrt(|cross(u,v)|) * / || u v (s-d) ||


			// NB: for cross(u,v) we invert the volume scaling (for the voxel
			// size normalization) to get the proper dimensions for
			// the scaling of the adjoint

			fScale = sqrt(scaled_cross3(u,v,Vec3(params.fVolScaleX,params.fVolScaleY,params.fVolScaleZ)).norm()) / det3(u, v, s-d);
		} else {
			// goal: 1/fDen = || u v s || / || u v (s-x) ||
			// fDen = || u v (s-x) || / || u v s ||
			// i.e., scale = 1 / || u v s ||

			fScale = 1.0 / det3(u, v, s);
		}

		p[i].fNumU.w = fScale * det3(s,v,d);
		p[i].fNumU.x = fScale * det3x(v,s-d);
		p[i].fNumU.y = fScale * det3y(v,s-d);
		p[i].fNumU.z = fScale * det3z(v,s-d);
		p[i].fNumV.w = -fScale * det3(s,u,d);
		p[i].fNumV.x = -fScale * det3x(u,s-d);
		p[i].fNumV.y = -fScale * det3y(u,s-d);
		p[i].fNumV.z = -fScale * det3z(u,s-d);
		p[i].fDen.w = fScale * det3(u, v, s); // == 1.0 for FDK
		p[i].fDen.x = -fScale * det3x(u, v);
		p[i].fDen.y = -fScale * det3y(u, v);
		p[i].fDen.z = -fScale * det3z(u, v);
	}

	// TODO: Check for errors
	hipMemcpyToSymbol(HIP_SYMBOL(gC_C), p, iProjAngles*sizeof(DevConeParams), 0, hipMemcpyHostToDevice);

	delete[] p;

	return true;
}


bool ConeBP_Array(hipPitchedPtr D_volumeData,
                  hipArray *D_projArray,
                  const SDimensions3D& dims, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	hipTextureObject_t D_texObj;
	if (!createTextureObject3D(D_projArray, D_texObj))
		return false;

	float fOutputScale;
	if (params.bFDKWeighting) {
		// NB: assuming cube voxels here
		fOutputScale = params.fOutputScale / (params.fVolScaleX);
	} else {
		fOutputScale = params.fOutputScale * (params.fVolScaleX * params.fVolScaleY * params.fVolScaleZ);
	}

	bool ok = true;

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		ok = transferConstants(angles, angleCount, params);
		if (!ok)
			break;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX/1+g_volBlockX-1)/(g_volBlockX))*((dims.iVolY/1+1*g_volBlockY-1)/(1*g_volBlockY)), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
		// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (params.bFDKWeighting) {
				if (dims.iVolZ == 1) {
					dev_cone_BP<true, 1><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				} else {
					dev_cone_BP<true, g_volBlockZ><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				}
			} else if (params.iRaysPerVoxelDim == 1) {
				if (dims.iVolZ == 1) {
					dev_cone_BP<false, 1><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				} else {
					dev_cone_BP<false, g_volBlockZ><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				}
			} else
				dev_cone_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, params.iRaysPerVoxelDim, fOutputScale);
		}

		// TODO: Consider not synchronizing here, if possible.
		ok = checkCuda(hipDeviceSynchronize(), "cone_bp");
		if (!ok)
			break;

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}

	hipDestroyTextureObject(D_texObj);

	return ok;
}

bool ConeBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = ConeBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}


}
