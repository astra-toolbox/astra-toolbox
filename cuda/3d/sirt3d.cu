#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "sirt3d.h"
#include "util3d.h"
#include "arith3d.h"
#include "cone_fp.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

namespace astraCUDA3d {

SIRT::SIRT() : ReconAlgo3D()
{
	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_projData.ptr = 0;
	D_tmpData.ptr = 0;

	D_lineWeight.ptr = 0;
	D_pixelWeight.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	useMinConstraint = false;
	useMaxConstraint = false;

	fRelaxation = 1.0f;
}


SIRT::~SIRT()
{
	reset();
}

void SIRT::reset()
{
	hipFree(D_projData.ptr);
	hipFree(D_tmpData.ptr);
	hipFree(D_lineWeight.ptr);
	hipFree(D_pixelWeight.ptr);

	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_projData.ptr = 0;
	D_tmpData.ptr = 0;

	D_lineWeight.ptr = 0;
	D_pixelWeight.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	fRelaxation = 1.0f;

	ReconAlgo3D::reset();
}

bool SIRT::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool SIRT::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool SIRT::init()
{
	D_pixelWeight = allocateVolumeData(dims);
	zeroVolumeData(D_pixelWeight, dims);

	D_tmpData = allocateVolumeData(dims);
	zeroVolumeData(D_tmpData, dims);

	D_projData = allocateProjectionData(dims);
	zeroProjectionData(D_projData, dims);

	D_lineWeight = allocateProjectionData(dims);
	zeroProjectionData(D_lineWeight, dims);

	// We can't precompute lineWeights and pixelWeights when using a mask
	if (!useVolumeMask && !useSinogramMask)
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool SIRT::setMinConstraint(float fMin)
{
	fMinConstraint = fMin;
	useMinConstraint = true;
	return true;
}

bool SIRT::setMaxConstraint(float fMax)
{
	fMaxConstraint = fMax;
	useMaxConstraint = true;
	return true;
}

bool SIRT::precomputeWeights()
{
	zeroProjectionData(D_lineWeight, dims);
	if (useVolumeMask) {
		callFP(D_maskData, D_lineWeight, 1.0f);
	} else {
		processVol3D<opSet>(D_tmpData, 1.0f, dims);
		callFP(D_tmpData, D_lineWeight, 1.0f);
	}
	processSino3D<opInvert>(D_lineWeight, dims);

	if (useSinogramMask) {
		// scale line weights with sinogram mask to zero out masked sinogram pixels
		processSino3D<opMul>(D_lineWeight, D_smaskData, dims);
	}

	zeroVolumeData(D_pixelWeight, dims);

	if (useSinogramMask) {
		callBP(D_pixelWeight, D_smaskData, 1.0f);
	} else {
		processSino3D<opSet>(D_projData, 1.0f, dims);
		callBP(D_pixelWeight, D_projData, 1.0f);
	}
#if 0
	float* bufp = new float[512*512];

	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "ray%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif

#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_pixelWeight.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "pix%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif
	processVol3D<opInvert>(D_pixelWeight, dims);

	if (useVolumeMask) {
		// scale pixel weights with mask to zero out masked pixels
		processVol3D<opMul>(D_pixelWeight, D_maskData, dims);
	}
	processVol3D<opMul>(D_pixelWeight, fRelaxation, dims);


	return true;
}


bool SIRT::setVolumeMask(hipPitchedPtr& _D_maskData)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;

	return true;
}

bool SIRT::setSinogramMask(hipPitchedPtr& _D_smaskData)
{
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;

	return true;
}

bool SIRT::setBuffers(hipPitchedPtr& _D_volumeData,
                      hipPitchedPtr& _D_projData)
{
	D_volumeData = _D_volumeData;
	D_sinoData = _D_projData;

	return true;
}

bool SIRT::iterate(unsigned int iterations)
{
	shouldAbort = false;

	if (useVolumeMask || useSinogramMask)
		precomputeWeights();

#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_pixelWeight.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "pix%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif
#if 0
	float* bufp = new float[512*512];

	for (int i = 0; i < 100; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_lineWeight.ptr)+100*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "ray%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif


	// iteration
	for (unsigned int iter = 0; iter < iterations && !shouldAbort; ++iter) {
		// copy sinogram to projection data
		duplicateProjectionData(D_projData, D_sinoData, dims);

		// do FP, subtracting projection from sinogram
		if (useVolumeMask) {
				duplicateVolumeData(D_tmpData, D_volumeData, dims);
				processVol3D<opMul>(D_tmpData, D_maskData, dims);
				callFP(D_tmpData, D_projData, -1.0f);
		} else {
				callFP(D_volumeData, D_projData, -1.0f);
		}

		processSino3D<opMul>(D_projData, D_lineWeight, dims);

		zeroVolumeData(D_tmpData, dims);
#if 0
	float* bufp = new float[512*512];
	printf("Dumping projData: %p\n", (void*)D_projData.ptr);
	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "diff%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif


		callBP(D_tmpData, D_projData, 1.0f);
#if 0
	printf("Dumping tmpData: %p\n", (void*)D_tmpData.ptr);
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_tmpData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "add%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

		// pixel weights also contain the volume mask and relaxation factor
		processVol3D<opAddMul>(D_volumeData, D_tmpData, D_pixelWeight, dims);

		if (useMinConstraint)
			processVol3D<opClampMin>(D_volumeData, fMinConstraint, dims);
		if (useMaxConstraint)
			processVol3D<opClampMax>(D_volumeData, fMaxConstraint, dims);
	}

	return true;
}

float SIRT::computeDiffNorm()
{
	// copy sinogram to projection data
	duplicateProjectionData(D_projData, D_sinoData, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_tmpData, D_volumeData, dims);
			processVol3D<opMul>(D_tmpData, D_maskData, dims);
			callFP(D_tmpData, D_projData, -1.0f);
	} else {
			callFP(D_volumeData, D_projData, -1.0f);
	}

	float s = dotProduct3D(D_projData, dims.iProjU, dims.iProjAngles, dims.iProjV);
	return sqrt(s);
}


bool doSIRT(hipPitchedPtr& D_volumeData, 
            hipPitchedPtr& D_sinoData,
            hipPitchedPtr& D_maskData,
            const SDimensions3D& dims, const SConeProjection* angles,
            unsigned int iterations)
{
	SIRT sirt;
	bool ok = true;

	ok &= sirt.setConeGeometry(dims, angles, SProjectorParams3D());
	if (D_maskData.ptr)
		ok &= sirt.enableVolumeMask();

	if (!ok)
		return false;

	ok = sirt.init();
	if (!ok)
		return false;

	if (D_maskData.ptr)
		ok &= sirt.setVolumeMask(D_maskData);

	ok &= sirt.setBuffers(D_volumeData, D_sinoData);
	if (!ok)
		return false;

	ok = sirt.iterate(iterations);

	return ok;
}

}

#ifdef STANDALONE

using namespace astraCUDA3d;

int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 100;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	SConeProjection angle[100];
	angle[0].fSrcX = -2905.6;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 694.4;
	angle[0].fDetSY = -122.4704;
	angle[0].fDetSZ = -122.4704;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = .4784;
	//angle[0].fDetUY = .5;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = .4784;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 100; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/100);
		ROTATE0(DetS, i, i*2*M_PI/100);
		ROTATE0(DetU, i, i*2*M_PI/100);
		ROTATE0(DetV, i, i*2*M_PI/100);
	}
#undef ROTATE0


	hipPitchedPtr volData = allocateVolumeData(dims);
	hipPitchedPtr projData = allocateProjectionData(dims);
	zeroProjectionData(projData, dims);

	float* pbuf = new float[100*512*512];
	copyProjectionsFromDevice(pbuf, projData, dims);
	copyProjectionsToDevice(pbuf, projData, dims);
	delete[] pbuf;

#if 0
	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256; ++i) {
		for (unsigned int y = 0; y < 256; ++y)
			for (unsigned int x = 0; x < 256; ++x)
				slice[y*256+x] = (i-127.5)*(i-127.5)+(y-127.5)*(y-127.5)+(x-127.5)*(x-127.5) < 4900 ? 1.0f : 0.0f;

		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
	}
	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

#else

	for (int i = 0; i < 100; ++i) {
		char fname[32];
		sprintf(fname, "Tiffs/%04d.png", 4*i);
		unsigned int w,h;
		float* bufp = loadImage(fname, w,h);

		for (int j = 0; j < 512*512; ++j) {
			float v = bufp[j];
			if (v > 236.0f) v = 236.0f;
			v = logf(236.0f / v);
			bufp[j] = 256*v;
		}

		for (int j = 0; j < 512; ++j) {
			hipMemcpy(((float*)projData.ptr)+100*512*j+512*i, bufp+512*j, 512*sizeof(float), hipMemcpyHostToDevice);
		}

		delete[] bufp;

	}
#endif

#if 0
	float* bufs = new float[100*512];

	for (int i = 0; i < 512; ++i) {
		hipMemcpy(bufs, ((float*)projData.ptr)+100*512*i, 100*512*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

		char fname[20];
		sprintf(fname, "sino%03d.png", i);
		saveImage(fname, 100, 512, bufs);
	}

	float* bufp = new float[512*512];

	for (int i = 0; i < 100; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)projData.ptr)+100*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "proj%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif

	zeroVolumeData(volData, dims);

	hipPitchedPtr maskData;
	maskData.ptr = 0;

	astraCUDA3d::doSIRT(volData, projData, maskData, dims, angle, 50);
#if 1
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)volData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "vol%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

	return 0;
}
#endif

