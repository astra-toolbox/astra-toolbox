#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneVolumeTexture;

namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
static const unsigned int g_blockSlices = 32;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_SrcZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];


bool bindVolumeDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneVolumeTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneVolumeTexture.filterMode = hipFilterModeLinear;
	gT_coneVolumeTexture.normalized = false;

	hipBindTextureToArray(gT_coneVolumeTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


// x=0, y=1, z=2
struct DIR_X {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return x; }
	__device__ float c1(float x, float y, float z) const { return y; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneVolumeTexture, f0, f1, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f0; }
	__device__ float y(float f0, float f1, float f2) const { return f1; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// y=0, x=1, z=2
struct DIR_Y {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return y; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneVolumeTexture, f1, f0, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f0; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// z=0, x=1, y=2
struct DIR_Z {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float c0(float x, float y, float z) const { return z; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return y; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_coneVolumeTexture, f1, f2, f0); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f2; }
	__device__ float z(float f0, float f1, float f2) const { return f0; }
};

struct SCALE_CUBE {
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1+a2*a2+1.0f) * fOutputScale; }
};

struct SCALE_NONCUBE {
	float fScale1;
	float fScale2;
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*fScale1+a2*a2*fScale2+1.0f) * fOutputScale; }
};


	// threadIdx: x = ??? detector  (u?)
	//            y = relative angle

	// blockIdx:  x = ??? detector  (u+v?)
    //            y = angle block

template<class COORD, class SCALE>
__global__ void cone_FP_t(float* D_projData, unsigned int projPitch,
                          unsigned int startSlice,
                          unsigned int startAngle, unsigned int endAngle,
                          const SDimensions3D dims,
                          SCALE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * x + (by) */
		/*        (z)   (az)       (bz) */

		const float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;

		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

template<class COORD>
__global__ void cone_FP_SS_t(float* D_projData, unsigned int projPitch,
                             unsigned int startSlice,
                             unsigned int startAngle, unsigned int endAngle,
                             const SDimensions3D dims, int iRaysPerDetDim,
                             SCALE_NONCUBE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fSrcZ = gC_SrcZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	const float fSubStep = 1.0f/iRaysPerDetDim;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray from Src to (detectorU,detectorV) from */
		/* X = startSlice to X = endSlice                   */

		float fV = 0.0f;

		float fdU = detectorU - 0.5f + 0.5f*fSubStep;
		for (int iSubU = 0; iSubU < iRaysPerDetDim; ++iSubU, fdU+=fSubStep) {
		float fdV = detectorV - 0.5f + 0.5f*fSubStep;
		for (int iSubV = 0; iSubV < iRaysPerDetDim; ++iSubV, fdV+=fSubStep) {

		const float fDetX = fDetSX + fdU*fDetUX + fdV*fDetVX;
		const float fDetY = fDetSY + fdU*fDetUY + fdV*fDetVY;
		const float fDetZ = fDetSZ + fdU*fDetUZ + fdV*fDetVZ;

		/*        (x)   ( 1)       ( 0) */
		/* ray:   (y) = (ay) * x + (by) */
		/*        (z)   (az)       (bz) */

		const float a1 = (c.c1(fSrcX,fSrcY,fSrcZ) - c.c1(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float a2 = (c.c2(fSrcX,fSrcY,fSrcZ) - c.c2(fDetX,fDetY,fDetZ)) / (c.c0(fSrcX,fSrcY,fSrcZ) - c.c0(fDetX,fDetY,fDetZ));
		const float b1 = c.c1(fSrcX,fSrcY,fSrcZ) - a1 * c.c0(fSrcX,fSrcY,fSrcZ);
		const float b2 = c.c2(fSrcX,fSrcY,fSrcZ) - a2 * c.c0(fSrcX,fSrcY,fSrcZ);

		const float fDistCorr = sc.scale(a1, a2);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;
		fV += fVal;

		}
		}

		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fV / (iRaysPerDetDim * iRaysPerDetDim);
	}
}


bool ConeFP_Array_internal(hipPitchedPtr D_projData,
                  const SDimensions3D& dims, unsigned int angleCount, const SConeProjection* angles,
                  const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[angleCount];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(SrcZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.fVolScaleX / params.fVolScaleY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.fVolScaleX / params.fVolScaleZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;

	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != angleCount) {
			float dX = fabsf(angles[a].fSrcX - (angles[a].fDetSX + dims.iProjU*angles[a].fDetUX*0.5f + dims.iProjV*angles[a].fDetVX*0.5f));
			float dY = fabsf(angles[a].fSrcY - (angles[a].fDetSY + dims.iProjU*angles[a].fDetUY*0.5f + dims.iProjV*angles[a].fDetVY*0.5f));
			float dZ = fabsf(angles[a].fSrcZ - (angles[a].fDetSZ + dims.iProjU*angles[a].fDetUZ*0.5f + dims.iProjV*angles[a].fDetVZ*0.5f));

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeX);
						else
							cone_FP_SS_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeX);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeY);
						else
							cone_FP_SS_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeY);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							if (cube)
								cone_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
							else
								cone_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeZ);
						else
							cone_FP_SS_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeZ);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();

	// printf("%f\n", toc(t));

	return true;
}


bool ConeFP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer volume to array

	hipArray* cuArray = allocateVolumeArray(dims);
	transferVolumeToArray(D_volumeData, cuArray, dims);
	bindVolumeDataTexture(cuArray);

	bool ret;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		hipPitchedPtr D_subprojData = D_projData;
		D_subprojData.ptr = (char*)D_projData.ptr + iAngle * D_projData.pitch;

		ret = ConeFP_Array_internal(D_subprojData,
		                            dims, iEndAngle - iAngle, angles + iAngle,
		                            params);
		if (!ret)
			break;
	}

	hipFreeArray(cuArray);

	return ret;
}


}

#ifdef STANDALONE
int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 32;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjV;
	extentP.depth = dims.iProjAngles;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipError_t err = hipMemcpy3D(&p);
		assert(!err);
	}


	SConeProjection angle[32];
	angle[0].fSrcX = -1536;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 200;

	angle[0].fDetSX = 512;
	angle[0].fDetSY = -256;
	angle[0].fDetSZ = -256;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = 1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 32; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*1*M_PI/180);
		ROTATE0(DetS, i, i*1*M_PI/180);
		ROTATE0(DetU, i, i*1*M_PI/180);
		ROTATE0(DetV, i, i*1*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

	float* buf = new float[512*512];

	hipMemcpy(buf, ((float*)projData.ptr)+512*512*8, 512*512*sizeof(float), hipMemcpyDeviceToHost);

	printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

	saveImage("proj.png", 512, 512, buf);
	

}
#endif
