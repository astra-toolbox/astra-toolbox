#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"
#include "astra/cuda/3d/arith3d.h"
#include "astra/cuda/3d/cone_bp.h"

#include "astra/cuda/2d/fft.h"

#include "astra/Logging.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>

namespace astraCUDA3d {

static const unsigned int g_anglesPerWeightBlock = 16;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 12000;

__constant__ float gC_angle[g_MaxAngles];

bool checkCufft(hipfftResult err, const char *msg)
{
	if (err != HIPFFT_SUCCESS) {
		ASTRA_ERROR("%s: CUFFT error %d.", msg, err);
		return false;
	} else {
		return true;
	}
}



// TODO: To support non-cube voxels, preweighting needs per-view
// parameters. NB: Need to properly take into account the
// anisotropic volume normalization done for that too.


__global__ void devFDK_preweight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fZShift, float fDetUSize, float fDetVSize, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the ray(s) to
	// our detector pixel(s).

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	const float fT = fCentralRayLength * fCentralRayLength + fU * fU;

	float fV = (startDetectorV - 0.5f*dims.iProjV + 0.5f) * fDetVSize + fZShift;

	// Contributions to the weighting factors:
	// fCentralRayLength / fRayLength   : the main FDK preweighting factor
	// fSrcOrigin / (fDetUSize * fCentralRayLength)
	//                                  : to adjust the filter to the det width
	// pi / (2 * iProjAngles)           : scaling of the integral over angles

	const float fW2 = fCentralRayLength / (fDetUSize * fSrcOrigin);
	const float fW = fCentralRayLength * fW2 * (M_PI / 2.0f) / (float)dims.iProjAngles;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		const float fRayLength = sqrtf(fT + fV * fV);

		const float fWeight = fW / fRayLength;

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

		fV += fDetVSize;
	}
}

__global__ void devFDK_ParkerWeight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fDetUSize, float fCentralFanAngle, float fScale, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the projection
	// of our ray onto the central slice

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	// TODO: Detector pixel size
	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	const float fGamma = atanf(fU / fCentralRayLength);
	float fBeta = gC_angle[angle];

	// compute the weight depending on the location in the central fan's radon
	// space
	float fWeight;

	if (fBeta <= 0.0f) {
		fWeight = 0.0f;
	} else if (fBeta <= 2.0f*(fCentralFanAngle + fGamma)) {
		fWeight = sinf((M_PI / 4.0f) * fBeta / (fCentralFanAngle + fGamma));
		fWeight *= fWeight;
	} else if (fBeta <= M_PI + 2*fGamma) {
		fWeight = 1.0f;
	} else if (fBeta <= M_PI + 2*fCentralFanAngle) {
		fWeight = sinf((M_PI / 4.0f) * (M_PI + 2.0f*fCentralFanAngle - fBeta) / (fCentralFanAngle - fGamma));
		fWeight *= fWeight;
	} else {
		fWeight = 0.0f;
	}

	fWeight *= fScale;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

	}
}



// Perform the FDK pre-weighting and filtering
bool FDK_PreWeight(hipPitchedPtr D_projData,
                float fSrcOrigin, float fDetOrigin,
                float fZShift,
                float fDetUSize, float fDetVSize,
				bool bShortScan,
                const SDimensions3D& dims, const float* angles)
{
	// The pre-weighting factor for a ray is the cosine of the angle between
	// the central line and the ray.

	dim3 dimBlock(g_detBlockU, g_anglesPerWeightBlock);
	dim3 dimGrid( ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
	              (dims.iProjAngles+g_anglesPerWeightBlock-1)/g_anglesPerWeightBlock);

	int projPitch = D_projData.pitch/sizeof(float);

	devFDK_preweight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fZShift, fDetUSize, fDetVSize, dims);

	if (!checkCuda(hipDeviceSynchronize(), "FDK_PreWeight"))
		return false;

	if (bShortScan && dims.iProjAngles > 1) {
		ASTRA_DEBUG("Doing Parker weighting");
		// We do short-scan Parker weighting

		// First, determine (in a very basic way) the interval that's
		// been scanned. We assume angles[0] is one of the endpoints of the
		// range.
		float fdA = angles[1] - angles[0];

		while (fdA < -M_PI)
			fdA += 2*M_PI;
		while (fdA >= M_PI)
			fdA -= 2*M_PI;

		float fAngleBase;
		if (fdA >= 0.0f) {
			// going up from angles[0]
			fAngleBase = angles[0];
			ASTRA_DEBUG("Second angle >= first angle, so assuming angles are incrementing");
		} else {
			// going down from angles[0]
			fAngleBase = angles[dims.iProjAngles - 1];
			ASTRA_DEBUG("Second angle < first angle, so assuming angles are decrementing");
		}

		// We pick the lowest end of the range, and then
		// move all angles so they fall in [0,2pi)

		float *fRelAngles = new float[dims.iProjAngles];
		for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
			float f = angles[i] - fAngleBase;
			while (f >= 2*M_PI)
				f -= 2*M_PI;
			while (f < 0)
				f += 2*M_PI;
			fRelAngles[i] = f;
		}

		float fRange = fabs(fRelAngles[dims.iProjAngles-1] - fRelAngles[0]);
		// Adjust for discretisation
		fRange /= dims.iProjAngles - 1;
		fRange *= dims.iProjAngles;

		ASTRA_DEBUG("Assuming angles are linearly ordered and equally spaced for Parker weighting. Angular range %f radians", fRange);
		float fScale = fRange / M_PI;

		hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle), fRelAngles,
		                                    dims.iProjAngles*sizeof(float), 0,
		                                    hipMemcpyHostToDevice);
		assert(!e1);
		delete[] fRelAngles;

		float fCentralFanAngle = fabs(atanf(fDetUSize * (dims.iProjU*0.5f) /
		                               (fSrcOrigin + fDetOrigin)));

		if (fRange + 1e-3 < M_PI + 2*fCentralFanAngle) {
			ASTRA_WARN("Angular range (%f rad) smaller than Parker weighting range (%f rad)", fRange, M_PI + 2*fCentralFanAngle);
		}

		devFDK_ParkerWeight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fDetUSize, fCentralFanAngle, fScale, dims);

		if (!checkCuda(hipDeviceSynchronize(), "FDK_PreWeight ParkerWeight"))
			return false;
	}

	return true;
}

bool FDK_Filter(hipPitchedPtr D_projData,
                const float *pfFilter,
                const SDimensions3D& dims)
{
	// The filtering is a regular ramp filter per detector line.

	// Generate filter
	// TODO: Check errors
	int iPaddedDetCount = calcNextPowerOfTwo(2 * dims.iProjU);
	int iHalfFFTSize = astra::calcFFTFourierSize(iPaddedDetCount);


	hipfftComplex *pHostFilter = new hipfftComplex[dims.iProjAngles * iHalfFFTSize];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * dims.iProjAngles * iHalfFFTSize);

	if (pfFilter == 0){
		astra::SFilterConfig filter;
		filter.m_eType = astra::FILTER_RAMLAK;
		astraCUDA::genCuFFTFilter(filter, dims.iProjAngles, pHostFilter, iPaddedDetCount, iHalfFFTSize);
	} else {
		for (int i = 0; i < dims.iProjAngles * iHalfFFTSize; i++) {
			pHostFilter[i].x = pfFilter[i];
			pHostFilter[i].y = 0;
		}
	}

	hipfftComplex * D_filter;

	astraCUDA::allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_filter);
	astraCUDA::uploadComplexArrayToDevice(dims.iProjAngles, iHalfFFTSize, pHostFilter, D_filter);

	delete [] pHostFilter;


	hipfftHandle planF;
	hipfftHandle planI;

	if (!checkCufft(hipfftPlan1d(&planF, iPaddedDetCount, HIPFFT_R2C, dims.iProjAngles), "FDK filter FFT plan")) {
		astraCUDA::freeComplexOnDevice(D_filter);
		return false;
	}

	if (!checkCufft(hipfftPlan1d(&planI, iPaddedDetCount, HIPFFT_C2R, dims.iProjAngles), "FDK filter IFFT plan")) {
		astraCUDA::freeComplexOnDevice(D_filter);
		hipfftDestroy(planF);
		return false;
	}

	int projPitch = D_projData.pitch/sizeof(float);
	

	// We process one sinogram at a time.
	float* D_sinoData = (float*)D_projData.ptr;

	hipfftComplex * D_pcSinoFFT = NULL;
	astraCUDA::allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_pcSinoFFT);

	bool ok = true;

	float * D_pfPadded = NULL;
	size_t bufferMemSize = sizeof(float) * dims.iProjAngles * iPaddedDetCount;
	if (!checkCuda(hipMalloc((void **)&D_pfPadded, bufferMemSize), "FDK filter malloc")) {
		astraCUDA::freeComplexOnDevice(D_pcSinoFFT);
		astraCUDA::freeComplexOnDevice(D_filter);
		hipfftDestroy(planF);
		hipfftDestroy(planI);
		return false;
	}


	for (int v = 0; v < dims.iProjV; ++v) {
		if (!checkCuda(hipMemset(D_pfPadded, 0, bufferMemSize), "FDK filter memset")) {
			ok = false;
			break;
		}

		// pitched memcpy 2D to handle both source pitch and target padding
		if (!checkCuda(hipMemcpy2D(D_pfPadded, iPaddedDetCount*sizeof(float), D_sinoData, projPitch*sizeof(float), dims.iProjU*sizeof(float), dims.iProjAngles, hipMemcpyDeviceToDevice), "FDK filter memcpy")) {
			ok = false;
			break;
		}


		if (!checkCufft(hipfftExecR2C(planF, (hipfftReal *)D_pfPadded, D_pcSinoFFT), "FDK filter forward exec")) {
			ok = false;
			break;
		}

		astraCUDA::applyFilter(dims.iProjAngles, iHalfFFTSize, D_pcSinoFFT, D_filter);

		// Getting rid of the const qualifier is due to cufft API issue?
		if (!checkCufft(hipfftExecC2R(planI, (hipfftComplex *)D_pcSinoFFT,
	                      (hipfftReal *)D_pfPadded), "FDK filter inverse exec"))
		{
			ok = false;
			break;
		}

		astraCUDA::rescaleInverseFourier(dims.iProjAngles, iPaddedDetCount, D_pfPadded);

		if (!checkCuda(hipMemset(D_sinoData, 0, sizeof(float) * dims.iProjAngles * projPitch), "FDK filter memset")) {
			ok = false;
			break;
		}

		// pitched memcpy 2D to handle both source padding and target pitch
		if (!checkCuda(hipMemcpy2D(D_sinoData, projPitch*sizeof(float), D_pfPadded, iPaddedDetCount*sizeof(float), dims.iProjU*sizeof(float), dims.iProjAngles, hipMemcpyDeviceToDevice), "FDK filter memcpy")) {
			ok = false;
			break;
		}

		D_sinoData += (dims.iProjAngles * projPitch);
	}

	if (!checkCuda(hipDeviceSynchronize(), "FDK filter sync")) {
		ok = false;
	}

	hipfftDestroy(planF);
	hipfftDestroy(planI);

	hipFree(D_pfPadded);
	astraCUDA::freeComplexOnDevice(D_pcSinoFFT);
	astraCUDA::freeComplexOnDevice(D_filter);

	return ok;
}


bool FDK(hipPitchedPtr D_volumeData,
         hipPitchedPtr D_projData,
         const SConeProjection* angles,
         const SDimensions3D& dims, SProjectorParams3D params, bool bShortScan,
	     const float* pfFilter)
{
	bool ok;

	// NB: We don't support arbitrary cone_vec geometries here.
	// Only those that are vertical sub-geometries
	// (cf. CompositeGeometryManager) of regular cone geometries.
	assert(dims.iProjAngles > 0);
	const SConeProjection& p0 = angles[0];

	// assuming U is in the XY plane, V is parallel to Z axis
	float fDetCX = p0.fDetSX + 0.5*dims.iProjU*p0.fDetUX;
	float fDetCY = p0.fDetSY + 0.5*dims.iProjU*p0.fDetUY;
	float fDetCZ = p0.fDetSZ + 0.5*dims.iProjV*p0.fDetVZ;

	float fSrcOrigin = sqrt(p0.fSrcX*p0.fSrcX + p0.fSrcY*p0.fSrcY);
	float fDetOrigin = sqrt(fDetCX*fDetCX + fDetCY*fDetCY);
	float fDetUSize = sqrt(p0.fDetUX*p0.fDetUX + p0.fDetUY*p0.fDetUY);
	float fDetVSize = abs(p0.fDetVZ);

	float fZShift = fDetCZ - p0.fSrcZ;

	float *pfAngles = new float[dims.iProjAngles];
	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		// FIXME: Sign/order
		pfAngles[i] = -atan2(angles[i].fSrcX, angles[i].fSrcY) + M_PI;
	}


#if 1
	ok = FDK_PreWeight(D_projData, fSrcOrigin, fDetOrigin,
	                fZShift, fDetUSize, fDetVSize,
	                bShortScan, dims, pfAngles);
#else
	ok = true;
#endif
	delete[] pfAngles;

	if (!ok)
		return false;

#if 1
	// Perform filtering
	ok = FDK_Filter(D_projData, pfFilter, dims);
#endif

	if (!ok)
		return false;

	// Perform BP

	params.bFDKWeighting = true;

	//ok = FDK_BP(D_volumeData, D_projData, fSrcOrigin, fDetOrigin, 0.0f, 0.0f, fDetUSize, fDetVSize, dims, pfAngles);
	ok = ConeBP(D_volumeData, D_projData, dims, angles, params);

	if (!ok)
		return false;

	return true;
}


}
