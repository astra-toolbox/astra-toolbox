#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "cone_fp.h"
#include "testutil.h"
#endif

#include "dims3d.h"
#include "arith3d.h"
#include "cone_bp.h"
#include "../2d/fft.h"

#include "../../include/astra/Logging.h"

namespace astraCUDA3d {

static const unsigned int g_anglesPerWeightBlock = 16;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 12000;

__constant__ float gC_angle[g_MaxAngles];


// per-detector u/v shifts?


__global__ void devFDK_preweight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fZShift, float fDetUSize, float fDetVSize, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the ray(s) to
	// our detector pixel(s).

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	const float fT = fCentralRayLength * fCentralRayLength + fU * fU;

	float fV = (startDetectorV - 0.5f*dims.iProjV + 0.5f) * fDetVSize + fZShift;

	//const float fW = fCentralRayLength;
	//const float fW = fCentralRayLength * (M_PI / 2.0f) / (float)dims.iProjAngles;
	const float fW1 = fSrcOrigin * fDetUSize * fDetVSize;
	const float fW = fCentralRayLength * fW1 * fW1 * (M_PI / 2.0f) / (float)dims.iProjAngles;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		const float fRayLength = sqrtf(fT + fV * fV);

		const float fWeight = fW / fRayLength;

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

		fV += fDetVSize;
	}
}

__global__ void devFDK_ParkerWeight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fDetUSize, float fCentralFanAngle, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the projection
	// of our ray onto the central slice

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	// TODO: Detector pixel size
	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	//const float fGamma = atanf(fU / fCentralRayLength);
	//const float fBeta = gC_angle[angle];
	const float fGamma = atanf(fU / fCentralRayLength);
	float fBeta = -gC_angle[angle];
	if (fBeta < 0.0f)
		fBeta += 2*M_PI;
	if (fBeta >= 2*M_PI)
		fBeta -= 2*M_PI;

	// compute the weight depending on the location in the central fan's radon
	// space
	float fWeight;

	if (fBeta <= 0.0f) {
		fWeight = 0.0f;
	} else if (fBeta <= 2.0f*(fCentralFanAngle + fGamma)) {
		fWeight = sinf((M_PI / 4.0f) * fBeta / (fCentralFanAngle + fGamma));
		fWeight *= fWeight;
	} else if (fBeta <= M_PI + 2*fGamma) {
		fWeight = 1.0f;
	} else if (fBeta <= M_PI + 2*fCentralFanAngle) {
		fWeight = sinf((M_PI / 4.0f) * (M_PI + 2.0f*fCentralFanAngle - fBeta) / (fCentralFanAngle - fGamma));
		fWeight *= fWeight;
	} else {
		fWeight = 0.0f;
	}

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

	}
}



// Perform the FDK pre-weighting and filtering
bool FDK_PreWeight(hipPitchedPtr D_projData,
                float fSrcOrigin, float fDetOrigin,
                float fZShift,
                float fDetUSize, float fDetVSize, bool bShortScan,
                const SDimensions3D& dims, const float* angles)
{
	// The pre-weighting factor for a ray is the cosine of the angle between
	// the central line and the ray.

	dim3 dimBlock(g_detBlockU, g_anglesPerWeightBlock);
	dim3 dimGrid( ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
	              (dims.iProjAngles+g_anglesPerWeightBlock-1)/g_anglesPerWeightBlock);

	int projPitch = D_projData.pitch/sizeof(float);

	devFDK_preweight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fZShift, fDetUSize, fDetVSize, dims);

	cudaTextForceKernelsCompletion();

	if (bShortScan && dims.iProjAngles > 1) {
		ASTRA_DEBUG("Doing Parker weighting");
		// We do short-scan Parker weighting

		// First, determine (in a very basic way) the interval that's
		// been scanned. We assume angles[0] is one of the endpoints of the
		// range.
		float fdA = angles[1] - angles[0];

		while (fdA < -M_PI)
			fdA += 2*M_PI;
		while (fdA >= M_PI)
			fdA -= 2*M_PI;

		float fAngleBase;
		if (fdA >= 0.0f) {
			// going up from angles[0]
			fAngleBase = angles[dims.iProjAngles - 1];
		} else {
			// going down from angles[0]
			fAngleBase = angles[0];
		}

		// We pick the highest end of the range, and then
		// move all angles so they fall in (-2pi,0]

		float *fRelAngles = new float[dims.iProjAngles];
		for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
			float f = angles[i] - fAngleBase;
			while (f > 0)
				f -= 2*M_PI;
			while (f <= -2*M_PI)
				f += 2*M_PI;
			fRelAngles[i] = f;

		}

		hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle), fRelAngles,
		                                    dims.iProjAngles*sizeof(float), 0,
		                                    hipMemcpyHostToDevice);
		assert(!e1);
		delete[] fRelAngles;

		float fCentralFanAngle = atanf(fDetUSize * (dims.iProjU*0.5f) /
		                               (fSrcOrigin + fDetOrigin));

		devFDK_ParkerWeight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fDetUSize, fCentralFanAngle, dims);

	}

	cudaTextForceKernelsCompletion();
	return true;
}

bool FDK_Filter(hipPitchedPtr D_projData,
                hipfftComplex * D_filter,
                const SDimensions3D& dims)
{

	// The filtering is a regular ramp filter per detector line.

	int iPaddedDetCount = calcNextPowerOfTwo(2 * dims.iProjU);
	int iHalfFFTSize = calcFFTFourSize(iPaddedDetCount);
	int projPitch = D_projData.pitch/sizeof(float);
	

	// We process one sinogram at a time.
	float* D_sinoData = (float*)D_projData.ptr;

	hipfftComplex * D_sinoFFT = NULL;
	allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_sinoFFT);

	bool ok = true;

	for (int v = 0; v < dims.iProjV; ++v) {

		ok = runCudaFFT(dims.iProjAngles, D_sinoData, projPitch,
		                dims.iProjU, iPaddedDetCount, iHalfFFTSize,
		                D_sinoFFT);

		if (!ok) break;

		applyFilter(dims.iProjAngles, iHalfFFTSize, D_sinoFFT, D_filter);


		ok = runCudaIFFT(dims.iProjAngles, D_sinoFFT, D_sinoData, projPitch,
		                 dims.iProjU, iPaddedDetCount, iHalfFFTSize);

		if (!ok) break;

		D_sinoData += (dims.iProjAngles * projPitch);
	}

	freeComplexOnDevice(D_sinoFFT);

	return ok;
}


bool FDK(hipPitchedPtr D_volumeData,
         hipPitchedPtr D_projData,
         const SConeProjection* angles,
         const SDimensions3D& dims, SProjectorParams3D params, bool bShortScan)
{
	bool ok;
	// Generate filter
	// TODO: Check errors
	hipfftComplex * D_filter;
	int iPaddedDetCount = calcNextPowerOfTwo(2 * dims.iProjU);
	int iHalfFFTSize = calcFFTFourSize(iPaddedDetCount);


	// NB: We don't support arbitrary cone_vec geometries here.
	// Only those that are vertical sub-geometries
	// (cf. CompositeGeometryManager) of regular cone geometries.
	assert(dims.iProjAngles > 0);
	const SConeProjection& p0 = angles[0];

	// assuming U is in the XY plane, V is parallel to Z axis
	float fDetCX = p0.fDetSX + 0.5*dims.iProjU*p0.fDetUX;
	float fDetCY = p0.fDetSY + 0.5*dims.iProjU*p0.fDetUY;
	float fDetCZ = p0.fDetSZ + 0.5*dims.iProjV*p0.fDetVZ;

	float fSrcOrigin = sqrt(p0.fSrcX*p0.fSrcX + p0.fSrcY*p0.fSrcY);
	float fDetOrigin = sqrt(fDetCX*fDetCX + fDetCY*fDetCY);
	float fDetUSize = sqrt(p0.fDetUX*p0.fDetUX + p0.fDetUY*p0.fDetUY);
	float fDetVSize = abs(p0.fDetVZ);

	float fZShift = fDetCZ - p0.fSrcZ;

	float *pfAngles = new float[dims.iProjAngles];
	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		// FIXME: Sign/order
		pfAngles[i] = -atan2(angles[i].fSrcX, angles[i].fSrcY) + M_PI;
	}


#if 1
	ok = FDK_PreWeight(D_projData, fSrcOrigin, fDetOrigin,
	                fZShift, fDetUSize, fDetVSize,
	                bShortScan, dims, pfAngles);
#else
	ok = true;
#endif
	delete[] pfAngles;

	if (!ok)
		return false;

#if 1
	hipfftComplex *pHostFilter = new hipfftComplex[dims.iProjAngles * iHalfFFTSize];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * dims.iProjAngles * iHalfFFTSize);

	genFilter(FILTER_RAMLAK, 1.0f, dims.iProjAngles, pHostFilter, iPaddedDetCount, iHalfFFTSize);


	allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_filter);
	uploadComplexArrayToDevice(dims.iProjAngles, iHalfFFTSize, pHostFilter, D_filter);

	delete [] pHostFilter;


	// Perform filtering



	ok = FDK_Filter(D_projData, D_filter, dims);

	// Clean up filter
	freeComplexOnDevice(D_filter);
#endif

	if (!ok)
		return false;

	// Perform BP

	params.bFDKWeighting = true;

	//ok = FDK_BP(D_volumeData, D_projData, fSrcOrigin, fDetOrigin, 0.0f, 0.0f, fDetUSize, fDetVSize, dims, pfAngles);
	ok = ConeBP(D_volumeData, D_projData, dims, angles, params);

	if (!ok)
		return false;

	return true;
}


}

#ifdef STANDALONE
void dumpVolume(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* buf = new float[dims.iVolX*dims.iVolY];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iVolZ; ++i) {
		hipMemcpy2D(buf, dims.iVolX*sizeof(float), ((float*)data.ptr)+pitch*dims.iVolY*i, data.pitch, dims.iVolX*sizeof(float), dims.iVolY, hipMemcpyDeviceToHost);

		char fname[512];
		sprintf(fname, filespec, dims.iVolZ-i-1);
		saveImage(fname, dims.iVolY, dims.iVolX, buf, fMin, fMax);
	}
}

void dumpSinograms(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* bufs = new float[dims.iProjAngles*dims.iProjU];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iProjV; ++i) {
		hipMemcpy2D(bufs, dims.iProjU*sizeof(float), ((float*)data.ptr)+pitch*dims.iProjAngles*i, data.pitch, dims.iProjU*sizeof(float), dims.iProjAngles, hipMemcpyDeviceToHost);

		char fname[512];
		sprintf(fname, filespec, i);
		saveImage(fname, dims.iProjAngles, dims.iProjU, bufs, fMin, fMax);
	}
}

void dumpProjections(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* bufp = new float[dims.iProjV*dims.iProjU];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iProjAngles; ++i) {
		for (int j = 0; j < dims.iProjV; ++j) {
			hipMemcpy(bufp+dims.iProjU*j, ((float*)data.ptr)+pitch*dims.iProjAngles*j+pitch*i, dims.iProjU*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[512];
		sprintf(fname, filespec, i);
		saveImage(fname, dims.iProjV, dims.iProjU, bufp, fMin, fMax);
	}
}




int main()
{
#if 0
	SDimensions3D dims;
	dims.iVolX = 512;
	dims.iVolY = 512;
	dims.iVolZ = 512;
	dims.iProjAngles = 180;
	dims.iProjU = 1024;
	dims.iProjV = 1024;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

#if 0
	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
#if 0
		if (i == 128) {
			for (unsigned int j = 0; j < 256*256; ++j)
				slice[j] = 0.0f;
		}
#endif 
	}
#endif

	SConeProjection angle[180];
	angle[0].fSrcX = -1536;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 1024;
	angle[0].fDetSY = -512;
	angle[0].fDetSZ = 512;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = -1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 180; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
		ROTATE0(DetV, i, i*2*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

	//dumpSinograms("sino%03d.png", projData, dims, 0, 512);
	//dumpProjections("proj%03d.png", projData, dims, 0, 512);

	astraCUDA3d::zeroVolumeData(volData, dims);

	float* angles = new float[dims.iProjAngles];
	for (int i = 0; i < 180; ++i)
		angles[i] = i*2*M_PI/180;

	astraCUDA3d::FDK(volData, projData, 1536, 512, 0, 0, dims, angles);

	dumpVolume("vol%03d.png", volData, dims, -20, 100);


#else

	SDimensions3D dims;
	dims.iVolX = 1000;
	dims.iVolY = 999;
	dims.iVolZ = 500;
	dims.iProjAngles = 376;
	dims.iProjU = 1024;
	dims.iProjV = 524;
	dims.iRaysPerDet = 1;

	float* angles = new float[dims.iProjAngles];
	for (int i = 0; i < dims.iProjAngles; ++i)
		angles[i] = -i*(M_PI)/360;

	hipPitchedPtr volData = astraCUDA3d::allocateVolumeData(dims);
	hipPitchedPtr projData = astraCUDA3d::allocateProjectionData(dims);
	astraCUDA3d::zeroProjectionData(projData, dims);
	astraCUDA3d::zeroVolumeData(volData, dims);

	timeval t;
	tic(t);

	for (int i = 0; i < dims.iProjAngles; ++i) {
		char fname[256];
		sprintf(fname, "/home/wpalenst/tmp/Elke/proj%04d.png", i);
		unsigned int w,h;
		float* bufp = loadImage(fname, w,h);

		int pitch = projData.pitch / sizeof(float);
		for (int j = 0; j < dims.iProjV; ++j) {
			hipMemcpy(((float*)projData.ptr)+dims.iProjAngles*pitch*j+pitch*i, bufp+dims.iProjU*j, dims.iProjU*sizeof(float), hipMemcpyHostToDevice);
		}

		delete[] bufp;
	}
	printf("Load time: %f\n", toc(t));

	//dumpSinograms("sino%03d.png", projData, dims, -8.0f, 256.0f);
	//astraCUDA3d::FDK(volData, projData, 7350, 62355, 0, 10, dims, angles);
	//astraCUDA3d::FDK(volData, projData, 7350, -380, 0, 10, dims, angles);

	tic(t);

	astraCUDA3d::FDK(volData, projData, 7383.29867, 0, 0, 10, dims, angles);

	printf("FDK time: %f\n", toc(t));
	tic(t);

	dumpVolume("vol%03d.png", volData, dims, -65.9f, 200.0f);
	//dumpVolume("vol%03d.png", volData, dims, 0.0f, 256.0f);
	printf("Save time: %f\n", toc(t));

#endif


}
#endif
