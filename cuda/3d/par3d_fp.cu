#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_par3DVolumeTexture;

namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
static const unsigned int g_blockSlices = 32;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_RayX[g_MaxAngles];
__constant__ float gC_RayY[g_MaxAngles];
__constant__ float gC_RayZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];


static bool bindVolumeDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_par3DVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_par3DVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_par3DVolumeTexture.addressMode[2] = hipAddressModeBorder;
	gT_par3DVolumeTexture.filterMode = hipFilterModeLinear;
	gT_par3DVolumeTexture.normalized = false;

	hipBindTextureToArray(gT_par3DVolumeTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


// x=0, y=1, z=2
struct DIR_X {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return x; }
	__device__ float c1(float x, float y, float z) const { return y; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f0, f1, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f0; }
	__device__ float y(float f0, float f1, float f2) const { return f1; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// y=0, x=1, z=2
struct DIR_Y {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return y; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f1, f0, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f0; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// z=0, x=1, y=2
struct DIR_Z {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float c0(float x, float y, float z) const { return z; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return y; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f1, f2, f0); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f2; }
	__device__ float z(float f0, float f1, float f2) const { return f0; }
};

struct SCALE_CUBE {
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1+a2*a2+1.0f) * fOutputScale; }
};

struct SCALE_NONCUBE {
	float fScale1;
	float fScale2;
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*fScale1+a2*a2*fScale2+1.0f) * fOutputScale; }
};



// threadIdx: x = u detector
//            y = relative angle
// blockIdx:  x = u/v detector
//            y = angle block


template<class COORD, class SCALE>
__global__ void par3D_FP_t(float* D_projData, unsigned int projPitch,
                           unsigned int startSlice,
                           unsigned int startAngle, unsigned int endAngle,
                           const SDimensions3D dims,
                           SCALE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);


	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;

		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

// Supersampling version
template<class COORD>
__global__ void par3D_FP_SS_t(float* D_projData, unsigned int projPitch,
                              unsigned int startSlice,
                              unsigned int startAngle, unsigned int endAngle,
                              const SDimensions3D dims, int iRaysPerDetDim,
                              SCALE_NONCUBE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	const float fSubStep = 1.0f/iRaysPerDetDim;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{

		float fV = 0.0f;

		float fdU = detectorU - 0.5f + 0.5f*fSubStep;
		for (int iSubU = 0; iSubU < iRaysPerDetDim; ++iSubU, fdU+=fSubStep) {
		float fdV = detectorV - 0.5f + 0.5f*fSubStep;
		for (int iSubV = 0; iSubV < iRaysPerDetDim; ++iSubV, fdV+=fSubStep) {

		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + fdU*fDetUX + fdV*fDetVX;
		const float fDetY = fDetSY + fdU*fDetUY + fdV*fDetVY;
		const float fDetZ = fDetSZ + fdU*fDetUZ + fdV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);


		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fV += fVal;

		}
		}

		fV *= fDistCorr;
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fV / (iRaysPerDetDim * iRaysPerDetDim);
	}
}


__device__ float dirWeights(float fX, float fN) {
	if (fX <= -0.5f) // outside image on left
		return 0.0f;
	if (fX <= 0.5f) // half outside image on left
		return (fX + 0.5f) * (fX + 0.5f);
	if (fX <= fN - 0.5f) { // inside image
		float t = fX + 0.5f - floorf(fX + 0.5f);
		return t*t + (1-t)*(1-t);
	}
	if (fX <= fN + 0.5f) // half outside image on right
		return (fN + 0.5f - fX) * (fN + 0.5f - fX);
	return 0.0f; // outside image on right
}

template<class COORD>
__global__ void par3D_FP_SumSqW_t(float* D_projData, unsigned int projPitch,
                                  unsigned int startSlice,
                                  unsigned int startAngle, unsigned int endAngle,
                                  const SDimensions3D dims,
                                  SCALE_NONCUBE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);


	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += dirWeights(f1, c.nDim1(dims)) * dirWeights(f2, c.nDim2(dims));
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr * fDistCorr;
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

// Supersampling version
// TODO


bool Par3DFP_Array_internal(hipPitchedPtr D_projData,
                   const SDimensions3D& dims, unsigned int angleCount, const SPar3DProjection* angles,
                   const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(RayX);
	TRANSFER_TO_CONSTANT(RayY);
	TRANSFER_TO_CONSTANT(RayZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.fVolScaleX / params.fVolScaleY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.fVolScaleX / params.fVolScaleZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;

	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != dims.iProjAngles) {
			float dX = fabsf(angles[a].fRayX);
			float dY = fabsf(angles[a].fRayY);
			float dZ = fabsf(angles[a].fRayZ);

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
								if (cube)
										par3D_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
								else
										par3D_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeX);
						else
							par3D_FP_SS_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeX);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
								if (cube)
										par3D_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
								else
										par3D_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeY);
						else
							par3D_FP_SS_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeY);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
								if (cube)
										par3D_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, scube);
								else
										par3D_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeZ);
						else
							par3D_FP_SS_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, params.iRaysPerDetDim, snoncubeZ);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();


	// printf("%f\n", toc(t));

	return true;
}

bool Par3DFP(hipPitchedPtr D_volumeData,
             hipPitchedPtr D_projData,
             const SDimensions3D& dims, const SPar3DProjection* angles,
             const SProjectorParams3D& params)
{
	// transfer volume to array
	hipArray* cuArray = allocateVolumeArray(dims);
	transferVolumeToArray(D_volumeData, cuArray, dims);
	bindVolumeDataTexture(cuArray);

	bool ret;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		hipPitchedPtr D_subprojData = D_projData;
		D_subprojData.ptr = (char*)D_projData.ptr + iAngle * D_projData.pitch;

		ret = Par3DFP_Array_internal(D_subprojData,
		                             dims, iEndAngle - iAngle, angles + iAngle,
		                             params);
		if (!ret)
			break;
	}

	hipFreeArray(cuArray);

	return ret;
}



bool Par3DFP_SumSqW(hipPitchedPtr D_volumeData,
                    hipPitchedPtr D_projData,
                    const SDimensions3D& dims, const SPar3DProjection* angles,
                    const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(RayX);
	TRANSFER_TO_CONSTANT(RayY);
	TRANSFER_TO_CONSTANT(RayZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;


	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= dims.iProjAngles; ++a) {
		int dir;
		if (a != dims.iProjAngles) {
			float dX = fabsf(angles[a].fRayX);
			float dY = fabsf(angles[a].fRayY);
			float dZ = fabsf(angles[a].fRayZ);

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == dims.iProjAngles || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeX);
						else
#if 0
							par3D_FP_SS_SumSqW_dirX<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeY);
						else
#if 0
							par3D_FP_SS_SumSqW_dirY<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeZ);
						else
#if 0
							par3D_FP_SS_SumSqW_dirZ<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();


	// printf("%f\n", toc(t));

	return true;
}







}

#ifdef STANDALONE

using namespace astraCUDA3d;

int main()
{
	hipSetDevice(1);


	SDimensions3D dims;
	dims.iVolX = 500;
	dims.iVolY = 500;
	dims.iVolZ = 81;
	dims.iProjAngles = 241;
	dims.iProjU = 600;
	dims.iProjV = 100;
	dims.iRaysPerDet = 1;

	SPar3DProjection base;
	base.fRayX = 1.0f;
	base.fRayY = 0.0f;
	base.fRayZ = 0.1f;

	base.fDetSX = 0.0f;
	base.fDetSY = -300.0f;
	base.fDetSZ = -50.0f;

	base.fDetUX = 0.0f;
	base.fDetUY = 1.0f;
	base.fDetUZ = 0.0f;

	base.fDetVX = 0.0f;
	base.fDetVY = 0.0f;
	base.fDetVZ = 1.0f;

	SPar3DProjection angle[dims.iProjAngles];

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	volData = allocateVolumeData(dims);

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	projData = allocateProjectionData(dims);

	unsigned int ix = 500,iy = 500;

	float* buf = new float[dims.iProjU*dims.iProjV];

	float* slice = new float[dims.iVolX*dims.iVolY];
	for (int i = 0; i < dims.iVolX*dims.iVolY; ++i)
		slice[i] = 1.0f;

	for (unsigned int a = 0; a < 241; a += dims.iProjAngles) {

		zeroProjectionData(projData, dims);

		for (int y = 0; y < iy; y += dims.iVolY) {
			for (int x = 0; x < ix; x += dims.iVolX) { 

				timeval st;
				tic(st);

				for (int z = 0; z < dims.iVolZ; ++z) {
//					char sfn[256];
//					sprintf(sfn, "/home/wpalenst/projects/cone_simulation/phantom_4096/mouse_fem_phantom_%04d.png", 30+z);
//					float* slice = loadSubImage(sfn, x, y, dims.iVolX, dims.iVolY);

					hipPitchedPtr ptr;
					ptr.ptr = slice;
					ptr.pitch = dims.iVolX*sizeof(float);
					ptr.xsize = dims.iVolX*sizeof(float);
					ptr.ysize = dims.iVolY;
					hipExtent extentS;
					extentS.width = dims.iVolX*sizeof(float);
					extentS.height = dims.iVolY;
					extentS.depth = 1;

					hipPos sp = { 0, 0, 0 };
					hipPos dp = { 0, 0, z };
					hipMemcpy3DParms p;
					p.srcArray = 0;
					p.srcPos = sp;
					p.srcPtr = ptr;
					p.dstArray = 0;
					p.dstPos = dp;
					p.dstPtr = volData;
					p.extent = extentS;
					p.kind = hipMemcpyHostToDevice;
					hipError_t err = hipMemcpy3D(&p);
					assert(!err);
//					delete[] slice;
				}

				printf("Load: %f\n", toc(st));

#if 0

	hipPos zp = { 0, 0, 0 };

	hipPitchedPtr t;
	t.ptr = new float[1024*1024];
	t.pitch = 1024*4;
	t.xsize = 1024*4;
	t.ysize = 1024;

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = volData;
	p.extent = extentS;
	p.dstArray = 0;
	p.dstPtr = t;
	p.dstPos = zp;
	p.kind = hipMemcpyDeviceToHost;
	hipError_t err = hipMemcpy3D(&p);
	assert(!err);

	char fn[32];
	sprintf(fn, "t%d%d.png", x / dims.iVolX, y / dims.iVolY);
	saveImage(fn, 1024, 1024, (float*)t.ptr);
	saveImage("s.png", 4096, 4096, slice);
	delete[] (float*)t.ptr;
#endif


#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = base.f##name##X * cos(alpha) - base.f##name##Y * sin(alpha); angle[i].f##name##Y = base.f##name##X * sin(alpha) + base.f##name##Y * cos(alpha); angle[i].f##name##Z = base.f##name##Z; } while(0)
#define SHIFT(name,i,x,y) do { angle[i].f##name##X += x; angle[i].f##name##Y += y; } while(0)
				for (int i = 0; i < dims.iProjAngles; ++i) {
					ROTATE0(Ray, i, (a+i)*.8*M_PI/180);
					ROTATE0(DetS, i, (a+i)*.8*M_PI/180);
					ROTATE0(DetU, i, (a+i)*.8*M_PI/180);
					ROTATE0(DetV, i, (a+i)*.8*M_PI/180);


//					SHIFT(Src, i, (-x+1536), (-y+1536));
//					SHIFT(DetS, i, (-x+1536), (-y+1536));
				}
#undef ROTATE0
#undef SHIFT
				tic(st);

				astraCUDA3d::Par3DFP(volData, projData, dims, angle, 1.0f);

				printf("FP: %f\n", toc(st));

			}
		}
		for (unsigned int aa = 0; aa < dims.iProjAngles; ++aa) {
			for (unsigned int v = 0; v < dims.iProjV; ++v)
				hipMemcpy(buf+v*dims.iProjU, ((float*)projData.ptr)+(v*dims.iProjAngles+aa)*(projData.pitch/sizeof(float)), dims.iProjU*sizeof(float), hipMemcpyDeviceToHost);

			char fname[32];
			sprintf(fname, "proj%03d.png", a+aa);
			saveImage(fname, dims.iProjV, dims.iProjU, buf, 0.0f, 1000.0f);
		}
	}

	delete[] buf;

}
#endif
