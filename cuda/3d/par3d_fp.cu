#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2021, imec Vision Lab, University of Antwerp
           2014-2021, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_par3DVolumeTexture;

namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
// TODO: Make this configurable for X/Y/Z independently
static const unsigned int g_blockSlices = 32;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_RayX[g_MaxAngles];
__constant__ float gC_RayY[g_MaxAngles];
__constant__ float gC_RayZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];


// startAngle, endAngle, startDetectorV
__constant__ int gC_params[3];


static bool bindVolumeDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_par3DVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_par3DVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_par3DVolumeTexture.addressMode[2] = hipAddressModeBorder;
	gT_par3DVolumeTexture.filterMode = hipFilterModeLinear;
	gT_par3DVolumeTexture.normalized = false;

	hipBindTextureToArray(gT_par3DVolumeTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


// x=0, y=1, z=2
struct DIR_X {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return x; }
	__device__ float c1(float x, float y, float z) const { return y; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f0, f1, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f0; }
	__device__ float y(float f0, float f1, float f2) const { return f1; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// y=0, x=1, z=2
struct DIR_Y {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float c0(float x, float y, float z) const { return y; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return z; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f1, f0, f2); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f0; }
	__device__ float z(float f0, float f1, float f2) const { return f2; }
};

// z=0, x=1, y=2
struct DIR_Z {
	__device__ float nSlices(const SDimensions3D& dims) const { return dims.iVolZ; }
	__device__ float nDim1(const SDimensions3D& dims) const { return dims.iVolX; }
	__device__ float nDim2(const SDimensions3D& dims) const { return dims.iVolY; }
	__device__ float c0(float x, float y, float z) const { return z; }
	__device__ float c1(float x, float y, float z) const { return x; }
	__device__ float c2(float x, float y, float z) const { return y; }
	__device__ float tex(float f0, float f1, float f2) const { return tex3D(gT_par3DVolumeTexture, f1, f2, f0); }
	__device__ float x(float f0, float f1, float f2) const { return f1; }
	__device__ float y(float f0, float f1, float f2) const { return f2; }
	__device__ float z(float f0, float f1, float f2) const { return f0; }
};

struct SCALE_CUBE {
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1+a2*a2+1.0f) * fOutputScale; }
};

struct SCALE_NONCUBE {
	float fScale1;
	float fScale2;
	float fOutputScale;
	__device__ float scale(float a1, float a2) const { return sqrt(a1*a1*fScale1+a2*a2*fScale2+1.0f) * fOutputScale; }
};



// threadIdx: x = u detector
//            y = relative angle
// blockIdx:  x = u/v detector
//            y = angle block


template<class COORD, class SCALE>
__global__ void par3D_FP_t(float* D_projData, unsigned int projPitch,
                           unsigned int startSlice,
                           const SDimensions3D dims,
                           SCALE sc)
{
	COORD c;

	int angle = gC_params[0] + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= gC_params[1])
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);


	const int detectorU = blockIdx.x * g_detBlockU + threadIdx.x;
	const int startDetectorV = gC_params[2];
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr;

		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

// Supersampling version
template<class COORD>
__global__ void par3D_FP_SS_t(float* D_projData, unsigned int projPitch,
                              unsigned int startSlice,
                              const SDimensions3D dims, int iRaysPerDetDim,
                              SCALE_NONCUBE sc)
{
	COORD c;

	int angle = gC_params[0] + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= gC_params[1])
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);

	const int detectorU = blockIdx.x * g_detBlockU + threadIdx.x;
	const int startDetectorV = gC_params[2];
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	const float fSubStep = 1.0f/iRaysPerDetDim;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{

		float fV = 0.0f;

		float fdU = detectorU - 0.5f + 0.5f*fSubStep;
		for (int iSubU = 0; iSubU < iRaysPerDetDim; ++iSubU, fdU+=fSubStep) {
		float fdV = detectorV - 0.5f + 0.5f*fSubStep;
		for (int iSubV = 0; iSubV < iRaysPerDetDim; ++iSubV, fdV+=fSubStep) {

		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + fdU*fDetUX + fdV*fDetVX;
		const float fDetY = fDetSY + fdU*fDetUY + fdV*fDetVY;
		const float fDetZ = fDetSZ + fdU*fDetUZ + fdV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);


		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += c.tex(f0, f1, f2);
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fV += fVal;

		}
		}

		fV *= fDistCorr;
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fV / (iRaysPerDetDim * iRaysPerDetDim);
	}
}


__device__ float dirWeights(float fX, float fN) {
	if (fX <= -0.5f) // outside image on left
		return 0.0f;
	if (fX <= 0.5f) // half outside image on left
		return (fX + 0.5f) * (fX + 0.5f);
	if (fX <= fN - 0.5f) { // inside image
		float t = fX + 0.5f - floorf(fX + 0.5f);
		return t*t + (1-t)*(1-t);
	}
	if (fX <= fN + 0.5f) // half outside image on right
		return (fN + 0.5f - fX) * (fN + 0.5f - fX);
	return 0.0f; // outside image on right
}

template<class COORD>
__global__ void par3D_FP_SumSqW_t(float* D_projData, unsigned int projPitch,
                                  unsigned int startSlice,
                                  unsigned int startAngle, unsigned int endAngle,
                                  const SDimensions3D dims,
                                  SCALE_NONCUBE sc)
{
	COORD c;

	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const float fRayX = gC_RayX[angle];
	const float fRayY = gC_RayY[angle];
	const float fRayZ = gC_RayZ[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];
	const float fDetUZ = gC_DetUZ[angle];
	const float fDetVX = gC_DetVX[angle];
	const float fDetVY = gC_DetVY[angle];
	const float fDetVZ = gC_DetVZ[angle];
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;

	const float a1 = c.c1(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float a2 = c.c2(fRayX,fRayY,fRayZ) / c.c0(fRayX,fRayY,fRayZ);
	const float fDistCorr = sc.scale(a1, a2);


	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	int endSlice = startSlice + g_blockSlices;
	if (endSlice > c.nSlices(dims))
		endSlice = c.nSlices(dims);

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		/* Trace ray in direction Ray to (detectorU,detectorV) from  */
		/* X = startSlice to X = endSlice                            */

		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;

		/*        (x)   ( 1)       ( 0)    */
		/* ray:   (y) = (ay) * x + (by)    */
		/*        (z)   (az)       (bz)    */

		const float b1 = c.c1(fDetX,fDetY,fDetZ) - a1 * c.c0(fDetX,fDetY,fDetZ);
		const float b2 = c.c2(fDetX,fDetY,fDetZ) - a2 * c.c0(fDetX,fDetY,fDetZ);

		float fVal = 0.0f;

		float f0 = startSlice + 0.5f;
		float f1 = a1 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b1 + 0.5f*c.nDim1(dims) - 0.5f + 0.5f;
		float f2 = a2 * (startSlice - 0.5f*c.nSlices(dims) + 0.5f) + b2 + 0.5f*c.nDim2(dims) - 0.5f + 0.5f;

		for (int s = startSlice; s < endSlice; ++s)
		{
			fVal += dirWeights(f1, c.nDim1(dims)) * dirWeights(f2, c.nDim2(dims));
			f0 += 1.0f;
			f1 += a1;
			f2 += a2;
		}

		fVal *= fDistCorr * fDistCorr;
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;
	}
}

// Supersampling version
// TODO


bool Par3DFP_Array_internal(hipPitchedPtr D_projData,
                   const SDimensions3D& dims, unsigned int angleCount, const SPar3DProjection* angles,
                   const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(RayX);
	TRANSFER_TO_CONSTANT(RayY);
	TRANSFER_TO_CONSTANT(RayZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	bool cube = true;
	if (abs(params.fVolScaleX / params.fVolScaleY - 1.0) > 0.00001)
		cube = false;
	if (abs(params.fVolScaleX / params.fVolScaleZ - 1.0) > 0.00001)
		cube = false;

	SCALE_CUBE scube;
	scube.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;

	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= angleCount; ++a) {
		int dir = -1;
		if (a != dims.iProjAngles) {
			float dX = fabsf(angles[a].fRayX);
			float dY = fabsf(angles[a].fRayY);
			float dZ = fabsf(angles[a].fRayZ);

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == angleCount || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {


				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				for (unsigned int startDetectorV = 0; startDetectorV < dims.iProjV; startDetectorV += g_detBlockV) {
					int p[3] = { (int)blockStart, (int)blockEnd, (int)startDetectorV };
					hipMemcpyToSymbol(HIP_SYMBOL(gC_params), p, 3*sizeof(int), 0, hipMemcpyHostToDevice);
					if (blockDirection == 0) {
						for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
							if (params.iRaysPerDetDim == 1)
								if (cube)
									par3D_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, scube);
								else
									par3D_FP_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, snoncubeX);
							else
								par3D_FP_SS_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, params.iRaysPerDetDim, snoncubeX);
					} else if (blockDirection == 1) {
						for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
							if (params.iRaysPerDetDim == 1)
								if (cube)
									par3D_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, scube);
								else
									par3D_FP_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, snoncubeY);
							else
								par3D_FP_SS_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, params.iRaysPerDetDim, snoncubeY);
					} else if (blockDirection == 2) {
						for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
							if (params.iRaysPerDetDim == 1)
								if (cube)
									par3D_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, scube);
								else
									par3D_FP_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, snoncubeZ);
							else
								par3D_FP_SS_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, dims, params.iRaysPerDetDim, snoncubeZ);
					}
				}
			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();


	// printf("%f\n", toc(t));

	return true;
}

bool Par3DFP(hipPitchedPtr D_volumeData,
             hipPitchedPtr D_projData,
             const SDimensions3D& dims, const SPar3DProjection* angles,
             const SProjectorParams3D& params)
{
	// transfer volume to array
	hipArray* cuArray = allocateVolumeArray(dims);
	transferVolumeToArray(D_volumeData, cuArray, dims);
	bindVolumeDataTexture(cuArray);

	bool ret;

	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;

		hipPitchedPtr D_subprojData = D_projData;
		D_subprojData.ptr = (char*)D_projData.ptr + iAngle * D_projData.pitch;

		ret = Par3DFP_Array_internal(D_subprojData,
		                             dims, iEndAngle - iAngle, angles + iAngle,
		                             params);
		if (!ret)
			break;
	}

	hipFreeArray(cuArray);

	return ret;
}



bool Par3DFP_SumSqW(hipPitchedPtr D_volumeData,
                    hipPitchedPtr D_projData,
                    const SDimensions3D& dims, const SPar3DProjection* angles,
                    const SProjectorParams3D& params)
{
	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(RayX);
	TRANSFER_TO_CONSTANT(RayY);
	TRANSFER_TO_CONSTANT(RayZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	SCALE_NONCUBE snoncubeX;
	float fS1 = params.fVolScaleY / params.fVolScaleX;
	snoncubeX.fScale1 = fS1 * fS1;
	float fS2 = params.fVolScaleZ / params.fVolScaleX;
	snoncubeX.fScale2 = fS2 * fS2;
	snoncubeX.fOutputScale = params.fOutputScale * params.fVolScaleX;

	SCALE_NONCUBE snoncubeY;
	fS1 = params.fVolScaleX / params.fVolScaleY;
	snoncubeY.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleY;
	snoncubeY.fScale2 = fS2 * fS2;
	snoncubeY.fOutputScale = params.fOutputScale * params.fVolScaleY;

	SCALE_NONCUBE snoncubeZ;
	fS1 = params.fVolScaleX / params.fVolScaleZ;
	snoncubeZ.fScale1 = fS1 * fS1;
	fS2 = params.fVolScaleY / params.fVolScaleZ;
	snoncubeZ.fScale2 = fS2 * fS2;
	snoncubeZ.fOutputScale = params.fOutputScale * params.fVolScaleZ;


	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= dims.iProjAngles; ++a) {
		int dir;
		if (a != dims.iProjAngles) {
			float dX = fabsf(angles[a].fRayX);
			float dY = fabsf(angles[a].fRayY);
			float dZ = fabsf(angles[a].fRayZ);

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == dims.iProjAngles || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_X><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeX);
						else
#if 0
							par3D_FP_SS_SumSqW_dirX<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_Y><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeY);
						else
#if 0
							par3D_FP_SS_SumSqW_dirY<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (params.iRaysPerDetDim == 1)
							par3D_FP_SumSqW_t<DIR_Z><<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, snoncubeZ);
						else
#if 0
							par3D_FP_SS_SumSqW_dirZ<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
#else
							assert(false);
#endif
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();


	// printf("%f\n", toc(t));

	return true;
}







}
