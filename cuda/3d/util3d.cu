#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"

#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA3d {


hipPitchedPtr allocateVolumeData(const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData;

	hipError_t err = hipMalloc3D(&volData, extentV);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU buffer", dims.iVolX, dims.iVolY, dims.iVolZ);
		volData.ptr = 0;
		// TODO: return 0 somehow?
	}

	return volData;
}
hipPitchedPtr allocateProjectionData(const SDimensions3D& dims)
{
	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData;

	hipError_t err = hipMalloc3D(&projData, extentP);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU buffer", dims.iProjU, dims.iProjAngles, dims.iProjV);
		projData.ptr = 0;
		// TODO: return 0 somehow?
	}

	return projData;
}
bool zeroVolumeData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;
	hipError_t err;

	for (unsigned int z = 0; z < dims.iVolZ; ++z) {
		err = hipMemset2D(t, D_data.pitch, 0, dims.iVolX*sizeof(float), dims.iVolY);
		ASTRA_CUDA_ASSERT(err);
		t += D_data.pitch * dims.iVolY;
	}
	return true;
}
bool zeroProjectionData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;
	hipError_t err;

	for (unsigned int z = 0; z < dims.iProjV; ++z) {
		err = hipMemset2D(t, D_data.pitch, 0, dims.iProjU*sizeof(float), dims.iProjAngles);
		ASTRA_CUDA_ASSERT(err);
		t += D_data.pitch * dims.iProjAngles;
	}

	return true;
}
bool copyVolumeToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool copyProjectionsToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool copyVolumeFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}
bool copyProjectionsFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool duplicateVolumeData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}
bool duplicateProjectionData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}



// TODO: Consider using a single array of size max(proj,volume) (per dim)
//       instead of allocating a new one each time

hipArray* allocateVolumeArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iVolX;
	extentA.height = dims.iVolY;
	extentA.depth = dims.iVolZ;
	hipError_t err = hipMalloc3DArray(&cuArray, &channelDesc, extentA);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU array", dims.iVolX, dims.iVolY, dims.iVolZ);
		return 0;
	}

	return cuArray;
}
hipArray* allocateProjectionArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;
	hipError_t err = hipMalloc3DArray(&cuArray, &channelDesc, extentA);

	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU array", dims.iProjU, dims.iProjAngles, dims.iProjV);
		return 0;
	}

	return cuArray;
}

bool transferVolumeToArray(hipPitchedPtr D_volumeData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iVolX;
	extentA.height = dims.iVolY;
	extentA.depth = dims.iVolZ;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_volumeData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);
	// TODO: check errors

	return true;
}
bool transferProjectionsToArray(hipPitchedPtr D_projData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_projData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	// TODO: check errors

	return true;
}
bool transferHostProjectionsToArray(const float *projData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)projData; // const cast away
	ptr.pitch = dims.iProjU*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyHostToDevice;

	hipError_t err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	// TODO: check errors

	return true;
}



float dotProduct3D(hipPitchedPtr data, unsigned int x, unsigned int y,
                   unsigned int z)
{
	return astraCUDA::dotProduct2D((float*)data.ptr, data.pitch/sizeof(float), x, y*z);
}


bool cudaTextForceKernelsCompletion()
{
	hipError_t returnedCudaError = hipDeviceSynchronize();

	if(returnedCudaError != hipSuccess) {
		ASTRA_ERROR("Failed to force completion of cuda kernels: %d: %s.", returnedCudaError, hipGetErrorString(returnedCudaError));
		return false;
	}

	return true;
}

int calcNextPowerOfTwo(int _iValue)
{
	int iOutput = 1;
	while(iOutput < _iValue)
		iOutput *= 2;
	return iOutput;
}

}
