#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "cgls3d.h"
#include "util3d.h"
#include "arith3d.h"
#include "cone_fp.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

namespace astraCUDA3d {

CGLS::CGLS() : ReconAlgo3D()
{
	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_r.ptr = 0;
	D_w.ptr = 0;
	D_z.ptr = 0;
	D_p.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;
}


CGLS::~CGLS()
{
	reset();
}

void CGLS::reset()
{
	hipFree(D_r.ptr);
	hipFree(D_w.ptr);
	hipFree(D_z.ptr);
	hipFree(D_p.ptr);

	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_r.ptr = 0;
	D_w.ptr = 0;
	D_z.ptr = 0;
	D_p.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	sliceInitialized = false;

	ReconAlgo3D::reset();
}

bool CGLS::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool CGLS::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool CGLS::init()
{
	D_z = allocateVolumeData(dims);
	D_p = allocateVolumeData(dims);
	D_r = allocateProjectionData(dims);
	D_w = allocateProjectionData(dims);

	// TODO: check if allocations succeeded
	return true;
}

bool CGLS::setVolumeMask(hipPitchedPtr& _D_maskData)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;

	return true;
}

bool CGLS::setSinogramMask(hipPitchedPtr& _D_smaskData)
{
	return false;
#if 0
	// TODO: Implement this
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;
	return true;
#endif
}

bool CGLS::setBuffers(hipPitchedPtr& _D_volumeData,
                      hipPitchedPtr& _D_projData)
{
	D_volumeData = _D_volumeData;
	D_sinoData = _D_projData;

	sliceInitialized = false;

	return true;
}

bool CGLS::iterate(unsigned int iterations)
{
	shouldAbort = false;

	if (!sliceInitialized) {

		// copy sinogram
		duplicateProjectionData(D_r, D_sinoData, dims);

		// r = sino - A*x
		if (useVolumeMask) {
				duplicateVolumeData(D_z, D_volumeData, dims);
				processVol3D<opMul>(D_z, D_maskData, dims);
				callFP(D_z, D_r, -1.0f);
		} else {
				callFP(D_volumeData, D_r, -1.0f);
		}

		// p = A'*r
		zeroVolumeData(D_p, dims);
		callBP(D_p, D_r, 1.0f);
		if (useVolumeMask)
			processVol3D<opMul>(D_p, D_maskData, dims);

		gamma = dotProduct3D(D_p, dims.iVolX, dims.iVolY, dims.iVolZ);

		sliceInitialized = true;

	}


	// iteration
	for (unsigned int iter = 0; iter < iterations && !shouldAbort; ++iter) {

		// w = A*p
		zeroProjectionData(D_w, dims);
		callFP(D_p, D_w, 1.0f);

		// alpha = gamma / <w,w>
		float ww = dotProduct3D(D_w, dims.iProjU, dims.iProjAngles, dims.iProjV);
		float alpha = gamma / ww;

		// x += alpha*p
		processVol3D<opAddScaled>(D_volumeData, D_p, alpha, dims);

		// r -= alpha*w
		processSino3D<opAddScaled>(D_r, D_w, -alpha, dims);

		// z = A'*r
		zeroVolumeData(D_z, dims);
		callBP(D_z, D_r, 1.0f);
		if (useVolumeMask)
			processVol3D<opMul>(D_z, D_maskData, dims);

		float beta = 1.0f / gamma;
		gamma = dotProduct3D(D_z, dims.iVolX, dims.iVolY, dims.iVolZ);

		beta *= gamma;

		// p = z + beta*p
		processVol3D<opScaleAndAdd>(D_p, D_z, beta, dims);
	}

	return true;
}

float CGLS::computeDiffNorm()
{
	// We can use w and z as temporary storage here since they're not
	// used outside of iterations.

	// copy sinogram to w
	duplicateProjectionData(D_w, D_sinoData, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_z, D_volumeData, dims);
			processVol3D<opMul>(D_z, D_maskData, dims);
			callFP(D_z, D_w, -1.0f);
	} else {
			callFP(D_volumeData, D_w, -1.0f);
	}

	float s = dotProduct3D(D_w, dims.iProjU, dims.iProjAngles, dims.iProjV);
	return sqrt(s);
}


bool doCGLS(hipPitchedPtr& D_volumeData, 
            hipPitchedPtr& D_sinoData,
            hipPitchedPtr& D_maskData,
            const SDimensions3D& dims, const SConeProjection* angles,
            unsigned int iterations)
{
	CGLS cgls;
	bool ok = true;

	ok &= cgls.setConeGeometry(dims, angles, SProjectorParams3D());
	if (D_maskData.ptr)
		ok &= cgls.enableVolumeMask();

	if (!ok)
		return false;

	ok = cgls.init();
	if (!ok)
		return false;

	if (D_maskData.ptr)
		ok &= cgls.setVolumeMask(D_maskData);

	ok &= cgls.setBuffers(D_volumeData, D_sinoData);
	if (!ok)
		return false;

	ok = cgls.iterate(iterations);

	return ok;
}

}

#ifdef STANDALONE

using namespace astraCUDA3d;

int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 100;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	SConeProjection angle[100];
	angle[0].fSrcX = -2905.6;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 694.4;
	angle[0].fDetSY = -122.4704;
	angle[0].fDetSZ = -122.4704;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = .4784;
	//angle[0].fDetUY = .5;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = .4784;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 100; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/100);
		ROTATE0(DetS, i, i*2*M_PI/100);
		ROTATE0(DetU, i, i*2*M_PI/100);
		ROTATE0(DetV, i, i*2*M_PI/100);
	}
#undef ROTATE0


	hipPitchedPtr volData = allocateVolumeData(dims);
	hipPitchedPtr projData = allocateProjectionData(dims);
	zeroProjectionData(projData, dims);

	float* pbuf = new float[100*512*512];
	copyProjectionsFromDevice(pbuf, projData, dims);
	copyProjectionsToDevice(pbuf, projData, dims);
	delete[] pbuf;

#if 0
	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256; ++i) {
		for (unsigned int y = 0; y < 256; ++y)
			for (unsigned int x = 0; x < 256; ++x)
				slice[y*256+x] = (i-127.5)*(i-127.5)+(y-127.5)*(y-127.5)+(x-127.5)*(x-127.5) < 4900 ? 1.0f : 0.0f;

		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
	}
	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

#else

	for (int i = 0; i < 100; ++i) {
		char fname[32];
		sprintf(fname, "Tiffs/%04d.png", 4*i);
		unsigned int w,h;
		float* bufp = loadImage(fname, w,h);

		for (int j = 0; j < 512*512; ++j) {
			float v = bufp[j];
			if (v > 236.0f) v = 236.0f;
			v = logf(236.0f / v);
			bufp[j] = 256*v;
		}

		for (int j = 0; j < 512; ++j) {
			hipMemcpy(((float*)projData.ptr)+100*512*j+512*i, bufp+512*j, 512*sizeof(float), hipMemcpyHostToDevice);
		}

		delete[] bufp;

	}
#endif

#if 0
	float* bufs = new float[100*512];

	for (int i = 0; i < 512; ++i) {
		hipMemcpy(bufs, ((float*)projData.ptr)+100*512*i, 100*512*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

		char fname[20];
		sprintf(fname, "sino%03d.png", i);
		saveImage(fname, 100, 512, bufs);
	}

	float* bufp = new float[512*512];

	for (int i = 0; i < 100; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)projData.ptr)+100*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "proj%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif

	zeroVolumeData(volData, dims);

	hipPitchedPtr maskData;
	maskData.ptr = 0;

	astraCUDA3d::doCGLS(volData, projData, maskData, dims, angle, 50);
#if 1
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)volData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "vol%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

	return 0;
}
#endif

