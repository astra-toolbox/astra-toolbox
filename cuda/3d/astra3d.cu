#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/cgls3d.h"
#include "astra/cuda/3d/sirt3d.h"
#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/cone_fp.h"
#include "astra/cuda/3d/cone_bp.h"
#include "astra/cuda/3d/par3d_fp.h"
#include "astra/cuda/3d/par3d_bp.h"
#include "astra/cuda/3d/fdk.h"
#include "astra/cuda/3d/arith3d.h"
#include "astra/cuda/3d/astra3d.h"
#include "astra/cuda/3d/mem3d.h"

#include "astra/ParallelProjectionGeometry3D.h"
#include "astra/ParallelVecProjectionGeometry3D.h"
#include "astra/ConeProjectionGeometry3D.h"
#include "astra/ConeVecProjectionGeometry3D.h"
#include "astra/VolumeGeometry3D.h"
#include "astra/Data3D.h"
#include "astra/Logging.h"

#include <iostream>
#include <cstdio>
#include <cassert>

using namespace astraCUDA3d;

namespace astra {

enum CUDAProjectionType3d {
	PROJ_PARALLEL,
	PROJ_CONE
};






// adjust pProjs to normalize volume geometry
template<typename ProjectionT>
static bool convertAstraGeometry_internal(const CVolumeGeometry3D* pVolGeom,
                          unsigned int iProjectionAngleCount,
                          ProjectionT*& pProjs,
                          SProjectorParams3D& params)
{
	assert(pVolGeom);
	assert(pProjs);

#if 0
	// TODO: Relative instead of absolute
	const float EPS = 0.00001f;
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthZ()) > EPS)
		return false;
#endif

	// Translate
	float dx = -(pVolGeom->getWindowMinX() + pVolGeom->getWindowMaxX()) / 2;
	float dy = -(pVolGeom->getWindowMinY() + pVolGeom->getWindowMaxY()) / 2;
	float dz = -(pVolGeom->getWindowMinZ() + pVolGeom->getWindowMaxZ()) / 2;

	float fx = 1.0f / pVolGeom->getPixelLengthX();
	float fy = 1.0f / pVolGeom->getPixelLengthY();
	float fz = 1.0f / pVolGeom->getPixelLengthZ();

	for (int i = 0; i < iProjectionAngleCount; ++i) {
		// CHECKME: Order of scaling and translation
		pProjs[i].translate(dx, dy, dz);
		pProjs[i].scale(fx, fy, fz);
	}

	params.fVolScaleX = pVolGeom->getPixelLengthX();
	params.fVolScaleY = pVolGeom->getPixelLengthY();
	params.fVolScaleZ = pVolGeom->getPixelLengthZ();

	// CHECKME: Check factor
	//params.fOutputScale *= pVolGeom->getPixelLengthX();

	return true;
}


bool convertAstraGeometry_dims(const CVolumeGeometry3D* pVolGeom,
                               const CProjectionGeometry3D* pProjGeom,
                               SDimensions3D& dims)
{
	dims.iVolX = pVolGeom->getGridColCount();
	dims.iVolY = pVolGeom->getGridRowCount();
	dims.iVolZ = pVolGeom->getGridSliceCount();
	dims.iProjAngles = pProjGeom->getProjectionCount();
	dims.iProjU = pProjGeom->getDetectorColCount();
	dims.iProjV = pProjGeom->getDetectorRowCount();

	if (dims.iVolX <= 0 || dims.iVolX <= 0 || dims.iVolX <= 0)
		return false;
	if (dims.iProjAngles <= 0 || dims.iProjU <= 0 || dims.iProjV <= 0)
		return false;

	return true;
}


bool convertAstraGeometry(const CVolumeGeometry3D* pVolGeom,
                          const CParallelProjectionGeometry3D* pProjGeom,
                          SPar3DProjection*& pProjs, SProjectorParams3D& params)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	int nth = pProjGeom->getProjectionCount();

	pProjs = genPar3DProjections(nth,
	                             pProjGeom->getDetectorColCount(),
	                             pProjGeom->getDetectorRowCount(),
	                             pProjGeom->getDetectorSpacingX(),
	                             pProjGeom->getDetectorSpacingY(),
	                             pProjGeom->getProjectionAngles());

	bool ok;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, params);

	return ok;
}

bool convertAstraGeometry(const CVolumeGeometry3D* pVolGeom,
                          const CParallelVecProjectionGeometry3D* pProjGeom,
                          SPar3DProjection*& pProjs, SProjectorParams3D& params)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionVectors());

	int nth = pProjGeom->getProjectionCount();

	pProjs = new SPar3DProjection[nth];
	for (int i = 0; i < nth; ++i)
		pProjs[i] = pProjGeom->getProjectionVectors()[i];

	bool ok;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, params);

	return ok;
}

bool convertAstraGeometry(const CVolumeGeometry3D* pVolGeom,
                          const CConeProjectionGeometry3D* pProjGeom,
                          SConeProjection*& pProjs, SProjectorParams3D& params)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	int nth = pProjGeom->getProjectionCount();

	pProjs = genConeProjections(nth,
	                            pProjGeom->getDetectorColCount(),
	                            pProjGeom->getDetectorRowCount(),
	                            pProjGeom->getOriginSourceDistance(),
	                            pProjGeom->getOriginDetectorDistance(),
	                            pProjGeom->getDetectorSpacingX(),
	                            pProjGeom->getDetectorSpacingY(),
	                            pProjGeom->getProjectionAngles());

	bool ok;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, params);

	return ok;
}

bool convertAstraGeometry(const CVolumeGeometry3D* pVolGeom,
                          const CConeVecProjectionGeometry3D* pProjGeom,
                          SConeProjection*& pProjs, SProjectorParams3D& params)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionVectors());

	int nth = pProjGeom->getProjectionCount();

	pProjs = new SConeProjection[nth];
	for (int i = 0; i < nth; ++i)
		pProjs[i] = pProjGeom->getProjectionVectors()[i];

	bool ok;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, params);

	return ok;
}


bool convertAstraGeometry(const CVolumeGeometry3D* pVolGeom,
                          const CProjectionGeometry3D* pProjGeom,
                          SPar3DProjection*& pParProjs,
                          SConeProjection*& pConeProjs,
                          SProjectorParams3D& params)
{
	const CConeProjectionGeometry3D* conegeom = dynamic_cast<const CConeProjectionGeometry3D*>(pProjGeom);
	const CParallelProjectionGeometry3D* par3dgeom = dynamic_cast<const CParallelProjectionGeometry3D*>(pProjGeom);
	const CParallelVecProjectionGeometry3D* parvec3dgeom = dynamic_cast<const CParallelVecProjectionGeometry3D*>(pProjGeom);
	const CConeVecProjectionGeometry3D* conevec3dgeom = dynamic_cast<const CConeVecProjectionGeometry3D*>(pProjGeom);

	pConeProjs = 0;
	pParProjs = 0;

	bool ok;

	if (conegeom) {
		ok = convertAstraGeometry(pVolGeom, conegeom, pConeProjs, params);
	} else if (conevec3dgeom) {
		ok = convertAstraGeometry(pVolGeom, conevec3dgeom, pConeProjs, params);
	} else if (par3dgeom) {
		ok = convertAstraGeometry(pVolGeom, par3dgeom, pParProjs, params);
	} else if (parvec3dgeom) {
		ok = convertAstraGeometry(pVolGeom, parvec3dgeom, pParProjs, params);
	} else {
		ok = false;
	}

	return ok;
}




class AstraSIRT3d_internal {
public:
	SDimensions3D dims;
	SProjectorParams3D params;
	CUDAProjectionType3d projType;

	float* angles;
	float fOriginSourceDistance;
	float fOriginDetectorDistance;
	float fRelaxation;

	SConeProjection* projs;
	SPar3DProjection* parprojs;

	bool initialized;
	bool setStartReconstruction;

	bool useVolumeMask;
	bool useSinogramMask;

	// Input/output
	hipPitchedPtr D_projData;
	hipPitchedPtr D_volumeData;
	hipPitchedPtr D_maskData;
	hipPitchedPtr D_smaskData;

	SIRT sirt;
};

AstraSIRT3d::AstraSIRT3d()
{
	pData = new AstraSIRT3d_internal();

	pData->angles = 0;
	pData->D_projData.ptr = 0;
	pData->D_volumeData.ptr = 0;
	pData->D_maskData.ptr = 0;
	pData->D_smaskData.ptr = 0;

	pData->dims.iVolX = 0;
	pData->dims.iVolY = 0;
	pData->dims.iVolZ = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.iProjU = 0;
	pData->dims.iProjV = 0;

	pData->projs = 0;
	pData->parprojs = 0;

	pData->fRelaxation = 1.0f;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->useVolumeMask = false;
	pData->useSinogramMask = false;
}

AstraSIRT3d::~AstraSIRT3d()
{
	delete[] pData->angles;
	pData->angles = 0;

	delete[] pData->projs;
	pData->projs = 0;

	delete[] pData->parprojs;
	pData->parprojs = 0;

	hipFree(pData->D_projData.ptr);
	pData->D_projData.ptr = 0;

	hipFree(pData->D_volumeData.ptr);
	pData->D_volumeData.ptr = 0;

	hipFree(pData->D_maskData.ptr);
	pData->D_maskData.ptr = 0;

	hipFree(pData->D_smaskData.ptr);
	pData->D_smaskData.ptr = 0;

	delete pData;
	pData = 0;
}

bool AstraSIRT3d::setGeometry(const CVolumeGeometry3D* pVolGeom,
	                      const CProjectionGeometry3D* pProjGeom)
{
	if (pData->initialized)
		return false;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, pData->dims);

	if (!ok)
		return false;

	pData->projs = 0;
	pData->parprojs = 0;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pData->parprojs, pData->projs,
	                          pData->params);
	if (!ok)
		return false;

	if (pData->projs) {
		assert(pData->parprojs == 0);
		pData->projType = PROJ_CONE;
	} else {
		assert(pData->parprojs != 0);
		pData->projType = PROJ_PARALLEL;
	}

	return true;
}


bool AstraSIRT3d::enableSuperSampling(unsigned int iVoxelSuperSampling,
                                      unsigned int iDetectorSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iVoxelSuperSampling == 0 || iDetectorSuperSampling == 0)
		return false;

	pData->params.iRaysPerVoxelDim = iVoxelSuperSampling;
	pData->params.iRaysPerDetDim = iDetectorSuperSampling;

	return true;
}

bool AstraSIRT3d::setRelaxation(float r)
{
	if (pData->initialized)
		return false;

	pData->fRelaxation = r;

	return true;
}

bool AstraSIRT3d::enableVolumeMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->sirt.enableVolumeMask();
	pData->useVolumeMask = ok;

	return ok;
}

bool AstraSIRT3d::enableSinogramMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->sirt.enableSinogramMask();
	pData->useSinogramMask = ok;

	return ok;
}
	
bool AstraSIRT3d::setGPUIndex(int index)
{
	if (index != -1) {
		hipSetDevice(index);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool AstraSIRT3d::init()
{
	if (pData->initialized)
		return false;

	if (pData->dims.iVolX == 0 || pData->dims.iProjAngles == 0)
		return false;

	bool ok;

	if (pData->projType == PROJ_PARALLEL) {
		ok = pData->sirt.setPar3DGeometry(pData->dims, pData->parprojs, pData->params);
	} else {
		ok = pData->sirt.setConeGeometry(pData->dims, pData->projs, pData->params);
	}

	if (!ok)
		return false;

	pData->sirt.setRelaxation(pData->fRelaxation);

	ok = pData->sirt.init();
	if (!ok)
		return false;

	pData->D_volumeData = allocateVolumeData(pData->dims);
	ok = pData->D_volumeData.ptr;
	if (!ok)
		return false;

	pData->D_projData = allocateProjectionData(pData->dims);
	ok = pData->D_projData.ptr;
	if (!ok) {
		hipFree(pData->D_volumeData.ptr);
		pData->D_volumeData.ptr = 0;
		return false;
	}

	if (pData->useVolumeMask) {
		pData->D_maskData = allocateVolumeData(pData->dims);
		ok = pData->D_maskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			return false;
		}
	}

	if (pData->useSinogramMask) {
		pData->D_smaskData = allocateProjectionData(pData->dims);
		ok = pData->D_smaskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			hipFree(pData->D_maskData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			pData->D_maskData.ptr = 0;
			return false;
		}
	}

	pData->initialized = true;

	return true;
}

bool AstraSIRT3d::setMinConstraint(float fMin)
{
	if (!pData->initialized)
		return false;
	return pData->sirt.setMinConstraint(fMin);
}

bool AstraSIRT3d::setMaxConstraint(float fMax)
{
	if (!pData->initialized)
		return false;
	return pData->sirt.setMaxConstraint(fMax);
}

bool AstraSIRT3d::setSinogram(const float* pfSinogram,
                              unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copyProjectionsToDevice(pfSinogram, pData->D_projData, pData->dims, iSinogramPitch);

	if (!ok)
		return false;

	ok = pData->sirt.setBuffers(pData->D_volumeData, pData->D_projData);
	if (!ok)
		return false;

	pData->setStartReconstruction = false;

	return true;
}

bool AstraSIRT3d::setVolumeMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useVolumeMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyVolumeToDevice(pfMask, pData->D_maskData,
	                             pData->dims, iMaskPitch);
	if (!ok)
		return false;

	ok = pData->sirt.setVolumeMask(pData->D_maskData);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::setSinogramMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useSinogramMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyProjectionsToDevice(pfMask, pData->D_smaskData, pData->dims, iMaskPitch);

	if (!ok)
		return false;

	ok = pData->sirt.setSinogramMask(pData->D_smaskData);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::setStartReconstruction(const float* pfReconstruction,
                                         unsigned int iReconstructionPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copyVolumeToDevice(pfReconstruction, pData->D_volumeData,
	                             pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	pData->setStartReconstruction = true;

	return true;
}

bool AstraSIRT3d::iterate(unsigned int iIterations)
{
	if (!pData->initialized)
		return false;

	if (!pData->setStartReconstruction)
		zeroVolumeData(pData->D_volumeData, pData->dims);

	bool ok = pData->sirt.iterate(iIterations);
	if (!ok)
		return false;

	return true;
}

bool AstraSIRT3d::getReconstruction(float* pfReconstruction,
                                    unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, pData->D_volumeData,
	                               pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	return true;
}

float AstraSIRT3d::computeDiffNorm()
{
	if (!pData->initialized)
		return 0.0f; // FIXME: Error?

	return pData->sirt.computeDiffNorm();
}




class AstraCGLS3d_internal {
public:
	SDimensions3D dims;
	SProjectorParams3D params;
	CUDAProjectionType3d projType;

	float* angles;
	float fOriginSourceDistance;
	float fOriginDetectorDistance;

	SConeProjection* projs;
	SPar3DProjection* parprojs;

	bool initialized;
	bool setStartReconstruction;

	bool useVolumeMask;
	bool useSinogramMask;

	// Input/output
	hipPitchedPtr D_projData;
	hipPitchedPtr D_volumeData;
	hipPitchedPtr D_maskData;
	hipPitchedPtr D_smaskData;

	CGLS cgls;
};

AstraCGLS3d::AstraCGLS3d()
{
	pData = new AstraCGLS3d_internal();

	pData->angles = 0;
	pData->D_projData.ptr = 0;
	pData->D_volumeData.ptr = 0;
	pData->D_maskData.ptr = 0;
	pData->D_smaskData.ptr = 0;

	pData->dims.iVolX = 0;
	pData->dims.iVolY = 0;
	pData->dims.iVolZ = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.iProjU = 0;
	pData->dims.iProjV = 0;

	pData->projs = 0;
	pData->parprojs = 0;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->useVolumeMask = false;
	pData->useSinogramMask = false;
}

AstraCGLS3d::~AstraCGLS3d()
{
	delete[] pData->angles;
	pData->angles = 0;

	delete[] pData->projs;
	pData->projs = 0;

	delete[] pData->parprojs;
	pData->parprojs = 0;

	hipFree(pData->D_projData.ptr);
	pData->D_projData.ptr = 0;

	hipFree(pData->D_volumeData.ptr);
	pData->D_volumeData.ptr = 0;

	hipFree(pData->D_maskData.ptr);
	pData->D_maskData.ptr = 0;

	hipFree(pData->D_smaskData.ptr);
	pData->D_smaskData.ptr = 0;

	delete pData;
	pData = 0;
}

bool AstraCGLS3d::setGeometry(const CVolumeGeometry3D* pVolGeom,
	                      const CProjectionGeometry3D* pProjGeom)
{
	if (pData->initialized)
		return false;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, pData->dims);

	if (!ok)
		return false;

	pData->projs = 0;
	pData->parprojs = 0;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pData->parprojs, pData->projs,
	                          pData->params);
	if (!ok)
		return false;

	if (pData->projs) {
		assert(pData->parprojs == 0);
		pData->projType = PROJ_CONE;
	} else {
		assert(pData->parprojs != 0);
		pData->projType = PROJ_PARALLEL;
	}

	return true;
}

bool AstraCGLS3d::enableSuperSampling(unsigned int iVoxelSuperSampling,
                                      unsigned int iDetectorSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iVoxelSuperSampling == 0 || iDetectorSuperSampling == 0)
		return false;

	pData->params.iRaysPerVoxelDim = iVoxelSuperSampling;
	pData->params.iRaysPerDetDim = iDetectorSuperSampling;

	return true;
}

bool AstraCGLS3d::enableVolumeMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->cgls.enableVolumeMask();
	pData->useVolumeMask = ok;

	return ok;
}

#if 0
bool AstraCGLS3d::enableSinogramMask()
{
	if (pData->initialized)
		return false;

	bool ok = pData->cgls.enableSinogramMask();
	pData->useSinogramMask = ok;

	return ok;
}
#endif
	
bool AstraCGLS3d::setGPUIndex(int index)
{
	if (index != -1) {
		hipSetDevice(index);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool AstraCGLS3d::init()
{
	if (pData->initialized)
		return false;

	if (pData->dims.iVolX == 0 || pData->dims.iProjAngles == 0)
		return false;

	bool ok;

	if (pData->projType == PROJ_PARALLEL) {
		ok = pData->cgls.setPar3DGeometry(pData->dims, pData->parprojs, pData->params);
	} else {
		ok = pData->cgls.setConeGeometry(pData->dims, pData->projs, pData->params);
	}

	if (!ok)
		return false;

	ok = pData->cgls.init();
	if (!ok)
		return false;

	pData->D_volumeData = allocateVolumeData(pData->dims);
	ok = pData->D_volumeData.ptr;
	if (!ok)
		return false;

	pData->D_projData = allocateProjectionData(pData->dims);
	ok = pData->D_projData.ptr;
	if (!ok) {
		hipFree(pData->D_volumeData.ptr);
		pData->D_volumeData.ptr = 0;
		return false;
	}

	if (pData->useVolumeMask) {
		pData->D_maskData = allocateVolumeData(pData->dims);
		ok = pData->D_maskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			return false;
		}
	}

	if (pData->useSinogramMask) {
		pData->D_smaskData = allocateProjectionData(pData->dims);
		ok = pData->D_smaskData.ptr;
		if (!ok) {
			hipFree(pData->D_volumeData.ptr);
			hipFree(pData->D_projData.ptr);
			hipFree(pData->D_maskData.ptr);
			pData->D_volumeData.ptr = 0;
			pData->D_projData.ptr = 0;
			pData->D_maskData.ptr = 0;
			return false;
		}
	}

	pData->initialized = true;

	return true;
}

#if 0
bool AstraCGLS3d::setMinConstraint(float fMin)
{
	if (!pData->initialized)
		return false;
	return pData->cgls.setMinConstraint(fMin);
}

bool AstraCGLS3d::setMaxConstraint(float fMax)
{
	if (!pData->initialized)
		return false;
	return pData->cgls.setMaxConstraint(fMax);
}
#endif

bool AstraCGLS3d::setSinogram(const float* pfSinogram,
                              unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copyProjectionsToDevice(pfSinogram, pData->D_projData, pData->dims, iSinogramPitch);

	if (!ok)
		return false;

	ok = pData->cgls.setBuffers(pData->D_volumeData, pData->D_projData);
	if (!ok)
		return false;

	pData->setStartReconstruction = false;

	return true;
}

bool AstraCGLS3d::setVolumeMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useVolumeMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyVolumeToDevice(pfMask, pData->D_maskData,
	                             pData->dims, iMaskPitch);
	if (!ok)
		return false;

	ok = pData->cgls.setVolumeMask(pData->D_maskData);
	if (!ok)
		return false;

	return true;
}

#if 0
bool AstraCGLS3d::setSinogramMask(const float* pfMask, unsigned int iMaskPitch)
{
	if (!pData->initialized)
		return false;
	if (!pData->useSinogramMask)
		return false;
	if (!pfMask)
		return false;

	bool ok = copyProjectionsToDevice(pfMask, pData->D_smaskData, pData->dims, iMaskPitch);

	if (!ok)
		return false;

	ok = pData->cgls.setSinogramMask(pData->D_smaskData);
	if (!ok)
		return false;

	return true;
}
#endif

bool AstraCGLS3d::setStartReconstruction(const float* pfReconstruction,
                                         unsigned int iReconstructionPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copyVolumeToDevice(pfReconstruction, pData->D_volumeData,
	                             pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	pData->setStartReconstruction = true;

	return true;
}

bool AstraCGLS3d::iterate(unsigned int iIterations)
{
	if (!pData->initialized)
		return false;

	if (!pData->setStartReconstruction)
		zeroVolumeData(pData->D_volumeData, pData->dims);

	bool ok = pData->cgls.iterate(iIterations);
	if (!ok)
		return false;

	return true;
}

bool AstraCGLS3d::getReconstruction(float* pfReconstruction,
                                    unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, pData->D_volumeData,
	                               pData->dims, iReconstructionPitch);
	if (!ok)
		return false;

	return true;
}

float AstraCGLS3d::computeDiffNorm()
{
	if (!pData->initialized)
		return 0.0f; // FIXME: Error?

	return pData->cgls.computeDiffNorm();
}



bool astraCudaFP(const float* pfVolume, float* pfProjections,
                 const CVolumeGeometry3D* pVolGeom,
                 const CProjectionGeometry3D* pProjGeom,
                 int iGPUIndex, int iDetectorSuperSampling,
                 Cuda3DProjectionKernel projKernel)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerDetDim = iDetectorSuperSampling;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	if (iDetectorSuperSampling == 0)
		return false;

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);


	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}


	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok)
		return false;

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		hipFree(D_volumeData.ptr);
		return false;
	}

	ok &= copyVolumeToDevice(pfVolume, D_volumeData, dims, dims.iVolX);

	ok &= zeroProjectionData(D_projData, dims);

	if (!ok) {
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	if (pParProjs) {
		switch (projKernel) {
		case ker3d_default:
			ok &= Par3DFP(D_volumeData, D_projData, dims, pParProjs, params);
			break;
		case ker3d_sum_square_weights:
			ok &= Par3DFP_SumSqW(D_volumeData, D_projData, dims, pParProjs, params);
			break;
		default:
			assert(false);
		}
	} else {
		switch (projKernel) {
		case ker3d_default:
			ok &= ConeFP(D_volumeData, D_projData, dims, pConeProjs, params);
			break;
		default:
			assert(false);
		}
	}

	ok &= copyProjectionsFromDevice(pfProjections, D_projData,
	                                dims, dims.iProjU);


	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	return ok;

}


bool astraCudaBP(float* pfVolume, const float* pfProjections,
                 const CVolumeGeometry3D* pVolGeom,
                 const CProjectionGeometry3D* pProjGeom,
                 int iGPUIndex, int iVoxelSuperSampling)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerVoxelDim = iVoxelSuperSampling;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess) {
			delete[] pParProjs;
			delete[] pConeProjs;
			return false;
		}
	}


	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		return false;
	}

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		hipFree(D_volumeData.ptr);
		return false;
	}

	ok &= copyProjectionsToDevice(pfProjections, D_projData,
	                              dims, dims.iProjU);

	ok &= zeroVolumeData(D_volumeData, dims);

	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	if (pParProjs)
		ok &= Par3DBP(D_volumeData, D_projData, dims, pParProjs, params);
	else
		ok &= ConeBP(D_volumeData, D_projData, dims, pConeProjs, params);

	ok &= copyVolumeFromDevice(pfVolume, D_volumeData, dims, dims.iVolX);

	delete[] pParProjs;
	delete[] pConeProjs;

	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	return ok;

}


// This computes the column weights, divides by them, and adds the
// result to the current volume. This is both more expensive and more
// GPU memory intensive than the regular BP, but allows saving system RAM.
bool astraCudaBP_SIRTWeighted(float* pfVolume,
                      const float* pfProjections,
                      const CVolumeGeometry3D* pVolGeom,
                      const CProjectionGeometry3D* pProjGeom,
                      int iGPUIndex, int iVoxelSuperSampling)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	params.iRaysPerVoxelDim = iVoxelSuperSampling;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;


	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess) {
			delete[] pParProjs;
			delete[] pConeProjs;
			return false;
		}
	}


	hipPitchedPtr D_pixelWeight = allocateVolumeData(dims);
	ok = D_pixelWeight.ptr;
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		return false;
	}

	hipPitchedPtr D_volumeData = allocateVolumeData(dims);
	ok = D_volumeData.ptr;
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		hipFree(D_pixelWeight.ptr);
		return false;
	}

	hipPitchedPtr D_projData = allocateProjectionData(dims);
	ok = D_projData.ptr;
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		return false;
	}

	// Compute weights
	ok &= zeroVolumeData(D_pixelWeight, dims);
	processSino3D<opSet>(D_projData, 1.0f, dims);

	if (pParProjs)
		ok &= Par3DBP(D_pixelWeight, D_projData, dims, pParProjs, params);
	else
		ok &= ConeBP(D_pixelWeight, D_projData, dims, pConeProjs, params);

	processVol3D<opInvert>(D_pixelWeight, dims);
	if (!ok) {
		delete[] pParProjs;
		delete[] pConeProjs;
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}

	ok &= copyProjectionsToDevice(pfProjections, D_projData,
	                              dims, dims.iProjU);
	ok &= zeroVolumeData(D_volumeData, dims);
	// Do BP into D_volumeData
	if (pParProjs)
		ok &= Par3DBP(D_volumeData, D_projData, dims, pParProjs, params);
	else
		ok &= ConeBP(D_volumeData, D_projData, dims, pConeProjs, params);

	// Multiply with weights
	processVol3D<opMul>(D_volumeData, D_pixelWeight, dims);

	// Upload previous iterate to D_pixelWeight...
	ok &= copyVolumeToDevice(pfVolume, D_pixelWeight, dims, dims.iVolX);
	if (!ok) {
		hipFree(D_pixelWeight.ptr);
		hipFree(D_volumeData.ptr);
		hipFree(D_projData.ptr);
		return false;
	}
	// ...and add it to the weighted BP
	processVol3D<opAdd>(D_volumeData, D_pixelWeight, dims);

	// Then copy the result back
	ok &= copyVolumeFromDevice(pfVolume, D_volumeData, dims, dims.iVolX);


	hipFree(D_pixelWeight.ptr);
	hipFree(D_volumeData.ptr);
	hipFree(D_projData.ptr);

	delete[] pParProjs;
	delete[] pConeProjs;

	return ok;

}

_AstraExport bool uploadMultipleProjections(CFloat32ProjectionData3D *proj,
                                         const float *data,
                                         unsigned int y_min, unsigned int y_max)
{
	assert(proj->getStorage()->isGPU());
	CDataGPU *storage = dynamic_cast<CDataGPU*>(proj->getStorage());
	astraCUDA3d::MemHandle3D hnd = storage->getHandle();

	astraCUDA3d::SDimensions3D dims1;
	dims1.iProjU = proj->getDetectorColCount();
	dims1.iProjV = proj->getDetectorRowCount();
	dims1.iProjAngles = y_max - y_min + 1;

	hipPitchedPtr D_proj = allocateProjectionData(dims1);
	bool ok = copyProjectionsToDevice(data, D_proj, dims1);
	if (!ok) {
		ASTRA_ERROR("Failed to upload projection to GPU");
		return false;
	}

	astraCUDA3d::MemHandle3D hnd1 = astraCUDA3d::wrapHandle(
			(float *)D_proj.ptr,
			dims1.iProjU, dims1.iProjAngles, dims1.iProjV,
			D_proj.pitch / sizeof(float));

	astraCUDA3d::SSubDimensions3D subdims;
	subdims.nx = dims1.iProjU;
	subdims.ny = proj->getAngleCount();
	subdims.nz = dims1.iProjV;
	subdims.pitch = D_proj.pitch / sizeof(float); // FIXME: Pitch for wrong obj!
	subdims.subnx = dims1.iProjU;
	subdims.subny = dims1.iProjAngles;
	subdims.subnz = dims1.iProjV;
	subdims.subx = 0;
	subdims.suby = y_min;
	subdims.subz = 0;

	ok = astraCUDA3d::copyIntoArray(hnd, hnd1, subdims);
	if (!ok) {
		ASTRA_ERROR("Failed to copy projection into 3d data");
		return false;
	}

	hipFree(D_proj.ptr);
	return true;
}


}
