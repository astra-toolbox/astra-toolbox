#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "par3d_fp.h"
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_par3DProjTexture;

namespace astraCUDA3d {

#define ZSIZE 6
static const unsigned int g_volBlockZ = ZSIZE;

static const unsigned int g_anglesPerBlock = 32;
static const unsigned int g_volBlockX = 16;
static const unsigned int g_volBlockY = 32;

static const unsigned g_MaxAngles = 1024;

__constant__ float gC_C[8*g_MaxAngles];


static bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_par3DProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_par3DProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_par3DProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_par3DProjTexture.filterMode = hipFilterModeLinear;
	gT_par3DProjTexture.normalized = false;

	hipBindTextureToArray(gT_par3DProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


__global__ void dev_par3D_BP(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, const SDimensions3D dims, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
	//            y = z


	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f;
	float fY = Y - 0.5f*dims.iVolY + 0.5f;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	float Z[ZSIZE];
	for(int i=0; i < ZSIZE; i++)
		Z[i] = 0.0f;

	{
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			float4 fCu = make_float4(gC_C[8*angle+0], gC_C[8*angle+1], gC_C[8*angle+2], gC_C[8*angle+3]);
			float4 fCv = make_float4(gC_C[8*angle+4], gC_C[8*angle+5], gC_C[8*angle+6], gC_C[8*angle+7]);

			float fU = fCu.w + fX * fCu.x + fY * fCu.y + fZ * fCu.z;
			float fV = fCv.w + fX * fCv.x + fY * fCv.y + fZ * fCv.z;

			for (int idx = 0; idx < ZSIZE; ++idx) {

				float fVal = tex3D(gT_par3DProjTexture, fU, fAngle, fV);
				Z[idx] += fVal;

				fU += fCu.z;
				fV += fCv.z;
			}

		}
	}

	int endZ = ZSIZE;
	if (endZ > dims.iVolZ - startZ)
		endZ = dims.iVolZ - startZ;

	for(int i=0; i < endZ; i++)
		volData[((startZ+i)*dims.iVolY+Y)*volPitch+X] += Z[i] * fOutputScale;
}

// supersampling version
__global__ void dev_par3D_BP_SS(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, const SDimensions3D dims, int iRaysPerVoxelDim, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;

	const float fSubStep = 1.0f/iRaysPerVoxelDim;

	fOutputScale /= (iRaysPerVoxelDim*iRaysPerVoxelDim*iRaysPerVoxelDim);


	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			const float fCux = gC_C[8*angle+0];
			const float fCuy = gC_C[8*angle+1];
			const float fCuz = gC_C[8*angle+2];
			const float fCuc = gC_C[8*angle+3];
			const float fCvx = gC_C[8*angle+4];
			const float fCvy = gC_C[8*angle+5];
			const float fCvz = gC_C[8*angle+6];
			const float fCvc = gC_C[8*angle+7];

			float fXs = fX;
			for (int iSubX = 0; iSubX < iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < iRaysPerVoxelDim; ++iSubZ) {

				const float fU = fCuc + fXs * fCux + fYs * fCuy + fZs * fCuz;
				const float fV = fCvc + fXs * fCvx + fYs * fCvy + fZs * fCvz;

				fVal += tex3D(gT_par3DProjTexture, fU, fAngle, fV);
				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal * fOutputScale;
	}

}

bool Par3DBP_Array(hipPitchedPtr D_volumeData,
                   hipArray *D_projArray,
                   const SDimensions3D& dims, const SPar3DProjection* angles,
                   const SProjectorParams3D& params)
{
	bindProjDataTexture(D_projArray);

	float fOutputScale = params.fOutputScale * params.fVolScaleX * params.fVolScaleY * params.fVolScaleZ;

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		// transfer angles to constant memory
		float* tmp = new float[8*dims.iProjAngles];

		// NB: We increment angles at the end of the loop body.


		// TODO: Use functions from dims3d.cu for this:

#define TRANSFER_TO_CONSTANT(expr,name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[8*i + name] = (expr) ; } while (0)

#define DENOM (angles[i].fRayX*angles[i].fDetUY*angles[i].fDetVZ - angles[i].fRayX*angles[i].fDetUZ*angles[i].fDetVY - angles[i].fRayY*angles[i].fDetUX*angles[i].fDetVZ + angles[i].fRayY*angles[i].fDetUZ*angles[i].fDetVX + angles[i].fRayZ*angles[i].fDetUX*angles[i].fDetVY - angles[i].fRayZ*angles[i].fDetUY*angles[i].fDetVX)

		TRANSFER_TO_CONSTANT( ( - (angles[i].fRayY*angles[i].fDetVZ - angles[i].fRayZ*angles[i].fDetVY)) / DENOM , 0 );
		TRANSFER_TO_CONSTANT( ( (angles[i].fRayX*angles[i].fDetVZ - angles[i].fRayZ*angles[i].fDetVX)) / DENOM , 1 );
		TRANSFER_TO_CONSTANT( (- (angles[i].fRayX*angles[i].fDetVY - angles[i].fRayY*angles[i].fDetVX) ) / DENOM , 2 );
		TRANSFER_TO_CONSTANT( (-(angles[i].fDetSY*angles[i].fDetVZ - angles[i].fDetSZ*angles[i].fDetVY)*angles[i].fRayX + (angles[i].fRayY*angles[i].fDetVZ - angles[i].fRayZ*angles[i].fDetVY)*angles[i].fDetSX - (angles[i].fRayY*angles[i].fDetSZ - angles[i].fRayZ*angles[i].fDetSY)*angles[i].fDetVX) / DENOM , 3 );

		TRANSFER_TO_CONSTANT( ((angles[i].fRayY*angles[i].fDetUZ - angles[i].fRayZ*angles[i].fDetUY) ) / DENOM , 4 );
		TRANSFER_TO_CONSTANT( (- (angles[i].fRayX*angles[i].fDetUZ - angles[i].fRayZ*angles[i].fDetUX) ) / DENOM , 5 );
		TRANSFER_TO_CONSTANT( ((angles[i].fRayX*angles[i].fDetUY - angles[i].fRayY*angles[i].fDetUX) ) / DENOM , 6 );
		TRANSFER_TO_CONSTANT( ((angles[i].fDetSY*angles[i].fDetUZ - angles[i].fDetSZ*angles[i].fDetUY)*angles[i].fRayX - (angles[i].fRayY*angles[i].fDetUZ - angles[i].fRayZ*angles[i].fDetUY)*angles[i].fDetSX + (angles[i].fRayY*angles[i].fDetSZ - angles[i].fRayZ*angles[i].fDetSY)*angles[i].fDetUX ) / DENOM , 7 );

#undef TRANSFER_TO_CONSTANT
#undef DENOM
		hipMemcpyToSymbol(HIP_SYMBOL(gC_C), tmp, angleCount*8*sizeof(float), 0, hipMemcpyHostToDevice); 

		delete[] tmp;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX+g_volBlockX-1)/g_volBlockX)*((dims.iVolY+g_volBlockY-1)/g_volBlockY), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
			// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (params.iRaysPerVoxelDim == 1)
				dev_par3D_BP<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims, fOutputScale);
			else
				dev_par3D_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims, params.iRaysPerVoxelDim, fOutputScale);
		}

		cudaTextForceKernelsCompletion();

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}

	return true;
}

bool Par3DBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SPar3DProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = Par3DBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}


}

#ifdef STANDALONE
int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 180;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
#if 0
		if (i == 128) {
			for (unsigned int j = 0; j < 256*256; ++j)
				slice[j] = 0.0f;
		}
#endif 
	}


	SPar3DProjection angle[180];
	angle[0].fRayX = 1;
	angle[0].fRayY = 0;
	angle[0].fRayZ = 0;

	angle[0].fDetSX = 512;
	angle[0].fDetSY = -256;
	angle[0].fDetSZ = -256;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = 1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 180; ++i) {
		angle[i] = angle[0];
		ROTATE0(Ray, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
		ROTATE0(DetV, i, i*2*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::Par3DFP(volData, projData, dims, angle, 1.0f);
#if 1
	float* bufs = new float[180*512];

	for (int i = 0; i < 512; ++i) {
		hipMemcpy(bufs, ((float*)projData.ptr)+180*512*i, 180*512*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

		char fname[20];
		sprintf(fname, "sino%03d.png", i);
		saveImage(fname, 180, 512, bufs, 0, 512);
	}

	float* bufp = new float[512*512];

	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "proj%03d.png", i);
		saveImage(fname, 512, 512, bufp, 0, 512);
	}
#endif		
	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 0.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
	}

	astraCUDA3d::Par3DBP(volData, projData, dims, angle, 1.0f);
#if 1
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)volData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", volData.pitch, volData.xsize, volData.ysize);

		char fname[20];
		sprintf(fname, "vol%03d.png", i);
		saveImage(fname, 256, 256, buf, 0, 60000);
	}
#endif

}
#endif
