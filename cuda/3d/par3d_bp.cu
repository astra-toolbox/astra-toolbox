#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>

namespace astraCUDA3d {

static const unsigned int g_volBlockZ = 6;

static const unsigned int g_anglesPerBlock = 32;
static const unsigned int g_volBlockX = 16;
static const unsigned int g_volBlockY = 32;

static const unsigned g_MaxAngles = 1024;

struct DevPar3DParams {
	float4 fNumU;
	float4 fNumV;
};

__constant__ DevPar3DParams gC_C[g_MaxAngles];
__constant__ float gC_scale[g_MaxAngles];


template<unsigned int ZSIZE>
__global__ void dev_par3D_BP(void* D_volData, unsigned int volPitch, hipTextureObject_t tex, int startAngle, int angleOffset, const SDimensions3D dims, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
	//            y = z


	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f;
	float fY = Y - 0.5f*dims.iVolY + 0.5f;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	float Z[ZSIZE];
	for(int i=0; i < ZSIZE; i++)
		Z[i] = 0.0f;

	{
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			float4 fCu = gC_C[angle].fNumU;
			float4 fCv = gC_C[angle].fNumV;
			float fS = gC_scale[angle];

			float fU = fCu.w + fX * fCu.x + fY * fCu.y + fZ * fCu.z;
			float fV = fCv.w + fX * fCv.x + fY * fCv.y + fZ * fCv.z;

			for (int idx = 0; idx < ZSIZE; ++idx) {

				float fVal = tex3D<float>(tex, fU, fAngle, fV);
				Z[idx] += fVal * fS;

				fU += fCu.z;
				fV += fCv.z;
			}

		}
	}

	int endZ = ZSIZE;
	if (endZ > dims.iVolZ - startZ)
		endZ = dims.iVolZ - startZ;

	for(int i=0; i < endZ; i++)
		volData[((startZ+i)*dims.iVolY+Y)*volPitch+X] += Z[i] * fOutputScale;
}

// supersampling version
__global__ void dev_par3D_BP_SS(void* D_volData, unsigned int volPitch, hipTextureObject_t tex, int startAngle, int angleOffset, const SDimensions3D dims, int iRaysPerVoxelDim, float fOutputScale)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/iRaysPerVoxelDim;

	const float fSubStep = 1.0f/iRaysPerVoxelDim;

	fOutputScale /= (iRaysPerVoxelDim*iRaysPerVoxelDim*iRaysPerVoxelDim);


	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{
			float4 fCu = gC_C[angle].fNumU;
			float4 fCv = gC_C[angle].fNumV;
			float fS = gC_scale[angle];

			float fXs = fX;
			for (int iSubX = 0; iSubX < iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < iRaysPerVoxelDim; ++iSubZ) {

				const float fU = fCu.w + fXs * fCu.x + fYs * fCu.y + fZs * fCu.z;
				const float fV = fCv.w + fXs * fCv.x + fYs * fCv.y + fZs * fCv.z;

				fVal += tex3D<float>(tex, fU, fAngle, fV) * fS;
				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal * fOutputScale;
	}

}

bool transferConstants(const SPar3DProjection* angles, unsigned int iProjAngles, const SProjectorParams3D& params)
{
	DevPar3DParams *p = new DevPar3DParams[iProjAngles];
	float *s = new float[iProjAngles];

	for (unsigned int i = 0; i < iProjAngles; ++i) {
		Vec3 u(angles[i].fDetUX, angles[i].fDetUY, angles[i].fDetUZ);
		Vec3 v(angles[i].fDetVX, angles[i].fDetVY, angles[i].fDetVZ);
		Vec3 r(angles[i].fRayX, angles[i].fRayY, angles[i].fRayZ);
		Vec3 d(angles[i].fDetSX, angles[i].fDetSY, angles[i].fDetSZ);

		double fDen = det3(r,u,v);
		p[i].fNumU.x = -det3x(r,v) / fDen;
		p[i].fNumU.y = -det3y(r,v) / fDen;
		p[i].fNumU.z = -det3z(r,v) / fDen;
		p[i].fNumU.w = -det3(r,d,v) / fDen;
		p[i].fNumV.x = det3x(r,u) / fDen;
		p[i].fNumV.y = det3y(r,u) / fDen;
		p[i].fNumV.z = det3z(r,u) / fDen;
		p[i].fNumV.w = det3(r,d,u) / fDen;

		s[i] = 1.0 / scaled_cross3(u,v,Vec3(params.fVolScaleX,params.fVolScaleY,params.fVolScaleZ)).norm();
	}

	hipMemcpyToSymbol(HIP_SYMBOL(gC_C), p, iProjAngles*sizeof(DevPar3DParams), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(gC_scale), s, iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);

	delete[] p;
	delete[] s;

	return true;
}

bool Par3DBP_Array(hipPitchedPtr D_volumeData,
                   hipArray *D_projArray,
                   const SDimensions3D& dims, const SPar3DProjection* angles,
                   const SProjectorParams3D& params)
{
	hipTextureObject_t D_texObj;
	if (!createTextureObject3D(D_projArray, D_texObj))
		return false;

	float fOutputScale = params.fOutputScale * params.fVolScaleX * params.fVolScaleY * params.fVolScaleZ;

	bool ok = true;

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		ok = transferConstants(angles, angleCount, params);
		if (!ok)
			break;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX+g_volBlockX-1)/g_volBlockX)*((dims.iVolY+g_volBlockY-1)/g_volBlockY), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
			// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (params.iRaysPerVoxelDim == 1) {
				if (dims.iVolZ == 1) {
					dev_par3D_BP<1><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				} else {
					dev_par3D_BP<g_volBlockZ><<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, fOutputScale);
				}
			} else
				dev_par3D_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), D_texObj, i, th, dims, params.iRaysPerVoxelDim, fOutputScale);
		}

		// TODO: Consider not synchronizing here, if possible.
		ok = checkCuda(hipDeviceSynchronize(), "cone_bp");
		if (!ok)
			break;

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}

	hipDestroyTextureObject(D_texObj);

	return true;
}

bool Par3DBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SPar3DProjection* angles,
            const SProjectorParams3D& params)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = Par3DBP_Array(D_volumeData, cuArray, dims, angles, params);

	hipFreeArray(cuArray);

	return ret;
}


}
