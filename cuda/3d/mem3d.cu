#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "util3d.h"

#include "mem3d.h"

#include "astra3d.h"
#include "cone_fp.h"
#include "cone_bp.h"
#include "par3d_fp.h"
#include "par3d_bp.h"
#include "fdk.h"

#include "astra/Logging.h"


namespace astraCUDA3d {


struct SMemHandle3D_internal
{
	hipPitchedPtr ptr;
	unsigned int nx;
	unsigned int ny;
	unsigned int nz;
};

size_t availableGPUMemory()
{
	size_t free, total;
	hipError_t err = hipMemGetInfo(&free, &total);
	if (err != hipSuccess)
		return 0;
	return free;
}

int maxBlockDimension()
{
	int dev;
	hipError_t err = hipGetDevice(&dev);
	if (err != hipSuccess) {
		ASTRA_WARN("Error querying device");
		return 0;
	}

	hipDeviceProp_t props;
	err = hipGetDeviceProperties(&props, dev);
	if (err != hipSuccess) {
		ASTRA_WARN("Error querying device %d properties", dev);
		return 0;
	}

	return std::min(props.maxTexture3D[0], std::min(props.maxTexture3D[1], props.maxTexture3D[2]));
}

MemHandle3D allocateGPUMemory(unsigned int x, unsigned int y, unsigned int z, Mem3DZeroMode zero)
{
	SMemHandle3D_internal hnd;
	hnd.nx = x;
	hnd.ny = y;
	hnd.nz = z;

	size_t free = availableGPUMemory();

	hipError_t err;
	err = hipMalloc3D(&hnd.ptr, make_hipExtent(sizeof(float)*x, y, z));

	if (err != hipSuccess) {
		return MemHandle3D();
	}

	size_t free2 = availableGPUMemory();

	ASTRA_DEBUG("Allocated %d x %d x %d on GPU. (Pre: %lu, post: %lu)", x, y, z, free, free2);



	if (zero == INIT_ZERO) {
		err = hipMemset3D(hnd.ptr, 0, make_hipExtent(sizeof(float)*x, y, z));
		if (err != hipSuccess) {
			hipFree(hnd.ptr.ptr);
			return MemHandle3D();
		}
	}

	MemHandle3D ret;
	ret.d = boost::shared_ptr<SMemHandle3D_internal>(new SMemHandle3D_internal);
	*ret.d = hnd;

	return ret;
}

bool freeGPUMemory(MemHandle3D handle)
{
	size_t free = availableGPUMemory();
	hipError_t err = hipFree(handle.d->ptr.ptr);
	size_t free2 = availableGPUMemory();

	ASTRA_DEBUG("Freeing memory. (Pre: %lu, post: %lu)", free, free2);

	return err == hipSuccess;
}

bool copyToGPUMemory(const float *src, MemHandle3D dst, const SSubDimensions3D &pos)
{
	ASTRA_DEBUG("Copying %d x %d x %d to GPU", pos.subnx, pos.subny, pos.subnz);
	ASTRA_DEBUG("Offset %d,%d,%d", pos.subx, pos.suby, pos.subz);
	hipPitchedPtr s;
	s.ptr = (void*)src; // const cast away
	s.pitch = pos.pitch * sizeof(float);
	s.xsize = pos.nx * sizeof(float);
	s.ysize = pos.ny;
	ASTRA_DEBUG("Pitch %d, xsize %d, ysize %d", s.pitch, s.xsize, s.ysize);

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = make_hipPos(pos.subx * sizeof(float), pos.suby, pos.subz);
	p.srcPtr = s;

	p.dstArray = 0;
	p.dstPos = make_hipPos(0, 0, 0);
	p.dstPtr = dst.d->ptr;

	p.extent = make_hipExtent(pos.subnx * sizeof(float), pos.subny, pos.subnz);

	p.kind = hipMemcpyHostToDevice;

	hipError_t err = hipMemcpy3D(&p);

	return err == hipSuccess;
}


bool copyFromGPUMemory(float *dst, MemHandle3D src, const SSubDimensions3D &pos)
{
	ASTRA_DEBUG("Copying %d x %d x %d from GPU", pos.subnx, pos.subny, pos.subnz);
	ASTRA_DEBUG("Offset %d,%d,%d", pos.subx, pos.suby, pos.subz);
	hipPitchedPtr d;
	d.ptr = (void*)dst;
	d.pitch = pos.pitch * sizeof(float);
	d.xsize = pos.nx * sizeof(float);
	d.ysize = pos.ny;
	ASTRA_DEBUG("Pitch %d, xsize %d, ysize %d", d.pitch, d.xsize, d.ysize);

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = make_hipPos(0, 0, 0);
	p.srcPtr = src.d->ptr;

	p.dstArray = 0;
	p.dstPos = make_hipPos(pos.subx * sizeof(float), pos.suby, pos.subz);
	p.dstPtr = d;

	p.extent = make_hipExtent(pos.subnx * sizeof(float), pos.subny, pos.subnz);

	p.kind = hipMemcpyDeviceToHost;

	hipError_t err = hipMemcpy3D(&p);

	return err == hipSuccess;

}


bool FP(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, int iDetectorSuperSampling, astra::Cuda3DProjectionKernel projKernel)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

#if 1
	params.iRaysPerDetDim = iDetectorSuperSampling;
	if (iDetectorSuperSampling == 0)
		return false;
#else
	astra::Cuda3DProjectionKernel projKernel = astra::ker3d_default;
#endif


	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (pParProjs) {
#if 0
		for (int i = 0; i < dims.iProjAngles; ++i) {
			ASTRA_DEBUG("Vec: %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f\n",
			    pParProjs[i].fRayX, pParProjs[i].fRayY, pParProjs[i].fRayZ,
			    pParProjs[i].fDetSX, pParProjs[i].fDetSY, pParProjs[i].fDetSZ,
			    pParProjs[i].fDetUX, pParProjs[i].fDetUY, pParProjs[i].fDetUZ,
			    pParProjs[i].fDetVX, pParProjs[i].fDetVY, pParProjs[i].fDetVZ);
		}
#endif

		switch (projKernel) {
		case astra::ker3d_default:
			ok &= Par3DFP(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
			break;
		case astra::ker3d_sum_square_weights:
			ok &= Par3DFP_SumSqW(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
			break;
		default:
			ok = false;
		}
	} else {
		switch (projKernel) {
		case astra::ker3d_default:
			ok &= ConeFP(volData.d->ptr, projData.d->ptr, dims, pConeProjs, params);
			break;
		default:
			ok = false;
		}
	}

	return ok;
}

bool BP(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, int iVoxelSuperSampling)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

#if 1
	params.iRaysPerVoxelDim = iVoxelSuperSampling;
#endif

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (pParProjs)
		ok &= Par3DBP(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
	else
		ok &= ConeBP(volData.d->ptr, projData.d->ptr, dims, pConeProjs, params);

	return ok;

}

bool FDK(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, bool bShortScan)
{
	SDimensions3D dims;
	SProjectorParams3D params;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (!ok || !pConeProjs)
		return false;

	ok &= FDK(volData.d->ptr, projData.d->ptr, pConeProjs, dims, params, bShortScan);

	return ok;



}




}
