#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/mem3d.h"
#include "astra/cuda/3d/astra3d.h"
#include "astra/cuda/3d/cone_fp.h"
#include "astra/cuda/3d/cone_bp.h"
#include "astra/cuda/3d/par3d_fp.h"
#include "astra/cuda/3d/par3d_bp.h"
#include "astra/cuda/3d/fdk.h"

#include "astra/cuda/2d/astra.h"

#include "astra/Logging.h"

#include <cstdio>
#include <cassert>



namespace astraCUDA3d {


struct SMemHandle3D_internal
{
	hipPitchedPtr ptr;
	hipArray *arr;
	unsigned int nx;
	unsigned int ny;
	unsigned int nz;
};

int maxBlockDimension()
{
	int dev;
	if (!checkCuda(hipGetDevice(&dev), "maxBlockDimension getDevice")) {
		ASTRA_WARN("Error querying device");
		return 0;
	}

	hipDeviceProp_t props;
	if (!checkCuda(hipGetDeviceProperties(&props, dev), "maxBlockDimension getDviceProps")) {
		ASTRA_WARN("Error querying device %d properties", dev);
		return 0;
	}

	return std::min(props.maxTexture3D[0], std::min(props.maxTexture3D[1], props.maxTexture3D[2]));
}

MemHandle3D allocateGPUMemory(unsigned int x, unsigned int y, unsigned int z, Mem3DZeroMode zero)
{
	SMemHandle3D_internal hnd;
	hnd.nx = x;
	hnd.ny = y;
	hnd.nz = z;
	hnd.arr = 0;

	size_t free = astraCUDA::availableGPUMemory();

	if (!checkCuda(hipMalloc3D(&hnd.ptr, make_hipExtent(sizeof(float)*x, y, z)), "allocateGPUMemory malloc3d")) {
		return MemHandle3D();
	}

	size_t free2 = astraCUDA::availableGPUMemory();

	ASTRA_DEBUG("Allocated %d x %d x %d on GPU. (Pre: %lu, post: %lu)", x, y, z, free, free2);



	if (zero == INIT_ZERO) {
		if (!checkCuda(hipMemset3D(hnd.ptr, 0, make_hipExtent(sizeof(float)*x, y, z)), "allocateGPUMemory memset3d")) {
			hipFree(hnd.ptr.ptr);
			return MemHandle3D();
		}
	}

	MemHandle3D ret;
	ret.d = std::make_shared<SMemHandle3D_internal>();
	*ret.d = hnd;

	return ret;
}

bool zeroGPUMemory(MemHandle3D handle, unsigned int x, unsigned int y, unsigned int z)
{
	SMemHandle3D_internal& hnd = *handle.d.get();
	assert(!hnd.arr);
	return checkCuda(hipMemset3D(hnd.ptr, 0, make_hipExtent(sizeof(float)*x, y, z)), "zeroGPUMemory");
}

bool freeGPUMemory(MemHandle3D handle)
{
	size_t free = astraCUDA::availableGPUMemory();
	bool ok;
	if (handle.d->arr)
		ok = checkCuda(hipFreeArray(handle.d->arr), "freeGPUMemory array");
	else
		ok = checkCuda(hipFree(handle.d->ptr.ptr), "freeGPUMemory");
	size_t free2 = astraCUDA::availableGPUMemory();

	ASTRA_DEBUG("Freeing memory. (Pre: %lu, post: %lu)", free, free2);

	return ok;
}

bool copyToGPUMemory(const float *src, MemHandle3D dst, const SSubDimensions3D &pos)
{
	ASTRA_DEBUG("Copying %d x %d x %d to GPU", pos.subnx, pos.subny, pos.subnz);
	ASTRA_DEBUG("Offset %d,%d,%d", pos.subx, pos.suby, pos.subz);
	assert(!dst.d->arr);
	hipPitchedPtr s;
	s.ptr = (void*)src; // const cast away
	s.pitch = pos.pitch * sizeof(float);
	s.xsize = pos.nx * sizeof(float);
	s.ysize = pos.ny;
	ASTRA_DEBUG("Pitch %d, xsize %d, ysize %d", s.pitch, s.xsize, s.ysize);

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = make_hipPos(pos.subx * sizeof(float), pos.suby, pos.subz);
	p.srcPtr = s;

	p.dstArray = 0;
	p.dstPos = make_hipPos(0, 0, 0);
	p.dstPtr = dst.d->ptr;

	p.extent = make_hipExtent(pos.subnx * sizeof(float), pos.subny, pos.subnz);

	p.kind = hipMemcpyHostToDevice;

	return checkCuda(hipMemcpy3D(&p), "copyToGPUMemory");
}


bool copyFromGPUMemory(float *dst, MemHandle3D src, const SSubDimensions3D &pos)
{
	ASTRA_DEBUG("Copying %d x %d x %d from GPU", pos.subnx, pos.subny, pos.subnz);
	ASTRA_DEBUG("Offset %d,%d,%d", pos.subx, pos.suby, pos.subz);
	assert(!src.d->arr);
	hipPitchedPtr d;
	d.ptr = (void*)dst;
	d.pitch = pos.pitch * sizeof(float);
	d.xsize = pos.nx * sizeof(float);
	d.ysize = pos.ny;
	ASTRA_DEBUG("Pitch %d, xsize %d, ysize %d", d.pitch, d.xsize, d.ysize);

	hipMemcpy3DParms p;
	p.srcPos = make_hipPos(0, 0, 0);

	p.dstArray = 0;
	p.dstPos = make_hipPos(pos.subx * sizeof(float), pos.suby, pos.subz);
	p.dstPtr = d;

        if (src.d->ptr.ptr) {
            p.srcArray = 0;
            p.srcPtr = src.d->ptr;
            p.extent = make_hipExtent(pos.subnx * sizeof(float), pos.subny, pos.subnz);
        } else {
            p.srcArray = src.d->arr;
            p.srcPtr.ptr = 0;
            p.extent = make_hipExtent(pos.subnx, pos.subny, pos.subnz);
        }

	p.kind = hipMemcpyDeviceToHost;

	return checkCuda(hipMemcpy3D(&p), "copyFromGPUMemory");
}


bool FP(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, int iDetectorSuperSampling, astra::Cuda3DProjectionKernel projKernel)
{
	assert(!projData.d->arr);
	assert(!volData.d->arr);
	SDimensions3D dims;
	SProjectorParams3D params;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

#if 1
	params.iRaysPerDetDim = iDetectorSuperSampling;
	if (iDetectorSuperSampling == 0)
		return false;
#else
	astra::Cuda3DProjectionKernel projKernel = astra::ker3d_default;
#endif


	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (pParProjs) {
#if 0
		for (int i = 0; i < dims.iProjAngles; ++i) {
			ASTRA_DEBUG("Vec: %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f %6.3f\n",
			    pParProjs[i].fRayX, pParProjs[i].fRayY, pParProjs[i].fRayZ,
			    pParProjs[i].fDetSX, pParProjs[i].fDetSY, pParProjs[i].fDetSZ,
			    pParProjs[i].fDetUX, pParProjs[i].fDetUY, pParProjs[i].fDetUZ,
			    pParProjs[i].fDetVX, pParProjs[i].fDetVY, pParProjs[i].fDetVZ);
		}
#endif

		switch (projKernel) {
		case astra::ker3d_default:
			ok &= Par3DFP(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
			break;
		case astra::ker3d_sum_square_weights:
			ok &= Par3DFP_SumSqW(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
			break;
		default:
			ok = false;
		}
	} else {
		switch (projKernel) {
		case astra::ker3d_default:
			ok &= ConeFP(volData.d->ptr, projData.d->ptr, dims, pConeProjs, params);
			break;
		default:
			ok = false;
		}
	}

	delete[] pParProjs;
	delete[] pConeProjs;

	return ok;
}

bool BP(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, int iVoxelSuperSampling)
{
	assert(!volData.d->arr);
	SDimensions3D dims;
	SProjectorParams3D params;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

#if 1
	params.iRaysPerVoxelDim = iVoxelSuperSampling;
#endif

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	params.bFDKWeighting = false;

	if (pParProjs) {
		if (projData.d->arr)
			ok &= Par3DBP_Array(volData.d->ptr, projData.d->arr, dims, pParProjs, params);
		else
			ok &= Par3DBP(volData.d->ptr, projData.d->ptr, dims, pParProjs, params);
	} else {
		if (projData.d->arr)
			ok &= ConeBP_Array(volData.d->ptr, projData.d->arr, dims, pConeProjs, params);
		else
			ok &= ConeBP(volData.d->ptr, projData.d->ptr, dims, pConeProjs, params);
	}

	delete[] pParProjs;
	delete[] pConeProjs;

	return ok;

}

bool FDK(const astra::CProjectionGeometry3D* pProjGeom, MemHandle3D projData, const astra::CVolumeGeometry3D* pVolGeom, MemHandle3D volData, bool bShortScan, const float *pfFilter, float fOutputScale)
{
	assert(!projData.d->arr);
	assert(!volData.d->arr);
	SDimensions3D dims;
	SProjectorParams3D params;
	params.fOutputScale = fOutputScale;

	bool ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);
	if (!ok)
		return false;

	SPar3DProjection* pParProjs;
	SConeProjection* pConeProjs;

	ok = convertAstraGeometry(pVolGeom, pProjGeom,
	                          pParProjs, pConeProjs,
	                          params);

	if (!ok || !pConeProjs) {
		delete[] pParProjs;
		delete[] pConeProjs;
		return false;
	}

	ok &= FDK(volData.d->ptr, projData.d->ptr, pConeProjs, dims, params, bShortScan, pfFilter);

	delete[] pParProjs;
	delete[] pConeProjs;

	return ok;



}

_AstraExport MemHandle3D wrapHandle(float *D_ptr, unsigned int x, unsigned int y, unsigned int z, unsigned int pitch)
{
	hipPitchedPtr ptr;
	ptr.ptr = D_ptr;
	ptr.xsize = sizeof(float) * x;
	ptr.pitch = sizeof(float) * pitch;
	ptr.ysize = y;

	SMemHandle3D_internal h;
	h.ptr = ptr;
	h.arr = 0;

	MemHandle3D hnd;
	hnd.d = std::make_shared<SMemHandle3D_internal>();
	*hnd.d = h;

	return hnd;
}

MemHandle3D createProjectionArrayHandle(const float *ptr, unsigned int x, unsigned int y, unsigned int z)
{
	SDimensions3D dims;
	dims.iProjU = x;
	dims.iProjAngles = y;
	dims.iProjV = z;

	hipArray* cuArray = allocateProjectionArray(dims);
	transferHostProjectionsToArray(ptr, cuArray, dims);

	SMemHandle3D_internal h;
	h.arr = cuArray;
	h.ptr.ptr = 0;

	MemHandle3D hnd;
	hnd.d = std::make_shared<SMemHandle3D_internal>();
	*hnd.d = h;

	return hnd;
}

bool copyIntoArray(MemHandle3D handle, MemHandle3D subdata, const SSubDimensions3D &pos)
{
	assert(handle.d->arr);
	assert(!handle.d->ptr.ptr);
	assert(!subdata.d->arr);
	assert(subdata.d->ptr.ptr);

	ASTRA_DEBUG("Copying %d x %d x %d into GPU array", pos.subnx, pos.subny, pos.subnz);
	ASTRA_DEBUG("Offset %d,%d,%d", pos.subx, pos.suby, pos.subz);
	ASTRA_DEBUG("Pitch %d, xsize %d, ysize %d", subdata.d->ptr.pitch, subdata.d->ptr.xsize, subdata.d->ptr.ysize);

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = make_hipPos(0, 0, 0);
	p.srcPtr = subdata.d->ptr;

	p.dstArray = handle.d->arr;
	p.dstPos = make_hipPos(pos.subx, pos.suby, pos.subz);
	p.dstPtr.ptr = 0;

	p.extent = make_hipExtent(pos.subnx, pos.subny, pos.subnz);

	p.kind = hipMemcpyHostToDevice;

	return checkCuda(hipMemcpy3D(&p), "copyIntoArray");

}



}
