#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "util.h"
#include "par_fp.h"
#include "fan_fp.h"
#include "par_bp.h"
#include "fan_bp.h"
#include "arith.h"
#include "astra.h"

#include "fft.h"

#include <fstream>
#include <hip/hip_runtime.h>

#include "../../include/astra/VolumeGeometry2D.h"
#include "../../include/astra/ParallelProjectionGeometry2D.h"
#include "../../include/astra/FanFlatProjectionGeometry2D.h"
#include "../../include/astra/FanFlatVecProjectionGeometry2D.h"

#include "../../include/astra/Logging.h"

// For fan beam FBP weighting
#include "../3d/fdk.h"

using namespace astraCUDA;
using namespace std;


namespace astra {

enum CUDAProjectionType {
	PROJ_PARALLEL,
	PROJ_FAN
};


class AstraFBP_internal {
public:
	SDimensions dims;
	float* angles;
	float* TOffsets;
	astraCUDA::SFanProjection* fanProjections;

	float fOriginSourceDistance;
	float fOriginDetectorDistance;

	float fPixelSize;

	bool bFanBeam;
	bool bShortScan;

	bool initialized;
	bool setStartReconstruction;

	float* D_sinoData;
	unsigned int sinoPitch;

	float* D_volumeData;
	unsigned int volumePitch;

	hipfftComplex * m_pDevFilter;
};

AstraFBP::AstraFBP()
{
	pData = new AstraFBP_internal();

	pData->angles = 0;
	pData->fanProjections = 0;
	pData->TOffsets = 0;
	pData->D_sinoData = 0;
	pData->D_volumeData = 0;

	pData->dims.iVolWidth = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.fDetScale = 1.0f;
	pData->dims.iRaysPerDet = 1;
	pData->dims.iRaysPerPixelDim = 1;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->m_pDevFilter = NULL;
}

AstraFBP::~AstraFBP()
{
	delete[] pData->angles;
	pData->angles = 0;

	delete[] pData->TOffsets;
	pData->TOffsets = 0;

	delete[] pData->fanProjections;
	pData->fanProjections = 0;

	hipFree(pData->D_sinoData);
	pData->D_sinoData = 0;

	hipFree(pData->D_volumeData);
	pData->D_volumeData = 0;

	if(pData->m_pDevFilter != NULL)
	{
		freeComplexOnDevice(pData->m_pDevFilter);
		pData->m_pDevFilter = NULL;
	}

	delete pData;
	pData = 0;
}

bool AstraFBP::setReconstructionGeometry(unsigned int iVolWidth,
                                          unsigned int iVolHeight,
                                          float fPixelSize)
{
	if (pData->initialized)
		return false;

	pData->dims.iVolWidth = iVolWidth;
	pData->dims.iVolHeight = iVolHeight;

	pData->fPixelSize = fPixelSize;

	return (iVolWidth > 0 && iVolHeight > 0 && fPixelSize > 0.0f);
}

bool AstraFBP::setProjectionGeometry(unsigned int iProjAngles,
                                      unsigned int iProjDets,
                                      const float* pfAngles,
                                      float fDetSize)
{
	if (pData->initialized)
		return false;

	pData->dims.iProjAngles = iProjAngles;
	pData->dims.iProjDets = iProjDets;
	pData->dims.fDetScale = fDetSize / pData->fPixelSize;

	if (iProjAngles == 0 || iProjDets == 0 || pfAngles == 0)
		return false;

	pData->angles = new float[iProjAngles];
	memcpy(pData->angles, pfAngles, iProjAngles * sizeof(pfAngles[0]));

	pData->bFanBeam = false;

	return true;
}

bool AstraFBP::setFanGeometry(unsigned int iProjAngles,
                              unsigned int iProjDets,
                              const astraCUDA::SFanProjection *fanProjs,
                              const float* pfAngles,
                              float fOriginSourceDistance,
                              float fOriginDetectorDistance,
                              float fDetSize,
                              bool bShortScan)
{
	// Slightly abusing setProjectionGeometry for this...
	if (!setProjectionGeometry(iProjAngles, iProjDets, pfAngles, fDetSize))
		return false;

	pData->fOriginSourceDistance = fOriginSourceDistance;
	pData->fOriginDetectorDistance = fOriginDetectorDistance;

	pData->fanProjections = new astraCUDA::SFanProjection[iProjAngles];
	memcpy(pData->fanProjections, fanProjs, iProjAngles * sizeof(fanProjs[0]));

	pData->bFanBeam = true;
	pData->bShortScan = bShortScan;

	return true;
}


bool AstraFBP::setPixelSuperSampling(unsigned int iPixelSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iPixelSuperSampling == 0)
		return false;

	pData->dims.iRaysPerPixelDim = iPixelSuperSampling;

	return true;
}


bool AstraFBP::setTOffsets(const float* pfTOffsets)
{
	if (pData->initialized)
		return false;

	if (pfTOffsets == 0)
		return false;

	pData->TOffsets = new float[pData->dims.iProjAngles];
	memcpy(pData->TOffsets, pfTOffsets, pData->dims.iProjAngles * sizeof(pfTOffsets[0]));

	return true;
}

bool AstraFBP::init(int iGPUIndex)
{
	if (pData->initialized)
	{
		return false;
	}

	if (pData->dims.iProjAngles == 0 || pData->dims.iVolWidth == 0)
	{
		return false;
	}

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
		{
			return false;
		}
	}

	bool ok = allocateVolumeData(pData->D_volumeData, pData->volumePitch, pData->dims);
	if (!ok)
	{
		return false;
	}

	ok = allocateProjectionData(pData->D_sinoData, pData->sinoPitch, pData->dims);
	if (!ok)
	{
		hipFree(pData->D_volumeData);
		pData->D_volumeData = 0;
		return false;
	}

	pData->initialized = true;

	return true;
}

bool AstraFBP::setSinogram(const float* pfSinogram,
                            unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copySinogramToDevice(pfSinogram, iSinogramPitch,
	                               pData->dims,
	                               pData->D_sinoData, pData->sinoPitch);
	if (!ok)
		return false;

	// rescale sinogram to adjust for pixel size
	processSino<opMul>(pData->D_sinoData,
	                       1.0f/(pData->fPixelSize*pData->fPixelSize),
	                       pData->sinoPitch, pData->dims);

	pData->setStartReconstruction = false;

	return true;
}

static int calcNextPowerOfTwo(int _iValue)
{
	int iOutput = 1;

	while(iOutput < _iValue)
	{
		iOutput *= 2;
	}

	return iOutput;
}

bool AstraFBP::run()
{
	if (!pData->initialized)
	{
		return false;
	}

	zeroVolumeData(pData->D_volumeData, pData->volumePitch, pData->dims);

	bool ok = false;

	if (pData->bFanBeam) {
		// Call FDK_PreWeight to handle fan beam geometry. We treat
		// this as a cone beam setup of a single slice:

		// TODO: TOffsets affects this preweighting...

		// We create a fake hipPitchedPtr
		hipPitchedPtr tmp;
		tmp.ptr = pData->D_sinoData;
		tmp.pitch = pData->sinoPitch * sizeof(float);
		tmp.xsize = pData->dims.iProjDets;
		tmp.ysize = pData->dims.iProjAngles;
		// and a fake Dimensions3D
		astraCUDA3d::SDimensions3D dims3d;
		dims3d.iVolX = pData->dims.iVolWidth;
		dims3d.iVolY = pData->dims.iVolHeight;
		dims3d.iVolZ = 1;
		dims3d.iProjAngles = pData->dims.iProjAngles;
		dims3d.iProjU = pData->dims.iProjDets;
		dims3d.iProjV = 1;

		astraCUDA3d::FDK_PreWeight(tmp, pData->fOriginSourceDistance,
		              pData->fOriginDetectorDistance, 0.0f,
		              pData->dims.fDetScale, 1.0f, // TODO: Are these correct?
		              pData->bShortScan, dims3d, pData->angles);
	}

	if (pData->m_pDevFilter) {

		int iFFTRealDetCount = calcNextPowerOfTwo(2 * pData->dims.iProjDets);
		int iFFTFourDetCount = calcFFTFourSize(iFFTRealDetCount);

		hipfftComplex * pDevComplexSinogram = NULL;

		allocateComplexOnDevice(pData->dims.iProjAngles, iFFTFourDetCount, &pDevComplexSinogram);

		runCudaFFT(pData->dims.iProjAngles, pData->D_sinoData, pData->sinoPitch, pData->dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount, pDevComplexSinogram);

		applyFilter(pData->dims.iProjAngles, iFFTFourDetCount, pDevComplexSinogram, pData->m_pDevFilter);

		runCudaIFFT(pData->dims.iProjAngles, pDevComplexSinogram, pData->D_sinoData, pData->sinoPitch, pData->dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount);

		freeComplexOnDevice(pDevComplexSinogram);

	}

	float fOutputScale = (M_PI / 2.0f) / (float)pData->dims.iProjAngles;

	if (pData->bFanBeam) {
		ok = FanBP_FBPWeighted(pData->D_volumeData, pData->volumePitch, pData->D_sinoData, pData->sinoPitch, pData->dims, pData->fanProjections, fOutputScale);

	} else {
		ok = BP(pData->D_volumeData, pData->volumePitch, pData->D_sinoData, pData->sinoPitch, pData->dims, pData->angles, pData->TOffsets, fOutputScale);
	}
	if(!ok)
	{
		return false;
	}

	return true;
}

bool AstraFBP::getReconstruction(float* pfReconstruction, unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, iReconstructionPitch,
	                               pData->dims,
	                               pData->D_volumeData, pData->volumePitch);
	if (!ok)
		return false;

	return true;
}

int AstraFBP::calcFourierFilterSize(int _iDetectorCount)
{
	int iFFTRealDetCount = calcNextPowerOfTwo(2 * _iDetectorCount);
	int iFreqBinCount = calcFFTFourSize(iFFTRealDetCount);

	// CHECKME: Matlab makes this at least 64. Do we also need to?
	return iFreqBinCount;
}

bool AstraFBP::setFilter(E_FBPFILTER _eFilter, const float * _pfHostFilter /* = NULL */, int _iFilterWidth /* = 0 */, float _fD /* = 1.0f */, float _fFilterParameter /* = -1.0f */)
{
	if(pData->m_pDevFilter != 0)
	{
		freeComplexOnDevice(pData->m_pDevFilter);
		pData->m_pDevFilter = 0;
	}

	if (_eFilter == FILTER_NONE)
		return true; // leave pData->m_pDevFilter set to 0


	int iFFTRealDetCount = calcNextPowerOfTwo(2 * pData->dims.iProjDets);
	int iFreqBinCount = calcFFTFourSize(iFFTRealDetCount);

	hipfftComplex * pHostFilter = new hipfftComplex[pData->dims.iProjAngles * iFreqBinCount];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * pData->dims.iProjAngles * iFreqBinCount);

	allocateComplexOnDevice(pData->dims.iProjAngles, iFreqBinCount, &(pData->m_pDevFilter));

	switch(_eFilter)
	{
		case FILTER_NONE:
			// handled above
			break;
		case FILTER_RAMLAK:
		case FILTER_SHEPPLOGAN:
		case FILTER_COSINE:
		case FILTER_HAMMING:
		case FILTER_HANN:
		case FILTER_TUKEY:
		case FILTER_LANCZOS:
		case FILTER_TRIANGULAR:
		case FILTER_GAUSSIAN:
		case FILTER_BARTLETTHANN:
		case FILTER_BLACKMAN:
		case FILTER_NUTTALL:
		case FILTER_BLACKMANHARRIS:
		case FILTER_BLACKMANNUTTALL:
		case FILTER_FLATTOP:
		case FILTER_PARZEN:
		{
			genFilter(_eFilter, _fD, pData->dims.iProjAngles, pHostFilter, iFFTRealDetCount, iFreqBinCount, _fFilterParameter);
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);

			break;
		}
		case FILTER_PROJECTION:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				float fValue = _pfHostFilter[iFreqBinIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);
			break;
		}
		case FILTER_SINOGRAM:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iFreqBinIndex + iProjectionIndex * _iFilterWidth];

					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);
			break;
		}
		case FILTER_RPROJECTION:
		{
			int iProjectionCount = pData->dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float * pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;

			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;
				float fValue = _pfHostFilter[iDetectorIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, pData->m_pDevFilter);

			hipFree(pfDevRealFilter);

			break;
		}
		case FILTER_RSINOGRAM:
		{
			int iProjectionCount = pData->dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float* pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;

			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iDetectorIndex + iProjectionIndex * _iFilterWidth];
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, pData->m_pDevFilter);

			hipFree(pfDevRealFilter);

			break;
		}
		default:
		{
			ASTRA_ERROR("AstraFBP::setFilter: Unknown filter type requested");
			delete [] pHostFilter;
			return false;
		}
	}

	delete [] pHostFilter;

	return true;
}

BPalgo::BPalgo()
{

}

BPalgo::~BPalgo()
{

}

bool BPalgo::init()
{
	return true;
}

bool BPalgo::iterate(unsigned int)
{
	// TODO: This zeroVolume makes an earlier memcpy of D_volumeData redundant
	zeroVolumeData(D_volumeData, volumePitch, dims);
	callBP(D_volumeData, volumePitch, D_sinoData, sinoPitch, 1.0f);
	return true;
}

float BPalgo::computeDiffNorm()
{
	float *D_projData;
	unsigned int projPitch;

	allocateProjectionData(D_projData, projPitch, dims);

	duplicateProjectionData(D_projData, D_sinoData, sinoPitch, dims);
	callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	hipFree(D_projData);

	return sqrt(s);
}


bool astraCudaFP(const float* pfVolume, float* pfSinogram,
                 unsigned int iVolWidth, unsigned int iVolHeight,
                 unsigned int iProjAngles, unsigned int iProjDets,
                 const float *pfAngles, const float *pfOffsets,
                 float fDetSize, unsigned int iDetSuperSampling,
                 float fOutputScale, int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pfAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;
	dims.fDetScale = fDetSize;

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);
	ok = FP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pfAngles, pfOffsets, fOutputScale);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);
	return true;
}

bool astraCudaFanFP(const float* pfVolume, float* pfSinogram,
                    unsigned int iVolWidth, unsigned int iVolHeight,
                    unsigned int iProjAngles, unsigned int iProjDets,
                    const SFanProjection *pAngles,
                    unsigned int iDetSuperSampling, float fOutputScale,
                    int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;
	dims.fDetScale = 1.0f; // TODO?

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);

	ok = FanFP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pAngles, fOutputScale);

	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);

	return true;

}


bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CParallelProjectionGeometry2D* pProjGeom,
                          float*& detectorOffsets, float*& projectionAngles,
                          float& detSize, float& outputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	const float EPS = 0.00001f;

	int nth = pProjGeom->getProjectionAngleCount();

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;


	// Scale volume pixels to 1x1
	detSize = pProjGeom->getDetectorWidth() / pVolGeom->getPixelLengthX();

	// Copy angles
	float *angles = new float[nth];
	for (int i = 0; i < nth; ++i)
		angles[i] = pProjGeom->getProjectionAngles()[i];
	projectionAngles = angles;

	// Check if we need to translate
	bool offCenter = false;
	if (abs(pVolGeom->getWindowMinX() + pVolGeom->getWindowMaxX()) > EPS ||
	    abs(pVolGeom->getWindowMinY() + pVolGeom->getWindowMaxY()) > EPS)
	{
		offCenter = true;
	}

	// If there are existing detector offsets, or if we need to translate,
	// we need to return offsets
	if (pProjGeom->getExtraDetectorOffset() || offCenter)
	{
		float* offset = new float[nth];

		if (pProjGeom->getExtraDetectorOffset()) {
			for (int i = 0; i < nth; ++i)
				offset[i] = pProjGeom->getExtraDetectorOffset()[i];
		} else {
			for (int i = 0; i < nth; ++i)
				offset[i] = 0.0f;
		}

		if (offCenter) {
			float dx = (pVolGeom->getWindowMinX() + pVolGeom->getWindowMaxX()) / 2;
			float dy = (pVolGeom->getWindowMinY() + pVolGeom->getWindowMaxY()) / 2;

			// CHECKME: Is d in pixels or in units?

			for (int i = 0; i < nth; ++i) {
				float d = dx * cos(angles[i]) + dy * sin(angles[i]);
				offset[i] += d;
			}
		}

		// CHECKME: Order of scaling and translation

		// Scale volume pixels to 1x1
		for (int i = 0; i < nth; ++i) {
			//offset[i] /= pVolGeom->getPixelLengthX();
			//offset[i] *= detSize;
		}


		detectorOffsets = offset;
	} else {
		detectorOffsets = 0;
	}

	outputScale = pVolGeom->getPixelLengthX();
	outputScale *= outputScale;

	return true;
}

static void convertAstraGeometry_internal(const CVolumeGeometry2D* pVolGeom,
                          unsigned int iProjectionAngleCount,
                          astraCUDA::SFanProjection*& pProjs,
                          float& outputScale)
{
	// Translate
	float dx = (pVolGeom->getWindowMinX() + pVolGeom->getWindowMaxX()) / 2;
	float dy = (pVolGeom->getWindowMinY() + pVolGeom->getWindowMaxY()) / 2;

	for (int i = 0; i < iProjectionAngleCount; ++i) {
		pProjs[i].fSrcX -= dx;
		pProjs[i].fSrcY -= dy;
		pProjs[i].fDetSX -= dx;
		pProjs[i].fDetSY -= dy;
	}

	// CHECKME: Order of scaling and translation

	// Scale
	float factor = 1.0f / pVolGeom->getPixelLengthX();
	for (int i = 0; i < iProjectionAngleCount; ++i) {
		pProjs[i].fSrcX *= factor;
		pProjs[i].fSrcY *= factor;
		pProjs[i].fDetSX *= factor;
		pProjs[i].fDetSY *= factor;
		pProjs[i].fDetUX *= factor;
		pProjs[i].fDetUY *= factor;

	}

	// CHECKME: Check factor
	outputScale = pVolGeom->getPixelLengthX();
//	outputScale *= outputScale;
}


bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CFanFlatProjectionGeometry2D* pProjGeom,
                          astraCUDA::SFanProjection*& pProjs,
                          float& outputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	const float EPS = 0.00001f;

	int nth = pProjGeom->getProjectionAngleCount();

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;

	// TODO: Deprecate this.
//	if (pProjGeom->getExtraDetectorOffset())
//		return false;


	float fOriginSourceDistance = pProjGeom->getOriginSourceDistance();
	float fOriginDetectorDistance = pProjGeom->getOriginDetectorDistance();
	float fDetSize = pProjGeom->getDetectorWidth();
	const float *pfAngles = pProjGeom->getProjectionAngles();

	pProjs = new SFanProjection[nth];

	float fSrcX0 = 0.0f;
	float fSrcY0 = -fOriginSourceDistance;
	float fDetUX0 = fDetSize;
	float fDetUY0 = 0.0f;
	float fDetSX0 = pProjGeom->getDetectorCount() * fDetUX0 / -2.0f;
	float fDetSY0 = fOriginDetectorDistance;

#define ROTATE0(name,i,alpha) do { pProjs[i].f##name##X = f##name##X0 * cos(alpha) - f##name##Y0 * sin(alpha); pProjs[i].f##name##Y = f##name##X0 * sin(alpha) + f##name##Y0 * cos(alpha); } while(0)
	for (int i = 0; i < nth; ++i) {
		ROTATE0(Src, i, pfAngles[i]);
		ROTATE0(DetS, i, pfAngles[i]);
		ROTATE0(DetU, i, pfAngles[i]);
	}

#undef ROTATE0

	convertAstraGeometry_internal(pVolGeom, nth, pProjs, outputScale);

	return true;

}

bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CFanFlatVecProjectionGeometry2D* pProjGeom,
                          astraCUDA::SFanProjection*& pProjs,
                          float& outputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionVectors());

	const float EPS = 0.00001f;

	int nx = pVolGeom->getGridColCount();
	int ny = pVolGeom->getGridRowCount();
	int nth = pProjGeom->getProjectionAngleCount();

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;

	pProjs = new SFanProjection[nth];

	// Copy vectors
	for (int i = 0; i < nth; ++i)
		pProjs[i] = pProjGeom->getProjectionVectors()[i];

	convertAstraGeometry_internal(pVolGeom, nth, pProjs, outputScale);

	return true;
}




}
