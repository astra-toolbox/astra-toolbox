#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/par_fp.h"
#include "astra/cuda/2d/fan_fp.h"
#include "astra/cuda/2d/par_bp.h"
#include "astra/cuda/2d/fan_bp.h"
#include "astra/cuda/2d/arith.h"
#include "astra/cuda/2d/astra.h"
#include "astra/cuda/2d/fft.h"

// For fan beam FBP weighting
#include "astra/cuda/3d/fdk.h"

#include "astra/GeometryUtil2D.h"
#include "astra/VolumeGeometry2D.h"
#include "astra/ParallelProjectionGeometry2D.h"
#include "astra/ParallelVecProjectionGeometry2D.h"
#include "astra/FanFlatProjectionGeometry2D.h"
#include "astra/FanFlatVecProjectionGeometry2D.h"
#include "astra/Logging.h"

#include <cstdio>
#include <cassert>
#include <fstream>

#include <hip/hip_runtime.h>

using namespace astraCUDA;
using namespace std;


namespace astra {

enum CUDAProjectionType {
	PROJ_PARALLEL,
	PROJ_FAN
};


BPalgo::BPalgo()
{

}

BPalgo::~BPalgo()
{

}

bool BPalgo::init()
{
	return true;
}

bool BPalgo::iterate(unsigned int)
{
	// TODO: This zeroVolume makes an earlier memcpy of D_volumeData redundant
	zeroVolumeData(D_volumeData, volumePitch, dims);
	callBP(D_volumeData, volumePitch, D_sinoData, sinoPitch, 1.0f);
	return true;
}

float BPalgo::computeDiffNorm()
{
	float *D_projData;
	unsigned int projPitch;

	allocateProjectionData(D_projData, projPitch, dims);

	duplicateProjectionData(D_projData, D_sinoData, sinoPitch, dims);
	callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	hipFree(D_projData);

	return sqrt(s);
}


bool astraCudaFP(const float* pfVolume, float* pfSinogram,
                 unsigned int iVolWidth, unsigned int iVolHeight,
                 unsigned int iProjAngles, unsigned int iProjDets,
                 const SParProjection *pAngles,
                 unsigned int iDetSuperSampling,
                 float fOutputScale, int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);
	ok = FP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pAngles, fOutputScale);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);
	return true;
}

bool astraCudaFanFP(const float* pfVolume, float* pfSinogram,
                    unsigned int iVolWidth, unsigned int iVolHeight,
                    unsigned int iProjAngles, unsigned int iProjDets,
                    const SFanProjection *pAngles,
                    unsigned int iDetSuperSampling, float fOutputScale,
                    int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);

	ok = FanFP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pAngles, fOutputScale);

	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);

	return true;

}


// adjust pProjs to normalize volume geometry
template<typename ProjectionT>
static bool convertAstraGeometry_internal(const CVolumeGeometry2D* pVolGeom,
                          unsigned int iProjectionAngleCount,
                          ProjectionT*& pProjs,
                          float& fOutputScale)
{
	// TODO: Make EPS relative
	const float EPS = 0.00001f;

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;

	float dx = -(pVolGeom->getWindowMinX() + pVolGeom->getWindowMaxX()) / 2;
	float dy = -(pVolGeom->getWindowMinY() + pVolGeom->getWindowMaxY()) / 2;

	float factor = 1.0f / pVolGeom->getPixelLengthX();

	for (int i = 0; i < iProjectionAngleCount; ++i) {
		// CHECKME: Order of scaling and translation
		pProjs[i].translate(dx, dy);
		pProjs[i].scale(factor);
	}
	// CHECKME: Check factor
	// NB: Only valid for square pixels
	fOutputScale *= pVolGeom->getPixelLengthX();

	return true;
}



bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CParallelProjectionGeometry2D* pProjGeom,
                          SParProjection*& pProjs,
                          float& fOutputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	int nth = pProjGeom->getProjectionAngleCount();

	pProjs = genParProjections(nth,
	                           pProjGeom->getDetectorCount(),
	                           pProjGeom->getDetectorWidth(),
	                           pProjGeom->getProjectionAngles(), 0);

	bool ok;
	fOutputScale = 1.0f;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, fOutputScale);

	if (!ok) {
		delete[] pProjs;
		pProjs = 0;
	}

	return ok;
}

bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CParallelVecProjectionGeometry2D* pProjGeom,
                          SParProjection*& pProjs,
                          float& fOutputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionVectors());

	int nth = pProjGeom->getProjectionAngleCount();

	pProjs = new SParProjection[nth];

	for (int i = 0; i < nth; ++i) {
		pProjs[i] = pProjGeom->getProjectionVectors()[i];
	}

	bool ok;
	fOutputScale = 1.0f;

	ok = convertAstraGeometry_internal(pVolGeom, nth, pProjs, fOutputScale);

	if (!ok) {
		delete[] pProjs;
		pProjs = 0;
	}

	return ok;
}



bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CFanFlatProjectionGeometry2D* pProjGeom,
                          astraCUDA::SFanProjection*& pProjs,
                          float& outputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionAngles());

	// TODO: Make EPS relative
	const float EPS = 0.00001f;

	int nth = pProjGeom->getProjectionAngleCount();

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;

	// TODO: Deprecate this.
//	if (pProjGeom->getExtraDetectorOffset())
//		return false;


	float fOriginSourceDistance = pProjGeom->getOriginSourceDistance();
	float fOriginDetectorDistance = pProjGeom->getOriginDetectorDistance();
	float fDetSize = pProjGeom->getDetectorWidth();
	const float *pfAngles = pProjGeom->getProjectionAngles();

	pProjs = genFanProjections(nth, pProjGeom->getDetectorCount(),
                               fOriginSourceDistance, fOriginDetectorDistance,
	                           fDetSize, pfAngles);

	convertAstraGeometry_internal(pVolGeom, nth, pProjs, outputScale);

	return true;

}

bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CFanFlatVecProjectionGeometry2D* pProjGeom,
                          astraCUDA::SFanProjection*& pProjs,
                          float& outputScale)
{
	assert(pVolGeom);
	assert(pProjGeom);
	assert(pProjGeom->getProjectionVectors());

	// TODO: Make EPS relative
	const float EPS = 0.00001f;

	int nx = pVolGeom->getGridColCount();
	int ny = pVolGeom->getGridRowCount();
	int nth = pProjGeom->getProjectionAngleCount();

	// Check if pixels are square
	if (abs(pVolGeom->getPixelLengthX() - pVolGeom->getPixelLengthY()) > EPS)
		return false;

	pProjs = new SFanProjection[nth];

	// Copy vectors
	for (int i = 0; i < nth; ++i)
		pProjs[i] = pProjGeom->getProjectionVectors()[i];

	convertAstraGeometry_internal(pVolGeom, nth, pProjs, outputScale);

	return true;
}

bool convertAstraGeometry(const CVolumeGeometry2D* pVolGeom,
                          const CProjectionGeometry2D* pProjGeom,
                          astraCUDA::SParProjection*& pParProjs,
                          astraCUDA::SFanProjection*& pFanProjs,
                          float& outputScale)
{
	const CParallelProjectionGeometry2D* parProjGeom = dynamic_cast<const CParallelProjectionGeometry2D*>(pProjGeom);
	const CParallelVecProjectionGeometry2D* parVecProjGeom = dynamic_cast<const CParallelVecProjectionGeometry2D*>(pProjGeom);
	const CFanFlatProjectionGeometry2D* fanProjGeom = dynamic_cast<const CFanFlatProjectionGeometry2D*>(pProjGeom);
	const CFanFlatVecProjectionGeometry2D* fanVecProjGeom = dynamic_cast<const CFanFlatVecProjectionGeometry2D*>(pProjGeom);

	bool ok = false;

	if (parProjGeom) {
		ok = convertAstraGeometry(pVolGeom, parProjGeom, pParProjs, outputScale);
	} else if (parVecProjGeom) {
		ok = convertAstraGeometry(pVolGeom, parVecProjGeom, pParProjs, outputScale);
	} else if (fanProjGeom) {
		ok = convertAstraGeometry(pVolGeom, fanProjGeom, pFanProjs, outputScale);
	} else if (fanVecProjGeom) {
		ok = convertAstraGeometry(pVolGeom, fanVecProjGeom, pFanProjs, outputScale);
	} else {
		ok = false;
	}

	return ok;
}

bool convertAstraGeometry_dims(const CVolumeGeometry2D* pVolGeom,
                               const CProjectionGeometry2D* pProjGeom,
                               SDimensions& dims)
{
	dims.iVolWidth = pVolGeom->getGridColCount();
	dims.iVolHeight = pVolGeom->getGridRowCount();

	dims.iProjAngles = pProjGeom->getProjectionAngleCount();
	dims.iProjDets = pProjGeom->getDetectorCount();

	dims.iRaysPerDet = 1;
	dims.iRaysPerPixelDim = 1;

	return true;
}



}

namespace astraCUDA {


_AstraExport std::string getCudaDeviceString(int device)
{
	char buf[1024];
	hipError_t err;
	if (device == -1) {
		err = hipGetDevice(&device);
		if (err != hipSuccess) {
			return "Error getting current GPU index";
		}
	}

	hipDeviceProp_t prop;
	err = hipGetDeviceProperties(&prop, device);
	if (err != hipSuccess) {
		snprintf(buf, 1024, "GPU #%d: Invalid device (%d): %s", device, err, hipGetErrorString(err));
		return buf;
	}

	long mem = prop.totalGlobalMem / (1024*1024);
	snprintf(buf, 1024, "GPU #%d: %s, with %ldMB, CUDA compute capability %d.%d", device, prop.name, mem, prop.major, prop.minor);
	return buf;
}

_AstraExport bool setGPUIndex(int iGPUIndex)
{
        if (iGPUIndex != -1) {
                hipError_t err = hipSetDevice(iGPUIndex);

                // Ignore errors caused by calling hipSetDevice multiple times
                if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
                        return false;
        }

        return true;
}

_AstraExport size_t availableGPUMemory()
{
    size_t free, total;
    hipError_t err = hipMemGetInfo(&free, &total);
    if (err != hipSuccess)
        return 0;
    return free;
}




}
