#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "sirt.h"
#include "util.h"
#include "arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

namespace astraCUDA {

SIRT::SIRT() : ReconAlgo()
{
	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;
	D_pixelWeight = 0;

	D_minMaskData = 0;
	D_maxMaskData = 0;

	fRelaxation = 1.0f;

	freeMinMaxMasks = false;
}


SIRT::~SIRT()
{
	reset();
}

void SIRT::reset()
{
	hipFree(D_projData);
	hipFree(D_tmpData);
	hipFree(D_lineWeight);
	hipFree(D_pixelWeight);
	if (freeMinMaxMasks) {
		hipFree(D_minMaskData);
		hipFree(D_maxMaskData);
	}

	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;
	D_pixelWeight = 0;

	freeMinMaxMasks = false;
	D_minMaskData = 0;
	D_maxMaskData = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	fRelaxation = 1.0f;

	ReconAlgo::reset();
}

bool SIRT::init()
{

	// TODO allocate primal and dual variables, declare them in header (and their pitch)

	allocateVolumeData(D_pixelWeight, pixelPitch, dims);
	zeroVolumeData(D_pixelWeight, pixelPitch, dims);

	allocateVolumeData(D_tmpData, tmpPitch, dims);
	zeroVolumeData(D_tmpData, tmpPitch, dims);

	allocateProjectionData(D_projData, projPitch, dims);
	zeroProjectionData(D_projData, projPitch, dims);

	allocateProjectionData(D_lineWeight, linePitch, dims);
	zeroProjectionData(D_lineWeight, linePitch, dims);

	// We can't precompute lineWeights and pixelWeights when using a mask
	if (!useVolumeMask && !useSinogramMask)
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool SIRT::precomputeWeights()
{
	zeroProjectionData(D_lineWeight, linePitch, dims);
	if (useVolumeMask) {
		callFP(D_maskData, maskPitch, D_lineWeight, linePitch, 1.0f);
	} else {
		processVol<opSet>(D_tmpData, 1.0f, tmpPitch, dims);
		callFP(D_tmpData, tmpPitch, D_lineWeight, linePitch, 1.0f);
	}
	processSino<opInvert>(D_lineWeight, linePitch, dims);

	if (useSinogramMask) {
		// scale line weights with sinogram mask to zero out masked sinogram pixels
		processSino<opMul>(D_lineWeight, D_smaskData, linePitch, dims);
	}


	zeroVolumeData(D_pixelWeight, pixelPitch, dims);
	if (useSinogramMask) {
		callBP(D_pixelWeight, pixelPitch, D_smaskData, smaskPitch, 1.0f);
	} else {
		processSino<opSet>(D_projData, 1.0f, projPitch, dims);
		callBP(D_pixelWeight, pixelPitch, D_projData, projPitch, 1.0f);
	}
	processVol<opInvert>(D_pixelWeight, pixelPitch, dims);

	if (useVolumeMask) {
		// scale pixel weights with mask to zero out masked pixels
		processVol<opMul>(D_pixelWeight, D_maskData, pixelPitch, dims);
	}

	// Also fold the relaxation factor into pixel weights
	processVol<opMul>(D_pixelWeight, fRelaxation, pixelPitch, dims);

	return true;
}

bool SIRT::doSlabCorrections()
{
	// This function compensates for effectively infinitely large slab-like
	// objects of finite thickness 1.

	// Each ray through the object has an intersection of length d/cos(alpha).
	// The length of the ray actually intersecting the reconstruction volume is
	// given by D_lineWeight. By dividing by 1/cos(alpha) and multiplying by the
	// lineweights, we correct for this missing attenuation outside of the
	// reconstruction volume, assuming the object is homogeneous.

	// This effectively scales the output values by assuming the thickness d
	// is 1 unit.


	// This function in its current implementation only works if there are no masks.
	// In this case, init() will also have already called precomputeWeights(),
	// so we can use D_lineWeight.
	if (useVolumeMask || useSinogramMask)
		return false;

	// multiply by line weights
	processSino<opDiv>(D_sinoData, D_lineWeight, projPitch, dims);

	SDimensions subdims = dims;
	subdims.iProjAngles = 1;

	// divide by 1/cos(angle)
	// ...but limit the correction to -80/+80 degrees.
	float bound = cosf(1.3963f);
	float* t = (float*)D_sinoData;
	for (int i = 0; i < dims.iProjAngles; ++i) {
		float f = fabs(cosf(angles[i]));

		if (f < bound)
			f = bound;

		processSino<opMul>(t, f, sinoPitch, subdims);
		t += sinoPitch;
	}

	return true;
}


bool SIRT::setMinMaxMasks(float* D_minMaskData_, float* D_maxMaskData_,
	                      unsigned int iPitch)
{
	D_minMaskData = D_minMaskData_;
	D_maxMaskData = D_maxMaskData_;
	minMaskPitch = iPitch;
	maxMaskPitch = iPitch;

	freeMinMaxMasks = false;
	return true;
}

bool SIRT::uploadMinMaxMasks(const float* pfMinMaskData, const float* pfMaxMaskData,
	                         unsigned int iPitch)
{
	freeMinMaxMasks = true;
	bool ok = true;
	if (pfMinMaskData) {
		allocateVolumeData(D_minMaskData, minMaskPitch, dims);
		ok = copyVolumeToDevice(pfMinMaskData, iPitch,
		                        dims,
		                        D_minMaskData, minMaskPitch);
	}
	if (!ok)
		return false;

	if (pfMaxMaskData) {
		allocateVolumeData(D_maxMaskData, maxMaskPitch, dims);
		ok = copyVolumeToDevice(pfMaxMaskData, iPitch,
		                        dims,
		                        D_maxMaskData, maxMaskPitch);
	}
	if (!ok)
		return false;

	return true;
}

bool SIRT::iterate(unsigned int iterations)
{
	shouldAbort = false;

	if (useVolumeMask || useSinogramMask)
		precomputeWeights();

	// iteration
	for (unsigned int iter = 0; iter < iterations && !shouldAbort; ++iter) {

		// copy sinogram to projection data
		duplicateProjectionData(D_projData, D_sinoData, projPitch, dims);

		// do FP, subtracting projection from sinogram
		if (useVolumeMask) {
				duplicateVolumeData(D_tmpData, D_volumeData, volumePitch, dims);
				processVol<opMul>(D_tmpData, D_maskData, tmpPitch, dims);
				callFP(D_tmpData, tmpPitch, D_projData, projPitch, -1.0f);
		} else {
				callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);
		}

		processSino<opMul>(D_projData, D_lineWeight, projPitch, dims);

		zeroVolumeData(D_tmpData, tmpPitch, dims);

		callBP(D_tmpData, tmpPitch, D_projData, projPitch, 1.0f);

		// pixel weights also contain the volume mask and relaxation factor
		processVol<opAddMul>(D_volumeData, D_pixelWeight, D_tmpData, volumePitch, dims);

		if (useMinConstraint)
			processVol<opClampMin>(D_volumeData, fMinConstraint, volumePitch, dims);
		if (useMaxConstraint)
			processVol<opClampMax>(D_volumeData, fMaxConstraint, volumePitch, dims);
		if (D_minMaskData)
			processVol<opClampMinMask>(D_volumeData, D_minMaskData, volumePitch, dims);
		if (D_maxMaskData)
			processVol<opClampMaxMask>(D_volumeData, D_maxMaskData, volumePitch, dims);
	}

	return true;
}

float SIRT::computeDiffNorm()
{
	// copy sinogram to projection data
	duplicateProjectionData(D_projData, D_sinoData, projPitch, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_tmpData, D_volumeData, volumePitch, dims);
			processVol<opMul>(D_tmpData, D_maskData, tmpPitch, dims);
			callFP(D_tmpData, tmpPitch, D_projData, projPitch, -1.0f);
	} else {
			callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);
	}


	// compute norm of D_projData

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	return sqrt(s);
}


bool doSIRT(float* D_volumeData, unsigned int volumePitch,
            float* D_sinoData, unsigned int sinoPitch,
            float* D_maskData, unsigned int maskPitch,
            const SDimensions& dims, const float* angles,
            const float* TOffsets, unsigned int iterations)
{
	SIRT sirt;
	bool ok = true;

	ok &= sirt.setGeometry(dims, angles);
	if (D_maskData)
		ok &= sirt.enableVolumeMask();
	if (TOffsets)
		ok &= sirt.setTOffsets(TOffsets);

	if (!ok)
		return false;

	ok = sirt.init();
	if (!ok)
		return false;

	if (D_maskData)
		ok &= sirt.setVolumeMask(D_maskData, maskPitch);

	ok &= sirt.setBuffers(D_volumeData, volumePitch, D_sinoData, sinoPitch);
	if (!ok)
		return false;

	ok = sirt.iterate(iterations);

	return ok;
}

}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_sinoData;

	SDimensions dims;
	dims.iVolWidth = 1024;
	dims.iVolHeight = 1024;
	dims.iProjAngles = 512;
	dims.iProjDets = 1536;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;
	unsigned int volumePitch, sinoPitch;

	allocateVolume(D_volumeData, dims.iVolWidth, dims.iVolHeight, volumePitch);
	zeroVolume(D_volumeData, volumePitch, dims.iVolWidth, dims.iVolHeight);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_sinoData, dims.iProjDets, dims.iProjAngles, sinoPitch);
	zeroVolume(D_sinoData, sinoPitch, dims.iProjDets, dims.iProjAngles);
	printf("pitch: %u\n", sinoPitch);

	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_sinoData, sinoPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	SIRT sirt;

	sirt.setGeometry(dims, angle);
	sirt.init();

	sirt.setBuffers(D_volumeData, volumePitch, D_sinoData, sinoPitch);

	sirt.iterate(25);


	delete[] angle;

	copyVolumeFromDevice(img, dims.iVolWidth, dims, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif

