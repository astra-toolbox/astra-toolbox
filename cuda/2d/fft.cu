#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2021, imec Vision Lab, University of Antwerp
           2014-2021, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/fft.h"
#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"
#include "astra/Fourier.h"

#include <iostream>
#include <fstream>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


using namespace astra;

// TODO: evaluate what we want to do in these situations:

#define CHECK_ERROR(errorMessage) do {                                     \
  hipError_t err = hipDeviceSynchronize();                               \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error %s : %s",                                    \
              errorMessage,hipGetErrorString( err));                      \
      exit(EXIT_FAILURE);                                                  \
  } } while (0)

#define SAFE_CALL( call) do {                                              \
  hipError_t err = call;                                                    \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error: %s ",                                       \
              hipGetErrorString( err));                                   \
      exit(EXIT_FAILURE);                                                  \
  }                                                                        \
  err = hipDeviceSynchronize();                                           \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error: %s : ",                                     \
              hipGetErrorString( err));                                   \
      exit(EXIT_FAILURE);                                                  \
  } } while (0)


namespace astraCUDA {

__global__ static void applyFilter_kernel(int _iProjectionCount,
                                          int _iFreqBinCount,
                                          hipfftComplex * _pSinogram,
                                          hipfftComplex * _pFilter)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iFreqBinCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	float fA = _pSinogram[iIndex].x;
	float fB = _pSinogram[iIndex].y;
	float fC = _pFilter[iIndex].x;
	float fD = _pFilter[iIndex].y;

	_pSinogram[iIndex].x = fA * fC - fB * fD;
	_pSinogram[iIndex].y = fA * fD + fC * fB;
}

__global__ static void rescaleInverseFourier_kernel(int _iProjectionCount,
                                                    int _iDetectorCount,
                                                    float* _pfInFourierOutput)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iDetectorCount;
	int iDetectorIndex = iIndex % _iDetectorCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	_pfInFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex] /= (float)_iDetectorCount;
}

static void rescaleInverseFourier(int _iProjectionCount, int _iDetectorCount,
                                  float * _pfInFourierOutput)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iDetectorCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	rescaleInverseFourier_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                            _iDetectorCount,
	                                                            _pfInFourierOutput);
	CHECK_ERROR("rescaleInverseFourier_kernel failed");
}

void applyFilter(int _iProjectionCount, int _iFreqBinCount,
                 hipfftComplex * _pSinogram, hipfftComplex * _pFilter)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iFreqBinCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	applyFilter_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                  _iFreqBinCount,
	                                                  _pSinogram, _pFilter);
	CHECK_ERROR("applyFilter_kernel failed");
}

static bool invokeCudaFFT(int _iProjectionCount, int _iDetectorCount,
                          const float * _pfDevSource,
                          hipfftComplex * _pDevTargetComplex)
{
	hipfftHandle plan;
	hipfftResult result;

	result = hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_R2C, _iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d r2c fft");
		return false;
	}

	result = hipfftExecR2C(plan, (hipfftReal *)_pfDevSource, _pDevTargetComplex);
	hipfftDestroy(plan);

	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d r2c fft");
		return false;
	}

	return true;
}

static bool invokeCudaIFFT(int _iProjectionCount, int _iDetectorCount,
                           const hipfftComplex * _pDevSourceComplex,
                           float * _pfDevTarget)
{
	hipfftHandle plan;
	hipfftResult result;

	result = hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_C2R, _iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d c2r fft");
		return false;
	}

	// todo: why do we have to get rid of the const qualifier?
	result = hipfftExecC2R(plan, (hipfftComplex *)_pDevSourceComplex,
	                      (hipfftReal *)_pfDevTarget);
	hipfftDestroy(plan);

	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d c2r fft");
		return false;
	}

	return true;
}

bool allocateComplexOnDevice(int _iProjectionCount, int _iDetectorCount,
                             hipfftComplex ** _ppDevComplex)
{
	size_t bufferSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	SAFE_CALL(hipMalloc((void **)_ppDevComplex, bufferSize));
	return true;
}

bool freeComplexOnDevice(hipfftComplex * _pDevComplex)
{
	SAFE_CALL(hipFree(_pDevComplex));
	return true;
}

bool uploadComplexArrayToDevice(int _iProjectionCount, int _iDetectorCount,
                                hipfftComplex * _pHostComplexSource,
                                hipfftComplex * _pDevComplexTarget)
{
	size_t memSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	SAFE_CALL(hipMemcpy(_pDevComplexTarget, _pHostComplexSource, memSize, hipMemcpyHostToDevice));

	return true;
}

bool runCudaFFT(int _iProjectionCount, const float * _pfDevRealSource,
                int _iSourcePitch, int _iProjDets,
                int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount,
                hipfftComplex * _pDevTargetComplex)
{
	float * pfDevRealFFTSource = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iFFTRealDetectorCount;

	SAFE_CALL(hipMalloc((void **)&pfDevRealFFTSource, bufferMemSize));
	SAFE_CALL(hipMemset(pfDevRealFFTSource, 0, bufferMemSize));

	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		const float * pfSourceLocation = _pfDevRealSource + iProjectionIndex * _iSourcePitch;
		float * pfTargetLocation = pfDevRealFFTSource + iProjectionIndex * _iFFTRealDetectorCount;

		SAFE_CALL(hipMemcpy(pfTargetLocation, pfSourceLocation, sizeof(float) * _iProjDets, hipMemcpyDeviceToDevice));
	}

	bool bResult = invokeCudaFFT(_iProjectionCount, _iFFTRealDetectorCount,
	                             pfDevRealFFTSource, _pDevTargetComplex);
	if(!bResult)
	{
		return false;
	}

	SAFE_CALL(hipFree(pfDevRealFFTSource));

	return true;
}

bool runCudaIFFT(int _iProjectionCount, const hipfftComplex* _pDevSourceComplex,
                 float * _pfRealTarget,
                 int _iTargetPitch, int _iProjDets,
                 int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	float * pfDevRealFFTTarget = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iFFTRealDetectorCount;

	SAFE_CALL(hipMalloc((void **)&pfDevRealFFTTarget, bufferMemSize));

	bool bResult = invokeCudaIFFT(_iProjectionCount, _iFFTRealDetectorCount,
	                              _pDevSourceComplex, pfDevRealFFTTarget);
	if(!bResult)
	{
		return false;
	}

	rescaleInverseFourier(_iProjectionCount, _iFFTRealDetectorCount,
	                      pfDevRealFFTTarget);

	SAFE_CALL(hipMemset(_pfRealTarget, 0, sizeof(float) * _iProjectionCount * _iTargetPitch));

	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		const float * pfSourceLocation = pfDevRealFFTTarget + iProjectionIndex * _iFFTRealDetectorCount;
		float* pfTargetLocation = _pfRealTarget + iProjectionIndex * _iTargetPitch;

		SAFE_CALL(hipMemcpy(pfTargetLocation, pfSourceLocation, sizeof(float) * _iProjDets, hipMemcpyDeviceToDevice));
	}

	SAFE_CALL(hipFree(pfDevRealFFTTarget));

	return true;
}

void genIdenFilter(int _iProjectionCount, hipfftComplex * _pFilter,
                   int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = 1.0f;
			_pFilter[iIndex].y = 0.0f;
		}
	}
}

void genCuFFTFilter(const SFilterConfig &_cfg, int _iProjectionCount,
               hipfftComplex * _pFilter, int _iFFTRealDetectorCount,
               int _iFFTFourierDetectorCount)
{
	float * pfFilt = astra::genFilter(_cfg,
	                                  _iFFTRealDetectorCount,
	                                  _iFFTFourierDetectorCount);

	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fFilterValue = pfFilt[iDetectorIndex];

		for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = fFilterValue;
			_pFilter[iIndex].y = 0.0f;
		}
	}

	delete[] pfFilt;
}


}
