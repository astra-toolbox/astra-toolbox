#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include "fft.h"
#include "util.h"

#include <hipfft/hipfft.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>

#include "../../include/astra/Logging.h"
#include "../../include/astra/Fourier.h"

using namespace astra;

// TODO: evaluate what we want to do in these situations:

#define CHECK_ERROR(errorMessage) do {                                     \
  hipError_t err = hipDeviceSynchronize();                               \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error %s : %s",                                    \
              errorMessage,hipGetErrorString( err));                      \
      exit(EXIT_FAILURE);                                                  \
  } } while (0)

#define SAFE_CALL( call) do {                                              \
  hipError_t err = call;                                                    \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error: %s ",                                       \
              hipGetErrorString( err));                                   \
      exit(EXIT_FAILURE);                                                  \
  }                                                                        \
  err = hipDeviceSynchronize();                                           \
  if( hipSuccess != err) {                                                \
      ASTRA_ERROR("Cuda error: %s : ",                                     \
              hipGetErrorString( err));                                   \
      exit(EXIT_FAILURE);                                                  \
  } } while (0)


__global__ static void applyFilter_kernel(int _iProjectionCount,
                                          int _iFreqBinCount,
                                          hipfftComplex * _pSinogram,
                                          hipfftComplex * _pFilter)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iFreqBinCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	float fA = _pSinogram[iIndex].x;
	float fB = _pSinogram[iIndex].y;
	float fC = _pFilter[iIndex].x;
	float fD = _pFilter[iIndex].y;

	_pSinogram[iIndex].x = fA * fC - fB * fD;
	_pSinogram[iIndex].y = fA * fD + fC * fB;
}

__global__ static void rescaleInverseFourier_kernel(int _iProjectionCount,
                                                    int _iDetectorCount,
                                                    float* _pfInFourierOutput)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iDetectorCount;
	int iDetectorIndex = iIndex % _iDetectorCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	_pfInFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex] /= (float)_iDetectorCount;
}

static void rescaleInverseFourier(int _iProjectionCount, int _iDetectorCount,
                                  float * _pfInFourierOutput)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iDetectorCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	rescaleInverseFourier_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                            _iDetectorCount,
	                                                            _pfInFourierOutput);
	CHECK_ERROR("rescaleInverseFourier_kernel failed");
}

void applyFilter(int _iProjectionCount, int _iFreqBinCount,
                 hipfftComplex * _pSinogram, hipfftComplex * _pFilter)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iFreqBinCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	applyFilter_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                  _iFreqBinCount,
	                                                  _pSinogram, _pFilter);
	CHECK_ERROR("applyFilter_kernel failed");
}

static bool invokeCudaFFT(int _iProjectionCount, int _iDetectorCount,
                          const float * _pfDevSource,
                          hipfftComplex * _pDevTargetComplex)
{
	hipfftHandle plan;
	hipfftResult result;

	result = hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_R2C, _iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d r2c fft");
		return false;
	}

	result = hipfftExecR2C(plan, (hipfftReal *)_pfDevSource, _pDevTargetComplex);
	hipfftDestroy(plan);

	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d r2c fft");
		return false;
	}

	return true;
}

static bool invokeCudaIFFT(int _iProjectionCount, int _iDetectorCount,
                           const hipfftComplex * _pDevSourceComplex,
                           float * _pfDevTarget)
{
	hipfftHandle plan;
	hipfftResult result;

	result = hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_C2R, _iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d c2r fft");
		return false;
	}

	// todo: why do we have to get rid of the const qualifier?
	result = hipfftExecC2R(plan, (hipfftComplex *)_pDevSourceComplex,
	                      (hipfftReal *)_pfDevTarget);
	hipfftDestroy(plan);

	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d c2r fft");
		return false;
	}

	return true;
}

bool allocateComplexOnDevice(int _iProjectionCount, int _iDetectorCount,
                             hipfftComplex ** _ppDevComplex)
{
	size_t bufferSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	SAFE_CALL(hipMalloc((void **)_ppDevComplex, bufferSize));
	return true;
}

bool freeComplexOnDevice(hipfftComplex * _pDevComplex)
{
	SAFE_CALL(hipFree(_pDevComplex));
	return true;
}

bool uploadComplexArrayToDevice(int _iProjectionCount, int _iDetectorCount,
                                hipfftComplex * _pHostComplexSource,
                                hipfftComplex * _pDevComplexTarget)
{
	size_t memSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	SAFE_CALL(hipMemcpy(_pDevComplexTarget, _pHostComplexSource, memSize, hipMemcpyHostToDevice));

	return true;
}

bool runCudaFFT(int _iProjectionCount, const float * _pfDevRealSource,
                int _iSourcePitch, int _iProjDets,
                int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount,
                hipfftComplex * _pDevTargetComplex)
{
	float * pfDevRealFFTSource = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iFFTRealDetectorCount;

	SAFE_CALL(hipMalloc((void **)&pfDevRealFFTSource, bufferMemSize));
	SAFE_CALL(hipMemset(pfDevRealFFTSource, 0, bufferMemSize));

	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		const float * pfSourceLocation = _pfDevRealSource + iProjectionIndex * _iSourcePitch;
		float * pfTargetLocation = pfDevRealFFTSource + iProjectionIndex * _iFFTRealDetectorCount;

		SAFE_CALL(hipMemcpy(pfTargetLocation, pfSourceLocation, sizeof(float) * _iProjDets, hipMemcpyDeviceToDevice));
	}

	bool bResult = invokeCudaFFT(_iProjectionCount, _iFFTRealDetectorCount,
	                             pfDevRealFFTSource, _pDevTargetComplex);
	if(!bResult)
	{
		return false;
	}

	SAFE_CALL(hipFree(pfDevRealFFTSource));

	return true;
}

bool runCudaIFFT(int _iProjectionCount, const hipfftComplex* _pDevSourceComplex,
                 float * _pfRealTarget,
                 int _iTargetPitch, int _iProjDets,
                 int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	float * pfDevRealFFTTarget = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iFFTRealDetectorCount;

	SAFE_CALL(hipMalloc((void **)&pfDevRealFFTTarget, bufferMemSize));

	bool bResult = invokeCudaIFFT(_iProjectionCount, _iFFTRealDetectorCount,
	                              _pDevSourceComplex, pfDevRealFFTTarget);
	if(!bResult)
	{
		return false;
	}

	rescaleInverseFourier(_iProjectionCount, _iFFTRealDetectorCount,
	                      pfDevRealFFTTarget);

	SAFE_CALL(hipMemset(_pfRealTarget, 0, sizeof(float) * _iProjectionCount * _iTargetPitch));

	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		const float * pfSourceLocation = pfDevRealFFTTarget + iProjectionIndex * _iFFTRealDetectorCount;
		float* pfTargetLocation = _pfRealTarget + iProjectionIndex * _iTargetPitch;

		SAFE_CALL(hipMemcpy(pfTargetLocation, pfSourceLocation, sizeof(float) * _iProjDets, hipMemcpyDeviceToDevice));
	}

	SAFE_CALL(hipFree(pfDevRealFFTTarget));

	return true;
}


// Because the input is real, the Fourier transform is symmetric.
// CUFFT only outputs the first half (ignoring the redundant second half),
// and expects the same as input for the IFFT.
int calcFFTFourSize(int _iFFTRealSize)
{
	int iFFTFourSize = _iFFTRealSize / 2 + 1;

	return iFFTFourSize;
}

void genIdenFilter(int _iProjectionCount, hipfftComplex * _pFilter,
                   int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = 1.0f;
			_pFilter[iIndex].y = 0.0f;
		}
	}
}

void genFilter(E_FBPFILTER _eFilter, float _fD, int _iProjectionCount,
               hipfftComplex * _pFilter, int _iFFTRealDetectorCount,
               int _iFFTFourierDetectorCount, float _fParameter /* = -1.0f */)
{
	float * pfFilt = new float[_iFFTFourierDetectorCount];
	float * pfW = new float[_iFFTFourierDetectorCount];

	// We cache one Fourier transform for repeated FBP's of the same size
	static float *pfData = 0;
	static int iFilterCacheSize = 0;

	if (!pfData || iFilterCacheSize != _iFFTRealDetectorCount) {
		// Compute filter in spatial domain

		delete[] pfData;
		pfData = new float[2*_iFFTRealDetectorCount];
		int *ip = new int[int(2+sqrt(_iFFTRealDetectorCount)+1)];
		ip[0] = 0;
		float32 *w = new float32[_iFFTRealDetectorCount/2];

		for (int i = 0; i < _iFFTRealDetectorCount; ++i) {
			pfData[2*i+1] = 0.0f;

			if (i & 1) {
				int j = i;
				if (2*j > _iFFTRealDetectorCount)
					j = _iFFTRealDetectorCount - j;
				float f = M_PI * j;
				pfData[2*i] = -1 / (f*f);
			} else {
				pfData[2*i] = 0.0f;
			}
		}

		pfData[0] = 0.25f;

		cdft(2*_iFFTRealDetectorCount, -1, pfData, ip, w);
		delete[] ip;
		delete[] w;

		iFilterCacheSize = _iFFTRealDetectorCount;
	}

	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fRelIndex = (float)iDetectorIndex / (float)_iFFTRealDetectorCount;

		pfFilt[iDetectorIndex] = 2.0f * pfData[2*iDetectorIndex];
		pfW[iDetectorIndex] = M_PI * 2.0f * fRelIndex;
	}

	switch(_eFilter)
	{
		case FILTER_RAMLAK:
		{
			// do nothing
			break;
		}
		case FILTER_SHEPPLOGAN:
		{
			// filt(2:end) = filt(2:end) .* (sin(w(2:end)/(2*d))./(w(2:end)/(2*d)))
			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				pfFilt[iDetectorIndex] = pfFilt[iDetectorIndex] * (sinf(pfW[iDetectorIndex] / 2.0f / _fD) / (pfW[iDetectorIndex] / 2.0f / _fD));
			}
			break;
		}
		case FILTER_COSINE:
		{
			// filt(2:end) = filt(2:end) .* cos(w(2:end)/(2*d))
			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				pfFilt[iDetectorIndex] = pfFilt[iDetectorIndex] * cosf(pfW[iDetectorIndex] / 2.0f / _fD);
			}
			break;
		}
		case FILTER_HAMMING:
		{
			// filt(2:end) = filt(2:end) .* (.54 + .46 * cos(w(2:end)/d))
			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				pfFilt[iDetectorIndex] = pfFilt[iDetectorIndex] * ( 0.54f + 0.46f * cosf(pfW[iDetectorIndex] / _fD));
			}
			break;
		}
		case FILTER_HANN:
		{
			// filt(2:end) = filt(2:end) .*(1+cos(w(2:end)./d)) / 2
			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				pfFilt[iDetectorIndex] = pfFilt[iDetectorIndex] * (1.0f + cosf(pfW[iDetectorIndex] / _fD)) / 2.0f;
			}
			break;
		}
		case FILTER_TUKEY:
		{
			float fAlpha = _fParameter;
			if(_fParameter < 0.0f) fAlpha = 0.5f;
			float fN = (float)_iFFTFourierDetectorCount;
			float fHalfN = fN / 2.0f;
			float fEnumTerm = fAlpha * fHalfN;
			float fDenom = (1.0f - fAlpha) * fHalfN;
			float fBlockStart = fHalfN - fEnumTerm;
			float fBlockEnd = fHalfN + fEnumTerm;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fAbsSmallN = fabs((float)iDetectorIndex);
				float fStoredValue = 0.0f;

				if((fBlockStart <= fAbsSmallN) && (fAbsSmallN <= fBlockEnd))
				{
					fStoredValue = 1.0f;
				}
				else
				{
					float fEnum = fAbsSmallN - fEnumTerm;
					float fCosInput = M_PI * fEnum / fDenom;
					fStoredValue = 0.5f * (1.0f + cosf(fCosInput));
				}

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_LANCZOS:
		{
			float fDenum = (float)(_iFFTFourierDetectorCount - 1);

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fX = 2.0f * fSmallN / fDenum - 1.0f;
				float fSinInput = M_PI * fX;
				float fStoredValue = 0.0f;

				if(fabsf(fSinInput) > 0.001f)
				{
					fStoredValue = sin(fSinInput)/fSinInput;
				}
				else
				{
					fStoredValue = 1.0f;
				}

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_TRIANGULAR:
		{
			float fNMinusOne = (float)(_iFFTFourierDetectorCount - 1);

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fAbsInput = fSmallN - fNMinusOne / 2.0f;
				float fParenInput = fNMinusOne / 2.0f - fabsf(fAbsInput);
				float fStoredValue = 2.0f / fNMinusOne * fParenInput;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_GAUSSIAN:
		{
			float fSigma = _fParameter;
			if(_fParameter < 0.0f) fSigma = 0.4f;
			float fN = (float)_iFFTFourierDetectorCount;
			float fQuotient = (fN - 1.0f) / 2.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fEnum = fSmallN - fQuotient;
				float fDenom = fSigma * fQuotient;
				float fPower = -0.5f * (fEnum / fDenom) * (fEnum / fDenom);
				float fStoredValue = expf(fPower);

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_BARTLETTHANN:
		{
			const float fA0 = 0.62f;
			const float fA1 = 0.48f;
			const float fA2 = 0.38f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount) - 1.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fAbsInput = fSmallN / fNMinusOne - 0.5f;
				float fFirstTerm = fA1 * fabsf(fAbsInput);
				float fCosInput = 2.0f * M_PI * fSmallN / fNMinusOne;
				float fSecondTerm = fA2 * cosf(fCosInput);
				float fStoredValue = fA0 - fFirstTerm - fSecondTerm;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_BLACKMAN:
		{
			float fAlpha = _fParameter;
			if(_fParameter < 0.0f) fAlpha = 0.16f;
			float fA0 = (1.0f - fAlpha) / 2.0f;
			float fA1 = 0.5f;
			float fA2 = fAlpha / 2.0f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount - 1);

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fCosInput1 = 2.0f * M_PI * 0.5f * fSmallN / fNMinusOne;
				float fCosInput2 = 4.0f * M_PI * 0.5f * fSmallN / fNMinusOne;
				float fStoredValue = fA0 - fA1 * cosf(fCosInput1) + fA2 * cosf(fCosInput2);

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_NUTTALL:
		{
			const float fA0 = 0.355768f;
			const float fA1 = 0.487396f;
			const float fA2 = 0.144232f;
			const float fA3 = 0.012604f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount) - 1.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fBaseCosInput = M_PI * fSmallN / fNMinusOne;
				float fFirstTerm = fA1 * cosf(2.0f * fBaseCosInput);
				float fSecondTerm = fA2 * cosf(4.0f * fBaseCosInput);
				float fThirdTerm = fA3 * cosf(6.0f * fBaseCosInput);
				float fStoredValue = fA0 - fFirstTerm + fSecondTerm - fThirdTerm;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_BLACKMANHARRIS:
		{
			const float fA0 = 0.35875f;
			const float fA1 = 0.48829f;
			const float fA2 = 0.14128f;
			const float fA3 = 0.01168f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount) - 1.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fBaseCosInput = M_PI * fSmallN / fNMinusOne;
				float fFirstTerm = fA1 * cosf(2.0f * fBaseCosInput);
				float fSecondTerm = fA2 * cosf(4.0f * fBaseCosInput);
				float fThirdTerm = fA3 * cosf(6.0f * fBaseCosInput);
				float fStoredValue = fA0 - fFirstTerm + fSecondTerm - fThirdTerm;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_BLACKMANNUTTALL:
		{
			const float fA0 = 0.3635819f;
			const float fA1 = 0.4891775f;
			const float fA2 = 0.1365995f;
			const float fA3 = 0.0106411f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount) - 1.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fBaseCosInput = M_PI * fSmallN / fNMinusOne;
				float fFirstTerm = fA1 * cosf(2.0f * fBaseCosInput);
				float fSecondTerm = fA2 * cosf(4.0f * fBaseCosInput);
				float fThirdTerm = fA3 * cosf(6.0f * fBaseCosInput);
				float fStoredValue = fA0 - fFirstTerm + fSecondTerm - fThirdTerm;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_FLATTOP:
		{
			const float fA0 = 1.0f;
			const float fA1 = 1.93f;
			const float fA2 = 1.29f;
			const float fA3 = 0.388f;
			const float fA4 = 0.032f;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount) - 1.0f;

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fBaseCosInput = M_PI * fSmallN / fNMinusOne;
				float fFirstTerm = fA1 * cosf(2.0f * fBaseCosInput);
				float fSecondTerm = fA2 * cosf(4.0f * fBaseCosInput);
				float fThirdTerm = fA3 * cosf(6.0f * fBaseCosInput);
				float fFourthTerm = fA4 * cosf(8.0f * fBaseCosInput);
				float fStoredValue = fA0 - fFirstTerm + fSecondTerm - fThirdTerm + fFourthTerm;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_KAISER:
		{
			float fAlpha = _fParameter;
			if(_fParameter < 0.0f) fAlpha = 3.0f;
			float fPiTimesAlpha = M_PI * fAlpha;
			float fNMinusOne = (float)(_iFFTFourierDetectorCount - 1);
			float fDenom = (float)j0((double)fPiTimesAlpha);

			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fSquareInput = 2.0f * fSmallN / fNMinusOne - 1;
				float fSqrtInput = 1.0f - fSquareInput * fSquareInput;
				float fBesselInput = fPiTimesAlpha * sqrt(fSqrtInput);
				float fEnum = (float)j0((double)fBesselInput);
				float fStoredValue = fEnum / fDenom;

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		case FILTER_PARZEN:
		{
			for(int iDetectorIndex = 1; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
			{
				float fSmallN = (float)iDetectorIndex;
				float fQ = fSmallN / (float)(_iFFTFourierDetectorCount - 1);
				float fStoredValue = 0.0f;

				if(fQ <= 0.5f)
				{
					fStoredValue = 1.0f - 6.0f * fQ * fQ * (1.0f - fQ);
				}
				else
				{
					float fCubedValue = 1.0f - fQ;
					fStoredValue = 2.0f * fCubedValue * fCubedValue * fCubedValue;
				}

				pfFilt[iDetectorIndex] *= fStoredValue;
			}

			break;
		}
		default:
		{
			ASTRA_ERROR("Cannot serve requested filter");
		}
	}

	// filt(w>pi*d) = 0;
	float fPiTimesD = M_PI * _fD;
	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fWValue = pfW[iDetectorIndex];

		if(fWValue > fPiTimesD)
		{
			pfFilt[iDetectorIndex] = 0.0f;
		}
	}

	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fFilterValue = pfFilt[iDetectorIndex];

		for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = fFilterValue;
			_pFilter[iIndex].y = 0.0f;
		}
	}

	delete[] pfFilt;
	delete[] pfW;
}

#ifdef STANDALONE

__global__ static void doubleFourierOutput_kernel(int _iProjectionCount,
                                                  int _iDetectorCount,
                                                  hipfftComplex* _pFourierOutput)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iDetectorCount;
	int iDetectorIndex = iIndex % _iDetectorCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	if(iDetectorIndex <= (_iDetectorCount / 2))
	{
		return;
	}

	int iOtherDetectorIndex = _iDetectorCount - iDetectorIndex;

	_pFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex].x = _pFourierOutput[iProjectionIndex * _iDetectorCount + iOtherDetectorIndex].x;
	_pFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex].y = -_pFourierOutput[iProjectionIndex * _iDetectorCount + iOtherDetectorIndex].y;
}

static void doubleFourierOutput(int _iProjectionCount, int _iDetectorCount,
                                hipfftComplex * _pFourierOutput)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iDetectorCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	doubleFourierOutput_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                          _iDetectorCount,
	                                                          _pFourierOutput);
	CHECK_ERROR("doubleFourierOutput_kernel failed");
}



static void writeToMatlabFile(const char * _fileName, const float * _pfData,
                              int _iRowCount, int _iColumnCount)
{
	std::ofstream out(_fileName);

	for(int iRowIndex = 0; iRowIndex < _iRowCount; iRowIndex++)
	{
		for(int iColumnIndex = 0; iColumnIndex < _iColumnCount; iColumnIndex++)
		{
			out << _pfData[iColumnIndex + iRowIndex * _iColumnCount] << " ";
		}

		out << std::endl;
	}
}

static void convertComplexToRealImg(const hipfftComplex * _pComplex,
                                    int _iElementCount,
                                    float * _pfReal, float * _pfImaginary)
{
	for(int iIndex = 0; iIndex < _iElementCount; iIndex++)
	{
		_pfReal[iIndex] = _pComplex[iIndex].x;
		_pfImaginary[iIndex] = _pComplex[iIndex].y;
	}
}

void testCudaFFT()
{
	const int iProjectionCount = 2;
	const int iDetectorCount = 1024;
	const int iTotalElementCount = iProjectionCount * iDetectorCount;

	float * pfHostProj = new float[iTotalElementCount];
	memset(pfHostProj, 0, sizeof(float) * iTotalElementCount);

	for(int iProjectionIndex = 0; iProjectionIndex < iProjectionCount; iProjectionIndex++)
	{
		for(int iDetectorIndex = 0; iDetectorIndex < iDetectorCount; iDetectorIndex++)
		{
//			int

//			pfHostProj[iIndex] = (float)rand() / (float)RAND_MAX;
		}
	}

	writeToMatlabFile("proj.mat", pfHostProj, iProjectionCount, iDetectorCount);

	float * pfDevProj = NULL;
	SAFE_CALL(hipMalloc((void **)&pfDevProj, sizeof(float) * iTotalElementCount));
	SAFE_CALL(hipMemcpy(pfDevProj, pfHostProj, sizeof(float) * iTotalElementCount, hipMemcpyHostToDevice));

	hipfftComplex * pDevFourProj = NULL;
	SAFE_CALL(hipMalloc((void **)&pDevFourProj, sizeof(hipfftComplex) * iTotalElementCount));

	hipfftHandle plan;
	hipfftResult result;

	result = hipfftPlan1d(&plan, iDetectorCount, HIPFFT_R2C, iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d r2c fft");
	}

	result = hipfftExecR2C(plan, pfDevProj, pDevFourProj);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d r2c fft");
	}

	hipfftDestroy(plan);

	doubleFourierOutput(iProjectionCount, iDetectorCount, pDevFourProj);

	hipfftComplex * pHostFourProj = new hipfftComplex[iTotalElementCount];
	SAFE_CALL(hipMemcpy(pHostFourProj, pDevFourProj, sizeof(hipfftComplex) * iTotalElementCount, hipMemcpyDeviceToHost));

	float * pfHostFourProjReal = new float[iTotalElementCount];
	float * pfHostFourProjImaginary = new float[iTotalElementCount];

	convertComplexToRealImg(pHostFourProj, iTotalElementCount, pfHostFourProjReal, pfHostFourProjImaginary);

	writeToMatlabFile("proj_four_real.mat", pfHostFourProjReal, iProjectionCount, iDetectorCount);
	writeToMatlabFile("proj_four_imaginary.mat", pfHostFourProjImaginary, iProjectionCount, iDetectorCount);

	float * pfDevInFourProj = NULL;
	SAFE_CALL(hipMalloc((void **)&pfDevInFourProj, sizeof(float) * iTotalElementCount));

	result = hipfftPlan1d(&plan, iDetectorCount, HIPFFT_C2R, iProjectionCount);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to plan 1d c2r fft");
	}

	result = hipfftExecC2R(plan, pDevFourProj, pfDevInFourProj);
	if(result != HIPFFT_SUCCESS)
	{
		ASTRA_ERROR("Failed to exec 1d c2r fft");
	}

	hipfftDestroy(plan);

	rescaleInverseFourier(iProjectionCount, iDetectorCount, pfDevInFourProj);

	float * pfHostInFourProj = new float[iTotalElementCount];
	SAFE_CALL(hipMemcpy(pfHostInFourProj, pfDevInFourProj, sizeof(float) * iTotalElementCount, hipMemcpyDeviceToHost));

	writeToMatlabFile("in_four.mat", pfHostInFourProj, iProjectionCount, iDetectorCount);

	SAFE_CALL(hipFree(pDevFourProj));
	SAFE_CALL(hipFree(pfDevProj));

	delete [] pfHostInFourProj;
	delete [] pfHostFourProjReal;
	delete [] pfHostFourProjImaginary;
	delete [] pfHostProj;
	delete [] pHostFourProj;
}

void downloadDebugFilterComplex(float * _pfHostSinogram, int _iProjectionCount,
                                int _iDetectorCount,
                                hipfftComplex * _pDevFilter,
                                int _iFilterDetCount)
{
	hipfftComplex * pHostFilter = NULL;
	size_t complMemSize = sizeof(hipfftComplex) * _iFilterDetCount * _iProjectionCount;
	pHostFilter = (hipfftComplex *)malloc(complMemSize);
	SAFE_CALL(hipMemcpy(pHostFilter, _pDevFilter, complMemSize, hipMemcpyDeviceToHost));

	for(int iTargetProjIndex = 0; iTargetProjIndex < _iProjectionCount; iTargetProjIndex++)
	{
		for(int iTargetDetIndex = 0; iTargetDetIndex < min(_iDetectorCount, _iFilterDetCount); iTargetDetIndex++)
		{
			hipfftComplex source = pHostFilter[iTargetDetIndex + iTargetProjIndex * _iFilterDetCount];
			float fReadValue = sqrtf(source.x * source.x + source.y * source.y);
			_pfHostSinogram[iTargetDetIndex + iTargetProjIndex * _iDetectorCount] = fReadValue;
		}
	}

	free(pHostFilter);
}

void downloadDebugFilterReal(float * _pfHostSinogram, int _iProjectionCount,
                             int _iDetectorCount, float * _pfDevFilter,
                             int _iFilterDetCount)
{
	float * pfHostFilter = NULL;
	size_t memSize = sizeof(float) * _iFilterDetCount * _iProjectionCount;
	pfHostFilter = (float *)malloc(memSize);
	SAFE_CALL(hipMemcpy(pfHostFilter, _pfDevFilter, memSize, hipMemcpyDeviceToHost));

	for(int iTargetProjIndex = 0; iTargetProjIndex < _iProjectionCount; iTargetProjIndex++)
	{
		for(int iTargetDetIndex = 0; iTargetDetIndex < min(_iDetectorCount, _iFilterDetCount); iTargetDetIndex++)
		{
			float fSource = pfHostFilter[iTargetDetIndex + iTargetProjIndex * _iFilterDetCount];
			_pfHostSinogram[iTargetDetIndex + iTargetProjIndex * _iDetectorCount] = fSource;
		}
	}

	free(pfHostFilter);
}

#endif
