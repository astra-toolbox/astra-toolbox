#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/fft.h"
#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"
#include "astra/Fourier.h"

#include <iostream>
#include <fstream>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


using namespace astra;

namespace astraCUDA {

bool checkCufft(hipfftResult err, const char *msg)
{
	if (err != HIPFFT_SUCCESS) {
		ASTRA_ERROR("%s: CUFFT error %d.", msg, err);
		return false;
	} else {
		return true;
	}
}

__global__ static void applyFilter_kernel(int _iProjectionCount,
                                          int _iFreqBinCount,
                                          hipfftComplex * _pSinogram,
                                          hipfftComplex * _pFilter)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iFreqBinCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	float fA = _pSinogram[iIndex].x;
	float fB = _pSinogram[iIndex].y;
	float fC = _pFilter[iIndex].x;
	float fD = _pFilter[iIndex].y;

	_pSinogram[iIndex].x = fA * fC - fB * fD;
	_pSinogram[iIndex].y = fA * fD + fC * fB;
}

__global__ static void rescaleInverseFourier_kernel(int _iProjectionCount,
                                                    int _iDetectorCount,
                                                    float* _pfInFourierOutput)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iDetectorCount;
	int iDetectorIndex = iIndex % _iDetectorCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	_pfInFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex] /= (float)_iDetectorCount;
}

void rescaleInverseFourier(int _iProjectionCount, int _iDetectorCount,
                           float * _pfInFourierOutput)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iDetectorCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	rescaleInverseFourier_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                            _iDetectorCount,
	                                                            _pfInFourierOutput);

	checkCuda(hipDeviceSynchronize(), "rescaleInverseFourier");
}

void applyFilter(int _iProjectionCount, int _iFreqBinCount,
                 hipfftComplex * _pSinogram, hipfftComplex * _pFilter)
{
	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iFreqBinCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	applyFilter_kernel<<< iBlockCount, iBlockSize >>>(_iProjectionCount,
	                                                  _iFreqBinCount,
	                                                  _pSinogram, _pFilter);

	checkCuda(hipDeviceSynchronize(), "applyFilter");
}

static bool invokeCudaFFT(int _iProjectionCount, int _iDetectorCount,
                          const float * _pfDevSource,
                          hipfftComplex * _pDevTargetComplex)
{
	hipfftHandle plan;

	if (!checkCufft(hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_R2C, _iProjectionCount), "invokeCudaFFT plan")) {
		return false;
	}

	if (!checkCufft(hipfftExecR2C(plan, (hipfftReal *)_pfDevSource, _pDevTargetComplex), "invokeCudaFFT exec")) {
		hipfftDestroy(plan);
		return false;
	}

	if (!checkCuda(hipDeviceSynchronize(), "invokeCudaFFT sync")) {
		hipfftDestroy(plan);
		return false;
	}

	hipfftDestroy(plan);
	return true;
}

static bool invokeCudaIFFT(int _iProjectionCount, int _iDetectorCount,
                           const hipfftComplex * _pDevSourceComplex,
                           float * _pfDevTarget)
{
	hipfftHandle plan;

	if (!checkCufft(hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_C2R, _iProjectionCount), "invokeCudaIFFT plan")) {
		return false;
	}

	// Getting rid of the const qualifier is due to cufft API issue?
	if (!checkCufft(hipfftExecC2R(plan, (hipfftComplex *)_pDevSourceComplex,
	                      (hipfftReal *)_pfDevTarget), "invokeCudaIFFT exec"))
	{
		hipfftDestroy(plan);
		return false;
	}

	if (!checkCuda(hipDeviceSynchronize(), "invokeCudaIFFT sync")) {
		hipfftDestroy(plan);
		return false;
	}

	hipfftDestroy(plan);
	return true;
}

bool allocateComplexOnDevice(int _iProjectionCount, int _iDetectorCount,
                             hipfftComplex ** _ppDevComplex)
{
	size_t bufferSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	return checkCuda(hipMalloc((void **)_ppDevComplex, bufferSize), "fft allocateComplexOnDevice");
}

bool freeComplexOnDevice(hipfftComplex * _pDevComplex)
{
	return checkCuda(hipFree(_pDevComplex), "fft freeComplexOnDevice");
}

bool uploadComplexArrayToDevice(int _iProjectionCount, int _iDetectorCount,
                                hipfftComplex * _pHostComplexSource,
                                hipfftComplex * _pDevComplexTarget)
{
	size_t memSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	return checkCuda(hipMemcpy(_pDevComplexTarget, _pHostComplexSource, memSize, hipMemcpyHostToDevice), "fft uploadComplexArrayToDevice");
}

bool runCudaFFT(int _iProjectionCount,
                const float * D_pfSource, int _iSourcePitch,
                int _iProjDets, int _iPaddedSize,
                hipfftComplex * D_pcTarget)
{
	float * D_pfPaddedSource = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iPaddedSize;

	if (!checkCuda(hipMalloc((void **)&D_pfPaddedSource, bufferMemSize), "runCudaFFT malloc"))
		return false;
	if (!checkCuda(hipMemset(D_pfPaddedSource, 0, bufferMemSize), "runCudaFFT memset")) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	// pitched memcpy 2D to handle both source pitch and target padding
	if (!checkCuda(hipMemcpy2D(D_pfPaddedSource, _iPaddedSize*sizeof(float), D_pfSource, _iSourcePitch*sizeof(float), _iProjDets*sizeof(float), _iProjectionCount, hipMemcpyDeviceToDevice), "runCudaFFT memcpy")) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	bool bResult = invokeCudaFFT(_iProjectionCount, _iPaddedSize,
	                             D_pfPaddedSource, D_pcTarget);
	if(!bResult)
		return false;

	hipFree(D_pfPaddedSource);

	return true;
}

bool runCudaIFFT(int _iProjectionCount, const hipfftComplex *D_pcSource,
                 float * D_pfTarget, int _iTargetPitch,
                 int _iProjDets, int _iPaddedSize)
{
	float * D_pfPaddedTarget = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iPaddedSize;

	if (!checkCuda(hipMalloc((void **)&D_pfPaddedTarget, bufferMemSize), "runCudaIFFT malloc"))
		return false;

	bool bResult = invokeCudaIFFT(_iProjectionCount, _iPaddedSize,
	                             D_pcSource, D_pfPaddedTarget);
	if(!bResult)
	{
		return false;
	}

	rescaleInverseFourier(_iProjectionCount, _iPaddedSize,
	                      D_pfPaddedTarget);

	if (!checkCuda(hipMemset(D_pfTarget, 0, sizeof(float) * _iProjectionCount * _iTargetPitch), "runCudaIFFT memset")) {
		hipFree(D_pfPaddedTarget);
		return false;
	}

	// pitched memcpy 2D to handle both source padding and target pitch
	if (!checkCuda(hipMemcpy2D(D_pfTarget, _iTargetPitch*sizeof(float), D_pfPaddedTarget, _iPaddedSize*sizeof(float), _iProjDets*sizeof(float), _iProjectionCount, hipMemcpyDeviceToDevice), "runCudaIFFT memcpy")) {
		hipFree(D_pfPaddedTarget);
		return false;
	}


	hipFree(D_pfPaddedTarget);

	return true;
}

void genIdenFilter(int _iProjectionCount, hipfftComplex * _pFilter,
                   int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = 1.0f;
			_pFilter[iIndex].y = 0.0f;
		}
	}
}

void genCuFFTFilter(const SFilterConfig &_cfg, int _iProjectionCount,
               hipfftComplex * _pFilter, int _iFFTRealDetectorCount,
               int _iFFTFourierDetectorCount)
{
	float * pfFilt = astra::genFilter(_cfg,
	                                  _iFFTRealDetectorCount,
	                                  _iFFTFourierDetectorCount);

	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fFilterValue = pfFilt[iDetectorIndex];

		for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = fFilterValue;
			_pFilter[iIndex].y = 0.0f;
		}
	}

	delete[] pfFilt;
}


}
