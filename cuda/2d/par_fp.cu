#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>
#include <cmath>

typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_volumeTexture;


namespace astraCUDA {

static const unsigned g_MaxAngles = 2560;
__constant__ float gC_angle[g_MaxAngles];
__constant__ float gC_angle_offset[g_MaxAngles];
__constant__ float gC_angle_detsize[g_MaxAngles];


// optimization parameters
static const unsigned int g_anglesPerBlock = 16;
static const unsigned int g_detBlockSize = 32;
static const unsigned int g_blockSlices = 64;

// fixed point scaling factor
#define fPREC_FACTOR 16.0f
#define iPREC_FACTOR 16


static bool bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	gT_volumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_volumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_volumeTexture.filterMode = hipFilterModeLinear;
	gT_volumeTexture.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
//	hipBindTexture2D(0, gT_volumeTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(gT_volumeTexture, dataArray, channelDesc);

	// TODO: error value?

	return true;
}


// projection for angles that are roughly horizontal
// (detector roughly vertical)
__global__ void FPhorizontal_simple(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;

	if (angle >= endAngle)
		return;

	const float theta = gC_angle[angle];
	const float cos_theta = __cosf(theta);
	const float sin_theta = __sinf(theta);

	// compute start detector for this block/angle:
	const int detRegion = blockIdx.y;

	const int detector = detRegion * g_detBlockSize + relDet;

	// Now project the part of the ray to angle,detector through
	// slices startSlice to startSlice+g_blockSlices-1

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	const float fDetStep = -gC_angle_detsize[angle] / sin_theta;
	float fSliceStep = cos_theta / sin_theta;
	float fDistCorr;
	if (sin_theta > 0.0f)
		fDistCorr = outputScale / sin_theta;
	else
		fDistCorr = -outputScale / sin_theta;

	float fVal = 0.0f;
	// project detector on slice
	float fP = (detector - 0.5f*dims.iProjDets + 0.5f - gC_angle_offset[angle]) * fDetStep + (startSlice - 0.5f*dims.iVolWidth + 0.5f) * fSliceStep + 0.5f*dims.iVolHeight - 0.5f + 0.5f;
	float fS = startSlice + 0.5f;
	int endSlice = startSlice + g_blockSlices;
	if (endSlice > dims.iVolWidth)
		endSlice = dims.iVolWidth;

	if (dims.iRaysPerDet > 1) {

		fP += (-0.5f*dims.iRaysPerDet + 0.5f)/dims.iRaysPerDet * fDetStep;
		const float fSubDetStep = fDetStep / dims.iRaysPerDet;
		fDistCorr /= dims.iRaysPerDet;

		fSliceStep -= dims.iRaysPerDet * fSubDetStep;

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
				fVal += tex2D(gT_volumeTexture, fS, fP);
				fP += fSubDetStep;
			}
			fP += fSliceStep;
			fS += 1.0f;
		}

	} else {

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fVal += tex2D(gT_volumeTexture, fS, fP);
			fP += fSliceStep;
			fS += 1.0f;
		}


	}

	D_projData[angle*projPitch+detector] += fVal * fDistCorr;
}


// projection for angles that are roughly vertical
// (detector roughly horizontal)
__global__ void FPvertical_simple(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;

	if (angle >= endAngle)
		return;

	const float theta = gC_angle[angle];
	const float cos_theta = __cosf(theta);
	const float sin_theta = __sinf(theta);

	// compute start detector for this block/angle:
	const int detRegion = blockIdx.y;

	const int detector = detRegion * g_detBlockSize + relDet;

	// Now project the part of the ray to angle,detector through
	// slices startSlice to startSlice+g_blockSlices-1

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	const float fDetStep = gC_angle_detsize[angle] / cos_theta;
	float fSliceStep = sin_theta / cos_theta;
	float fDistCorr;
	if (cos_theta < 0.0f)
		fDistCorr = -outputScale / cos_theta; 
	else
		fDistCorr = outputScale / cos_theta;

	float fVal = 0.0f;
	float fP = (detector - 0.5f*dims.iProjDets + 0.5f - gC_angle_offset[angle]) * fDetStep + (startSlice - 0.5f*dims.iVolHeight + 0.5f) * fSliceStep + 0.5f*dims.iVolWidth - 0.5f + 0.5f;
	float fS = startSlice+0.5f;
	int endSlice = startSlice + g_blockSlices;
	if (endSlice > dims.iVolHeight)
		endSlice = dims.iVolHeight;

	if (dims.iRaysPerDet > 1) {

		fP += (-0.5f*dims.iRaysPerDet + 0.5f)/dims.iRaysPerDet * fDetStep;
		const float fSubDetStep = fDetStep / dims.iRaysPerDet;
		fDistCorr /= dims.iRaysPerDet;

		fSliceStep -= dims.iRaysPerDet * fSubDetStep;

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
				fVal += tex2D(gT_volumeTexture, fP, fS);
				fP += fSubDetStep;
			}
			fP += fSliceStep;
			fS += 1.0f;
		}

	} else {

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fVal += tex2D(gT_volumeTexture, fP, fS);
			fP += fSliceStep;
			fS += 1.0f;
		}

	}

	D_projData[angle*projPitch+detector] += fVal * fDistCorr;
}




// Coordinates of center of detector pixel number t:
// x = (t - 0.5*nDets + 0.5 - fOffset) * fSize * cos(fAngle)
// y = - (t - 0.5*nDets + 0.5 - fOffset) * fSize * sin(fAngle)


static void convertAndUploadAngles(const SParProjection *projs, unsigned int nth, unsigned int ndets)
{
	float *angles = new float[nth];
	float *offsets = new float[nth];
	float *detsizes = new float[nth];

	for (int i = 0; i < nth; ++i)
		getParParameters(projs[i], ndets, angles[i], detsizes[i], offsets[i]);

	hipMemcpyToSymbol(HIP_SYMBOL(gC_angle), angles, nth*sizeof(float), 0, hipMemcpyHostToDevice); 
	hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_offset), offsets, nth*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_detsize), detsizes, nth*sizeof(float), 0, hipMemcpyHostToDevice); 
}



bool FP_simple_internal(float* D_volumeData, unsigned int volumePitch,
               float* D_projData, unsigned int projPitch,
               const SDimensions& dims, const SParProjection* angles,
               float outputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	assert(angles);

	hipArray* D_dataArray;
	bindVolumeDataTexture(D_volumeData, D_dataArray, volumePitch, dims.iVolWidth, dims.iVolHeight);


	convertAndUploadAngles(angles, dims.iProjAngles, dims.iProjDets);


	dim3 dimBlock(g_detBlockSize, g_anglesPerBlock); // detector block size, angles

	std::list<hipStream_t> streams;


	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	// TODO: Check if it's worth it to store this info instead
	// of recomputing it every FP.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	bool blockVertical = false;
	for (unsigned int a = 0; a <= dims.iProjAngles; ++a) {
		bool vertical = false;
		// TODO: Having <= instead of < below causes a 5% speedup.
		// Maybe we should detect corner cases and put them in the optimal
		// group of angles.
		if (a != dims.iProjAngles)
			vertical = (fabsf(angles[a].fRayX) <= fabsf(angles[a].fRayY));
		if (a == dims.iProjAngles || vertical != blockVertical) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {
				dim3 dimGrid((blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock,
				             (dims.iProjDets+g_detBlockSize-1)/g_detBlockSize); // angle blocks, detector blocks

				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);
				//printf("angle block: %d to %d, %d\n", blockStart, blockEnd, blockVertical);
				if (!blockVertical)
					for (unsigned int i = 0; i < dims.iVolWidth; i += g_blockSlices)
						FPhorizontal_simple<<<dimGrid, dimBlock, 0, stream>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);
				else
					for (unsigned int i = 0; i < dims.iVolHeight; i += g_blockSlices)
						FPvertical_simple<<<dimGrid, dimBlock, 0, stream>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);
			}
			blockVertical = vertical;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipFreeArray(D_dataArray);
		

	return true;
}

bool FP_simple(float* D_volumeData, unsigned int volumePitch,
               float* D_projData, unsigned int projPitch,
               const SDimensions& dims, const SParProjection* angles,
               float outputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FP_simple_internal(D_volumeData, volumePitch,
		                         D_projData + iAngle * projPitch, projPitch,
		                         subdims, angles + iAngle,
		                         outputScale);
		if (!ret)
			return false;
	}
	return true;
}

bool FP(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const SParProjection* angles,
        float outputScale)
{
	return FP_simple(D_volumeData, volumePitch, D_projData, projPitch,
	                 dims, angles, outputScale);

}


}
