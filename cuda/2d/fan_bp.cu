#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>
#include <iostream>


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

struct DevFanParams {
	float fNumC;
	float fNumX;
	float fNumY;
	float fDenC;
	float fDenX;
	float fDenY;
};

__constant__ DevFanParams gC_C[g_MaxAngles];

template<bool FBPWEIGHT>
__global__ void devFanBP(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = - ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float fNumC = gC_C[angle].fNumC;
		const float fNumX = gC_C[angle].fNumX;
		const float fNumY = gC_C[angle].fNumY;
		const float fDenX = gC_C[angle].fDenX;
		const float fDenY = gC_C[angle].fDenY;

		const float fNum = fNumC + fNumX * fX + fNumY * fY;
		const float fDen = (FBPWEIGHT ? 1.0 : gC_C[angle].fDenC) + fDenX * fX + fDenY * fY;

		// Scale factor is the approximate number of rays traversing this pixel,
		// given by the inverse size of a detector pixel scaled by the magnification
		// factor of this pixel.
		// Magnification factor is || u (d-s) || / || u (x-s) ||

		const float fr = __fdividef(1.0f, fDen);
		const float fT = fNum * fr;
		fVal += tex2D<float>(tex, fT, fA) * (FBPWEIGHT ? fr * fr : fr);
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// supersampling version
__global__ void devFanBP_SS(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fXb = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);
	const float fYb = - ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);

	const float fSubStep = 1.0f/dims.iRaysPerPixelDim;

	float* volData = (float*)D_volData;

	fOutputScale /= (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float fNumC = gC_C[angle].fNumC;
		const float fNumX = gC_C[angle].fNumX;
		const float fNumY = gC_C[angle].fNumY;
		const float fDenC = gC_C[angle].fDenC;
		const float fDenX = gC_C[angle].fDenX;
		const float fDenY = gC_C[angle].fDenY;

		// TODO: Optimize these loops...
		float fX = fXb;
		for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
			float fY = fYb;
			for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {

				const float fNum = fNumC + fNumX * fX + fNumY * fY;
				const float fDen = fDenC + fDenX * fX + fDenY * fY;
				const float fr = __fdividef(1.0f, fDen);

				const float fT = fNum * fr;
				fVal += tex2D<float>(tex, fT, fA) * fr;
				fY -= fSubStep;
			}
			fX += fSubStep;
		}
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}


// BP specifically for SART.
// It includes (free) weighting with voxel weight.
// It assumes the proj texture is set up _without_ padding, unlike regular BP.
__global__ void devFanBP_SART(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = - ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	const float fNumC = gC_C[0].fNumC;
	const float fNumX = gC_C[0].fNumX;
	const float fNumY = gC_C[0].fNumY;
	const float fDenC = gC_C[0].fDenC;
	const float fDenX = gC_C[0].fDenX;
	const float fDenY = gC_C[0].fDenY;

	const float fNum = fNumC + fNumX * fX + fNumY * fY;
	const float fDen = fDenC + fDenX * fX + fDenY * fY;

	const float fr = __fdividef(1.0f, fDen);
	const float fT = fNum * fr;
	// NB: The scale constant in devBP is cancelled out by the SART weighting
	const float fVal = tex2D<float>(tex, fT, 0.5f);

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

struct Vec2 {
	double x;
	double y;
	Vec2(double x_, double y_) : x(x_), y(y_) { }
	Vec2 operator+(const Vec2 &b) const {
		return Vec2(x + b.x, y + b.y);
	}
	Vec2 operator-(const Vec2 &b) const {
		return Vec2(x - b.x, y - b.y);
	}
	Vec2 operator-() const {
		return Vec2(-x, -y);
	}
	double norm() const {
		return sqrt(x*x + y*y);
	}
};

double det2(const Vec2 &a, const Vec2 &b) {
	return a.x * b.y - a.y * b.x;
}


bool transferConstants(const SFanProjection* angles, unsigned int iProjAngles, bool FBP)
{
	DevFanParams *p = new DevFanParams[iProjAngles];

	// We need three values in the kernel:
	// projected coordinates of pixels on the detector:
	// || x (s-d) || + ||s d|| / || u (s-x) ||

	// ray density weighting factor for the adjoint
	// || u (s-d) || / ( |u| * || u (s-x) || )

	// fan-beam FBP weighting factor
	// ( || u s || / || u (s-x) || ) ^ 2



	for (unsigned int i = 0; i < iProjAngles; ++i) {
		Vec2 u(angles[i].fDetUX, angles[i].fDetUY);
		Vec2 s(angles[i].fSrcX, angles[i].fSrcY);
		Vec2 d(angles[i].fDetSX, angles[i].fDetSY);



		double fScale;
		if (!FBP) {
			// goal: 1/fDen = || u (s-d) || / ( |u| * || u (s-x) || )
			// fDen = ( |u| * || u (s-x) || ) / || u (s-d) ||
			// i.e. scale = |u| /  || u (s-d) ||

			fScale = u.norm() / det2(u, s-d);
		} else {
			// goal: 1/fDen = || u s || / || u (s-x) ||
			// fDen = || u (s-x) || / || u s ||
			// i.e., scale = 1 / || u s ||

			fScale = 1.0 / det2(u, s);
		}

		p[i].fNumC = fScale * det2(s,d);
		p[i].fNumX = fScale * (s-d).y;
		p[i].fNumY = -fScale * (s-d).x;
		p[i].fDenC = fScale * det2(u, s); // == 1.0 for FBP
		p[i].fDenX = fScale * u.y;
		p[i].fDenY = -fScale * u.x;
	}

	// TODO: Check for errors
	hipMemcpyToSymbol(HIP_SYMBOL(gC_C), p, iProjAngles*sizeof(DevFanParams), 0, hipMemcpyHostToDevice);

	delete [] p;

	return true;
}


bool FanBP_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipTextureObject_t D_texObj;
	if (!createTextureObjectPitch2D(D_projData, D_texObj, projPitch, dims.iProjDets, dims.iProjAngles))
		return false;

	bool ok = transferConstants(angles, dims.iProjAngles, false);
	if (!ok) {
		hipDestroyTextureObject(D_texObj);
		return false;
	}

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {
		if (dims.iRaysPerPixelDim > 1)
			devFanBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, D_texObj, i, dims, fOutputScale);
		else
			devFanBP<false><<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, D_texObj, i, dims, fOutputScale);
	}

	ok = checkCuda(hipStreamSynchronize(stream), "FanBP");

	hipStreamDestroy(stream);

	hipDestroyTextureObject(D_texObj);

	return ok;
}

bool FanBP_FBPWeighted_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipTextureObject_t D_texObj;
	if (!createTextureObjectPitch2D(D_projData, D_texObj, projPitch, dims.iProjDets, dims.iProjAngles))
		return false;

	bool ok = transferConstants(angles, dims.iProjAngles, true);
	if (!ok) {
		hipDestroyTextureObject(D_texObj);
		return false;
	}

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {
		devFanBP<true><<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, D_texObj, i, dims, fOutputScale);
	}

	ok = checkCuda(hipStreamSynchronize(stream), "FanBP_FBPWeighted");

	hipStreamDestroy(stream);

	hipDestroyTextureObject(D_texObj);

	return ok;
}

// D_projData is a pointer to one padded sinogram line
bool FanBP_SART(float* D_volumeData, unsigned int volumePitch,
                float* D_projData, unsigned int projPitch,
                unsigned int angle,
                const SDimensions& dims, const SFanProjection* angles,
                float fOutputScale)
{
	// only one angle
	hipTextureObject_t D_texObj;
	if (!createTextureObjectPitch2D(D_projData, D_texObj, projPitch, dims.iProjDets, 1, hipAddressModeClamp))
		return false;

	bool ok = transferConstants(angles + angle, 1, false);
	if (!ok) {
		hipDestroyTextureObject(D_texObj);
		return false;
	}

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devFanBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, D_texObj, dims, fOutputScale);

	ok = checkCuda(hipDeviceSynchronize(), "FanBP_SART");

	hipDestroyTextureObject(D_texObj);

	return ok;
}

bool FanBP(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanBP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);
		if (!ret)
			return false;
	}
	return true;
}

bool FanBP_FBPWeighted(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanBP_FBPWeighted_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);

		if (!ret)
			return false;
	}
	return true;
}


}
