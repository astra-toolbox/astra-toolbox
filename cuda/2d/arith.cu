#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cassert>

namespace astraCUDA {


struct opAddScaled {
	__device__ void operator()(float& out, const float in, const float inp) {
		out += in * inp;
	}
};
struct opScaleAndAdd {
	__device__ void operator()(float& out, const float in, const float inp) {
		out = in + out * inp;
	}
};
struct opAddMulScaled {
	__device__ void operator()(float& out, const float in1, const float in2, const float inp) {
		out += in1 * in2 * inp;
	}
};
struct opAddMul {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 * in2;
	}
};
struct opAdd {
	__device__ void operator()(float& out, const float in) {
		out += in;
	}
};
struct opAdd2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 + in2;
	}
};
struct opMul {
	__device__ void operator()(float& out, const float in) {
		out *= in;
	}
};
struct opDiv {
	__device__ void operator()(float& out, const float in) {
		if (in > 0.000001f) // out is assumed to be positive
			out /= in;
		else
			out = 0.0f;
	}
};
struct opMul2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out *= in1 * in2;
	}
};
struct opDividedBy {
	__device__ void operator()(float& out, const float in) {
		if (out > 0.000001f) // out is assumed to be positive
			out = in / out;
		else
			out = 0.0f;
	}
};
struct opInvert {
	__device__ void operator()(float& out) {
		if (out > 0.000001f) // out is assumed to be positive
			out = 1 / out;
		else
			out = 0.0f;
	}
};
struct opSet {
	__device__ void operator()(float& out, const float inp) {
		out = inp;
	}
};
struct opClampMin {
	__device__ void operator()(float& out, const float inp) {
		if (out < inp)
			out = inp;
	}
};
struct opClampMax {
	__device__ void operator()(float& out, const float inp) {
		if (out > inp)
			out = inp;
	}
};
struct opClampMinMask {
	__device__ void operator()(float& out, const float in) {
		if (out < in)
			out = in;
	}
};
struct opClampMaxMask {
	__device__ void operator()(float& out, const float in) {
		if (out > in)
			out = in;
	}
};
struct opSetMaskedValues {
	__device__ void operator()(float& out, const float in, const float inp) {
		if (!in)
			out = inp;
	}
};
struct opSegmentAndMask {
	__device__ void operator()(float& out1, float& out2, const float inp1, const float inp2) {
		if (out1 >= inp1) {
			out1 = inp2;
			out2 = 0.0f;
		}

	}

};
struct opMulMask {
	__device__ void operator()(float& out, const float mask, const float in) {
		if (mask > 0.0f) {
			out *= in;
		}
	}
};



template<class op, unsigned int repeat>
__global__ void devtoD(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devFtoD(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devFFtoDD(float* pfOut1, float* pfOut2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut1[off], pfOut2[off], fParam1, fParam2);
		off += pitch;
		y++;
	}
}



template<class op, unsigned int repeat>
__global__ void devDtoD(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDFtoD(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDtoD(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int repeat>
__global__ void devDDFtoD(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = y*pitch+x;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off], fParam);
		off += pitch;
		y++;
	}
}















template<typename op>
void processVolCopy(float* out, const SDimensions& dims)
{
	float* D_out;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);

	processVol<op>(D_out, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
}

template<typename op>
void processVolCopy(float* out, float param, const SDimensions& dims)
{
	float* D_out;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);

	processVol<op>(D_out, param, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
}

template<typename op>
void processVolCopy(float* out1, float* out2, float param1, float param2, const SDimensions& dims)
{
	float* D_out1;
	float* D_out2;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out1, pitch, dims);
	copyVolumeToDevice(out1, width, dims, D_out1, pitch);
	allocateVolumeData(D_out2, pitch, dims);
	copyVolumeToDevice(out2, width, dims, D_out2, pitch);

	processVol<op>(D_out1, D_out2, param1, param2, pitch, dims);

	copyVolumeFromDevice(out1, width, dims, D_out1, pitch);
	copyVolumeFromDevice(out2, width, dims, D_out2, pitch);

	hipFree(D_out1);
	hipFree(D_out2);
}


template<typename op>
void processVolCopy(float* out, const float* in, const SDimensions& dims)
{
	float* D_out;
	float* D_in;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);
	allocateVolumeData(D_in, pitch, dims);
	copyVolumeToDevice(in, width, dims, D_in, pitch);

	processVol<op>(D_out, D_in, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in);
}

template<typename op>
void processVolCopy(float* out, const float* in, float param, const SDimensions& dims)
{
	float* D_out;
	float* D_in;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);
	allocateVolumeData(D_in, pitch, dims);
	copyVolumeToDevice(in, width, dims, D_in, pitch);

	processVol<op>(D_out, D_in, param, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in);
}

template<typename op>
void processVolCopy(float* out, const float* in1, const float* in2, const SDimensions& dims)
{
	float* D_out;
	float* D_in1;
	float* D_in2;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);
	allocateVolumeData(D_in1, pitch, dims);
	copyVolumeToDevice(in1, width, dims, D_in1, pitch);
	allocateVolumeData(D_in2, pitch, dims);
	copyVolumeToDevice(in2, width, dims, D_in2, pitch);

	processVol<op>(D_out, D_in1, D_in2, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in1);
	hipFree(D_in2);
}

template<typename op>
void processVolCopy(float* out, const float* in1, const float* in2, float param, const SDimensions& dims)
{
	float* D_out;
	float* D_in1;
	float* D_in2;
	size_t width = dims.iVolWidth;

	unsigned int pitch;
	allocateVolumeData(D_out, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_out, pitch);
	allocateVolumeData(D_in1, pitch, dims);
	copyVolumeToDevice(in1, width, dims, D_in1, pitch);
	allocateVolumeData(D_in2, pitch, dims);
	copyVolumeToDevice(in2, width, dims, D_in2, pitch);

	processVol<op>(D_out, D_in1, D_in2, param, pitch, dims);

	copyVolumeFromDevice(out, width, dims, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in1);
	hipFree(D_in2);
}








template<typename op>
void processData(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+511)/512);

	devtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processData(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processData(float* pfOut1, float* pfOut2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devFFtoDD<op, 32><<<gridSize, blockSize>>>(pfOut1, pfOut2, fParam1, fParam2, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}


template<typename op>
void processData(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processData(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processData(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDDFtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}

template<typename op>
void processData(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDDtoD<op, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, pitch, width, height);

	checkCuda(hipDeviceSynchronize(), __FUNCTION__);
}








template<typename op>
void processVol(float* out, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, pitch, dims.iVolWidth, dims.iVolHeight);
}

template<typename op>
void processVol(float* out, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, param, pitch, dims.iVolWidth, dims.iVolHeight);
}

template<typename op>
void processVol(float* out1, float* out2, float param1, float param2, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out1, out2, param1, param2, pitch, dims.iVolWidth, dims.iVolHeight);
}


template<typename op>
void processVol(float* out, const float* in, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in, pitch, dims.iVolWidth, dims.iVolHeight);
}

template<typename op>
void processVol(float* out, const float* in, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in, param, pitch, dims.iVolWidth, dims.iVolHeight);
}

template<typename op>
void processVol(float* out, const float* in1, const float* in2, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in1, in2, pitch, dims.iVolWidth, dims.iVolHeight);
}

template<typename op>
void processVol(float* out, const float* in1, const float* in2, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in2, in2, param, pitch, dims.iVolWidth, dims.iVolHeight);
}




template<typename op>
void processSino(float* out, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, pitch, dims.iProjDets, dims.iProjAngles);
}

template<typename op>
void processSino(float* out, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, param, pitch, dims.iProjDets, dims.iProjAngles);
}

template<typename op>
void processSino(float* out1, float* out2, float param1, float param2, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out1, out2, param1, param2, pitch, dims.iProjDets, dims.iProjAngles);
}


template<typename op>
void processSino(float* out, const float* in, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in, pitch, dims.iProjDets, dims.iProjAngles);
}

template<typename op>
void processSino(float* out, const float* in, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in, param, pitch, dims.iProjDets, dims.iProjAngles);
}

template<typename op>
void processSino(float* out, const float* in1, const float* in2, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in1, in2, pitch, dims.iProjDets, dims.iProjAngles);
}

template<typename op>
void processSino(float* out, const float* in1, const float* in2, float param, unsigned int pitch, const SDimensions& dims)
{
	processData<op>(out, in2, in2, param, pitch, dims.iProjDets, dims.iProjAngles);
}























#define INST_DFtoD(name) \
  template void processVolCopy<name>(float* out, const float* in, float param, const SDimensions& dims); \
  template void processVol<name>(float* out, const float* in, float param, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, const float* in, float param, unsigned int pitch, const SDimensions& dims);

#define INST_DtoD(name) \
  template void processVolCopy<name>(float* out, const float* in, const SDimensions& dims); \
  template void processVol<name>(float* out, const float* in, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, const float* in, unsigned int pitch, const SDimensions& dims);

#define INST_DDtoD(name) \
  template void processVolCopy<name>(float* out, const float* in1, const float* in2, const SDimensions& dims); \
  template void processVol<name>(float* out, const float* in1, const float* in2, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, const float* in1, const float* in2, unsigned int pitch, const SDimensions& dims);

#define INST_DDFtoD(name) \
  template void processVolCopy<name>(float* out, const float* in1, const float* in2, float fParam, const SDimensions& dims); \
  template void processVol<name>(float* out, const float* in1, const float* in2, float fParam, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, const float* in1, const float* in2, float fParam, unsigned int pitch, const SDimensions& dims);


#define INST_toD(name) \
  template void processVolCopy<name>(float* out, const SDimensions& dims); \
  template void processVol<name>(float* out, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, unsigned int pitch, const SDimensions& dims);

#define INST_FtoD(name) \
  template void processVolCopy<name>(float* out, float param, const SDimensions& dims); \
  template void processVol<name>(float* out, float param, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out, float param, unsigned int pitch, const SDimensions& dims);

#define INST_FFtoDD(name) \
  template void processVolCopy<name>(float* out1, float* out2, float fParam1, float fParam2, const SDimensions& dims); \
  template void processVol<name>(float* out1, float* out2, float fParam1, float fParam2, unsigned int pitch, const SDimensions& dims); \
  template void processSino<name>(float* out1, float* out2, float fParam1, float fParam2, unsigned int pitch, const SDimensions& dims);



INST_DFtoD(opAddScaled)
INST_DFtoD(opScaleAndAdd)
INST_DDFtoD(opAddMulScaled)
INST_DDtoD(opAddMul)
INST_DDtoD(opMul2)
INST_DDtoD(opAdd2)
INST_DtoD(opMul)
INST_DDtoD(opMulMask)
INST_DtoD(opAdd)
INST_DtoD(opDividedBy)
INST_toD(opInvert)
INST_FtoD(opSet)
INST_FtoD(opMul)
INST_DtoD(opDiv)
INST_DFtoD(opMulMask)
INST_FtoD(opAdd)
INST_FtoD(opClampMin)
INST_FtoD(opClampMax)
INST_DtoD(opClampMinMask)
INST_DtoD(opClampMaxMask)

// PDART-specific:
INST_DFtoD(opSetMaskedValues)
INST_FFtoDD(opSegmentAndMask)

}
