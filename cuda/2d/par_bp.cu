#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>
#include <iostream>


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

__constant__ float gC_angle_scaled_sin[g_MaxAngles];
__constant__ float gC_angle_scaled_cos[g_MaxAngles];
__constant__ float gC_angle_offset[g_MaxAngles];
__constant__ float gC_angle_scale[g_MaxAngles];

// TODO: Templated version with/without scale? (Or only the global outputscale)
__global__ void devBP(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f );

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float scaled_cos_theta = gC_angle_scaled_cos[angle];
		const float scaled_sin_theta = gC_angle_scaled_sin[angle];
		const float TOffset = gC_angle_offset[angle];
		const float scale = gC_angle_scale[angle];

		const float fT = fX * scaled_cos_theta - fY * scaled_sin_theta + TOffset;
		fVal += tex2D<float>(tex, fT, fA) * scale;
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// supersampling version
__global__ void devBP_SS(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, unsigned int startAngle, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim);

	const float fSubStep = 1.0f/(dims.iRaysPerPixelDim); // * dims.fDetScale);

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;

	fOutputScale /= (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);

	for (int angle = startAngle; angle < endAngle; ++angle)
	{
		const float cos_theta = gC_angle_scaled_cos[angle];
		const float sin_theta = gC_angle_scaled_sin[angle];
		const float TOffset = gC_angle_offset[angle];
		const float scale = gC_angle_scale[angle];

		float fT = fX * cos_theta - fY * sin_theta + TOffset;

		for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
			float fTy = fT;
			fT += fSubStep * cos_theta;
			for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
				fVal += tex2D<float>(tex, fTy, fA) * scale;
				fTy -= fSubStep * sin_theta;
			}
		}
		fA += 1.0f;
	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

__global__ void devBP_SART(float* D_volData, unsigned int volPitch, hipTextureObject_t tex, float offset, float angle_sin, float angle_cos, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f );
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f );

	const float fT = fX * angle_cos - fY * angle_sin + offset;
	const float fVal = tex2D<float>(tex, fT, 0.5f);

	// NB: The 'scale' constant in devBP is cancelled out by the SART weighting

	D_volData[Y*volPitch+X] += fVal * fOutputScale;
}


bool BP_internal(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const SParProjection* angles,
        float fOutputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipTextureObject_t D_texObj;
	if (!createTextureObjectPitch2D(D_projData, D_texObj, projPitch, dims.iProjDets, dims.iProjAngles))
		return false;

	float* angle_scaled_sin = new float[dims.iProjAngles];
	float* angle_scaled_cos = new float[dims.iProjAngles];
	float* angle_offset = new float[dims.iProjAngles];
	float* angle_scale = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		double d = angles[i].fDetUX * angles[i].fRayY - angles[i].fDetUY * angles[i].fRayX;
		angle_scaled_cos[i] = angles[i].fRayY / d;
		angle_scaled_sin[i] = -angles[i].fRayX / d;
		angle_offset[i] = (angles[i].fDetSY * angles[i].fRayX - angles[i].fDetSX * angles[i].fRayY) / d;
		angle_scale[i] = sqrt(angles[i].fRayX * angles[i].fRayX + angles[i].fRayY * angles[i].fRayY) / abs(d);
	}
	//fprintf(stderr, "outputscale in BP_internal: %f, %f\n", fOutputScale, angle_scale[0]);
	//fprintf(stderr, "ray in BP_internal: %f,%f (length %f)\n", angles[0].fRayX, angles[0].fRayY, sqrt(angles[0].fRayX * angles[0].fRayX + angles[0].fRayY * angles[0].fRayY));

	hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_scaled_sin), angle_scaled_sin, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_scaled_cos), angle_scaled_cos, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e3 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_offset), angle_offset, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e4 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_scale), angle_scale, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	assert(e1 == hipSuccess);
	assert(e2 == hipSuccess);
	assert(e3 == hipSuccess);
	assert(e4 == hipSuccess);


	delete[] angle_scaled_sin;
	delete[] angle_scaled_cos;
	delete[] angle_offset;
	delete[] angle_scale;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {

		if (dims.iRaysPerPixelDim > 1)
			devBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, D_texObj, i, dims, fOutputScale);
		else
			devBP<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, D_texObj, i, dims, fOutputScale);
	}

	bool ok = checkCuda(hipStreamSynchronize(stream), "par_bp");

	hipStreamDestroy(stream);

	hipDestroyTextureObject(D_texObj);

	return ok;
}

bool BP(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const SParProjection* angles, float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = BP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle, fOutputScale);
		if (!ret)
			return false;
	}
	return true;
}


bool BP_SART(float* D_volumeData, unsigned int volumePitch,
             float* D_projData, unsigned int projPitch,
             unsigned int angle, const SDimensions& dims,
             const SParProjection* angles, float fOutputScale)
{
	// Only one angle.
	// We need to Clamp to the border pixels instead of to zero, because
	// SART weights with ray length.
	hipTextureObject_t D_texObj;
	if (!createTextureObjectPitch2D(D_projData, D_texObj, projPitch, dims.iProjDets, 1, hipAddressModeClamp))
		return false;

	double d = angles[angle].fDetUX * angles[angle].fRayY - angles[angle].fDetUY * angles[angle].fRayX;
	float angle_scaled_cos = angles[angle].fRayY / d;
	float angle_scaled_sin = -angles[angle].fRayX / d; // TODO: Check signs
	float angle_offset = (angles[angle].fDetSY * angles[angle].fRayX - angles[angle].fDetSX * angles[angle].fRayY) / d;
	// NB: The adjoint scaling factor from regular BP is cancelled out by the SART weighting
	//fOutputScale *= sqrt(angles[angle].fRayX * angles[angle].fRayX + angles[angle].fRayY * angles[angle].fRayY) / abs(d);

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, D_texObj, angle_offset, angle_scaled_sin, angle_scaled_cos, dims, fOutputScale);

	bool ok = checkCuda(hipDeviceSynchronize(), "BP_SART");

	hipDestroyTextureObject(D_texObj);

	return ok;
}


}
