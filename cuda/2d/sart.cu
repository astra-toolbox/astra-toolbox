#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/sart.h"
#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"
#include "astra/cuda/2d/fan_fp.h"
#include "astra/cuda/2d/fan_bp.h"
#include "astra/cuda/2d/par_fp.h"
#include "astra/cuda/2d/par_bp.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA {

// FIXME: Remove these functions. (Outdated)
__global__ void devMUL_SART(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	pfOut[x] *= pfIn[x];
}

void MUL_SART(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, 1);

	devMUL_SART<<<gridSize, blockSize>>>(pfOut, pfIn, pitch, width);

	checkCuda(hipDeviceSynchronize(), "MUL_SART");
}



SART::SART() : ReconAlgo()
{
	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;

	projectionOrder = 0;
	projectionCount = 0;
	iteration = 0;
	customOrder = false;

	fRelaxation = 1.0f;
}


SART::~SART()
{
	reset();
}

void SART::reset()
{
	hipFree(D_projData);
	hipFree(D_tmpData);
	hipFree(D_lineWeight);

	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	if (projectionOrder != NULL) delete[] projectionOrder;
	projectionOrder = 0;
	projectionCount = 0;
	iteration = 0;
	customOrder = false;
	fRelaxation = 1.0f;

	ReconAlgo::reset();
}

bool SART::init()
{
	if (useVolumeMask) {
		allocateVolumeData(D_tmpData, tmpPitch, dims);
		zeroVolumeData(D_tmpData, tmpPitch, dims);
	}

	// NB: Non-standard dimensions
	SDimensions linedims = dims;
	linedims.iProjAngles = 1;
	allocateProjectionData(D_projData, projPitch, linedims);
	zeroProjectionData(D_projData, projPitch, linedims);
	
	allocateProjectionData(D_lineWeight, linePitch, dims);
	zeroProjectionData(D_lineWeight, linePitch, dims);

	// We can't precompute lineWeights when using a mask
	if (!useVolumeMask)
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool SART::setProjectionOrder(int* _projectionOrder, int _projectionCount)
{
	customOrder = true;
	projectionCount = _projectionCount;
	projectionOrder = new int[projectionCount];
	for (int i = 0; i < projectionCount; i++) {
		projectionOrder[i] = _projectionOrder[i];
	}

	return true;
}


bool SART::precomputeWeights()
{
	zeroProjectionData(D_lineWeight, linePitch, dims);
	if (useVolumeMask) {
		callFP(D_maskData, maskPitch, D_lineWeight, linePitch, 1.0f);
	} else {
		// Allocate tmpData temporarily
		allocateVolumeData(D_tmpData, tmpPitch, dims);
		zeroVolumeData(D_tmpData, tmpPitch, dims);


		processVol<opSet>(D_tmpData, 1.0f, tmpPitch, dims);
		callFP(D_tmpData, tmpPitch, D_lineWeight, linePitch, 1.0f);


		hipFree(D_tmpData);
		D_tmpData = 0;
	}
	processSino<opInvert>(D_lineWeight, linePitch, dims);

	return true;
}

bool SART::iterate(unsigned int iterations)
{
	if (useVolumeMask)
		precomputeWeights();

	// iteration
	for (unsigned int iter = 0; iter < iterations && !astra::shouldAbort(); ++iter) {

		int angle;
		if (customOrder) {
			angle = projectionOrder[iteration % projectionCount];
		} else {
			angle = iteration % dims.iProjAngles;  
		}

		// copy one line of sinogram to projection data
		// NB: Non-standard dimensions
		SDimensions linedims = dims;
		linedims.iProjAngles = 1;
		duplicateProjectionData(D_projData, D_sinoData + angle*sinoPitch, sinoPitch, linedims);

		// do FP, subtracting projection from sinogram
		if (useVolumeMask) {
				duplicateVolumeData(D_tmpData, D_volumeData, volumePitch, dims);
				processVol<opMul>(D_tmpData, D_maskData, tmpPitch, dims);
				callFP_SART(D_tmpData, tmpPitch, D_projData, projPitch, angle, -1.0f);
		} else {
				callFP_SART(D_volumeData, volumePitch, D_projData, projPitch, angle, -1.0f);
		}

		MUL_SART(D_projData, D_lineWeight + angle*linePitch, projPitch, dims.iProjDets);

		if (useVolumeMask) {
			// BP, mask, and add back
			// TODO: Try putting the masking directly in the BP
			zeroVolumeData(D_tmpData, tmpPitch, dims);
			callBP_SART(D_tmpData, tmpPitch, D_projData, projPitch, angle, fRelaxation);
			processVol<opAddMul>(D_volumeData, D_maskData, D_tmpData, volumePitch, dims);
		} else {
			callBP_SART(D_volumeData, volumePitch, D_projData, projPitch, angle, fRelaxation);
		}

		if (useMinConstraint)
			processVol<opClampMin>(D_volumeData, fMinConstraint, volumePitch, dims);
		if (useMaxConstraint)
			processVol<opClampMax>(D_volumeData, fMaxConstraint, volumePitch, dims);

		iteration++;

	}

	return true;
}

float SART::computeDiffNorm()
{
	unsigned int pPitch;
	float *D_p;
	allocateProjectionData(D_p, pPitch, dims);

	// copy sinogram to D_p
	duplicateProjectionData(D_p, D_sinoData, sinoPitch, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_tmpData, D_volumeData, volumePitch, dims);
			processVol<opMul>(D_tmpData, D_maskData, tmpPitch, dims);
			callFP(D_tmpData, tmpPitch, D_p, pPitch, -1.0f);
	} else {
			callFP(D_volumeData, volumePitch, D_p, pPitch, -1.0f);
	}


	// compute norm of D_p
	float s = dotProduct2D(D_p, pPitch, dims.iProjDets, dims.iProjAngles);

	hipFree(D_p);

	return sqrt(s);
}

bool SART::callFP_SART(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       unsigned int angle, float outputScale)
{
	SDimensions d = dims;
	d.iProjAngles = 1;
	if (parProjs) {
		assert(!fanProjs);
		return FP(D_volumeData, volumePitch, D_projData, projPitch,
		          d, &parProjs[angle], outputScale * fProjectorScale);
	} else {
		assert(fanProjs);
		return FanFP(D_volumeData, volumePitch, D_projData, projPitch,
		             d, &fanProjs[angle], outputScale * fProjectorScale);
	}
}

bool SART::callBP_SART(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       unsigned int angle, float outputScale)
{
	// NB: No fProjectorScale here, as that it is cancelled out in the SART weighting
	if (parProjs) {
		assert(!fanProjs);
		return BP_SART(D_volumeData, volumePitch, D_projData, projPitch,
		               angle, dims, parProjs, outputScale);
	} else {
		assert(fanProjs);
		return FanBP_SART(D_volumeData, volumePitch, D_projData, projPitch,
		                  angle, dims, fanProjs, outputScale);
	}

}


}


