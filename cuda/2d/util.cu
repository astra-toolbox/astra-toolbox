#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA {

bool copyVolumeToDevice(const float* in_data, unsigned int in_pitch,
		const SDimensions& dims,
		float* outD_data, unsigned int out_pitch)
{
	size_t width = dims.iVolWidth;
	size_t height = dims.iVolHeight;

	return checkCuda(hipMemcpy2D(outD_data, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice), "copyVolumeToDevice");
}

bool copyVolumeFromDevice(float* out_data, unsigned int out_pitch,
		const SDimensions& dims,
		float* inD_data, unsigned int in_pitch)
{
	size_t width = dims.iVolWidth;
	size_t height = dims.iVolHeight;

	return checkCuda(hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost), "copyVolumeFromDevice");
}


bool copySinogramFromDevice(float* out_data, unsigned int out_pitch,
		const SDimensions& dims,
		float* inD_data, unsigned int in_pitch)
{
	size_t width = dims.iProjDets;
	size_t height = dims.iProjAngles;

	return checkCuda(hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost), "copySinogramFromDevice");
}

bool copySinogramToDevice(const float* in_data, unsigned int in_pitch,
		const SDimensions& dims,
		float* outD_data, unsigned int out_pitch)
{
	size_t width = dims.iProjDets;
	size_t height = dims.iProjAngles;

	return checkCuda(hipMemcpy2D(outD_data, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice), "copySinogramToDevice");
}


bool allocateVolume(float*& ptr, unsigned int width, unsigned int height, unsigned int& pitch)
{
	size_t p;
	if (!checkCuda(hipMallocPitch((void**)&ptr, &p, sizeof(float)*width, height), "allocateVolume")) {
		ASTRA_ERROR("Failed to allocate %dx%d GPU buffer", width, height);
		return false;
	}

	assert(p % sizeof(float) == 0);

	pitch = p / sizeof(float);

	return true;
}

bool zeroVolume(float* data, unsigned int pitch, unsigned int width, unsigned int height)
{
	return checkCuda(hipMemset2D(data, sizeof(float)*pitch, 0, sizeof(float)*width, height), "zeroVolume");
}

bool allocateVolumeData(float*& D_ptr, unsigned int& pitch, const SDimensions& dims)
{
	return allocateVolume(D_ptr, dims.iVolWidth, dims.iVolHeight, pitch);
}

bool allocateProjectionData(float*& D_ptr, unsigned int& pitch, const SDimensions& dims)
{
	return allocateVolume(D_ptr, dims.iProjDets, dims.iProjAngles, pitch);
}

bool zeroVolumeData(float* D_ptr, unsigned int pitch, const SDimensions& dims)
{
	return zeroVolume(D_ptr, pitch, dims.iVolWidth, dims.iVolHeight);
}

bool zeroProjectionData(float* D_ptr, unsigned int pitch, const SDimensions& dims)
{
	return zeroVolume(D_ptr, pitch, dims.iProjDets, dims.iProjAngles);
}

void duplicateVolumeData(float* D_dst, float* D_src, unsigned int pitch, const SDimensions& dims)
{
	hipMemcpy2D(D_dst, sizeof(float)*pitch, D_src, sizeof(float)*pitch, sizeof(float)*dims.iVolWidth, dims.iVolHeight, hipMemcpyDeviceToDevice);
}

void duplicateProjectionData(float* D_dst, float* D_src, unsigned int pitch, const SDimensions& dims)
{
	hipMemcpy2D(D_dst, sizeof(float)*pitch, D_src, sizeof(float)*pitch, sizeof(float)*dims.iProjDets, dims.iProjAngles, hipMemcpyDeviceToDevice);
}

bool createArrayAndTextureObject2D(float* data, hipArray*& dataArray, hipTextureObject_t& texObj, unsigned int pitch, unsigned int width, unsigned int height)
{
	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.

	hipChannelFormatDesc channelDesc =
	    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	dataArray = 0;
	if (!checkCuda(hipMallocArray(&dataArray, &channelDesc, width, height), "createTextureObject2D malloc"))
		return false;
	if (!checkCuda(hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice), "createTextureObject2D memcpy")) {
		hipFreeArray(dataArray);
		return false;
	}

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = dataArray;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	texObj = 0;

	if (!checkCuda(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "createTextureObject2D")) {
		hipFreeArray(dataArray);
		return false;
	}

	return true;
}

bool createTextureObjectPitch2D(float* data, hipTextureObject_t& texObj, unsigned int pitch, unsigned int width, unsigned int height, hipTextureAddressMode mode)
{
	hipChannelFormatDesc channelDesc =
	    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = (void*)data;
	resDesc.res.pitch2D.desc = channelDesc;
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;
	resDesc.res.pitch2D.pitchInBytes = sizeof(float)*pitch;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = mode;
	texDesc.addressMode[1] = mode;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	texObj = 0;

	return checkCuda(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "createTextureObjectPitch2D");
}




template <unsigned int blockSize>
__global__ void reduce1D(float *g_idata, float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;

	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		volatile float* smem = sdata;
		if (blockSize >= 64) smem[tid] += smem[tid + 32];
		if (blockSize >= 32) smem[tid] += smem[tid + 16];
		if (blockSize >= 16) smem[tid] += smem[tid + 8];
		if (blockSize >= 8) smem[tid] += smem[tid + 4];
		if (blockSize >= 4) smem[tid] += smem[tid + 2];
		if (blockSize >= 2) smem[tid] += smem[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2D(float *g_idata, float *g_odata,
                         unsigned int pitch,
                         unsigned int nx, unsigned int ny)
{
	extern __shared__ float sdata[];
	const unsigned int tidx = threadIdx.x;
	const unsigned int tidy = threadIdx.y;
	const unsigned int tid = tidy * 16 + tidx;

	unsigned int x = blockIdx.x*16 + tidx;
	unsigned int y = blockIdx.y*16 + tidy;

	sdata[tid] = 0;

	if (x < nx) {

		while (y < ny) {
			sdata[tid] += (g_idata[pitch*y+x] * g_idata[pitch*y+x]);
			y += 16 * gridDim.y;
		}

	}

	__syncthreads();

	if (tid < 128)
		sdata[tid] += sdata[tid + 128];
	__syncthreads();

	if (tid < 64)
		sdata[tid] += sdata[tid + 64];
	__syncthreads();

	if (tid < 32) { // 32 is warp size
		volatile float* smem = sdata;
		smem[tid] += smem[tid + 32];
		smem[tid] += smem[tid + 16];
		smem[tid] += smem[tid + 8];
		smem[tid] += smem[tid + 4];
		smem[tid] += smem[tid + 2];
		smem[tid] += smem[tid + 1];
	} 

	if (tid == 0)
		g_odata[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
}

float dotProduct2D(float* D_data, unsigned int pitch,
                   unsigned int width, unsigned int height)
{
	unsigned int bx = (width + 15) / 16;
	unsigned int by = (height + 127) / 128;
	unsigned int shared_mem2 = sizeof(float) * 16 * 16;

	dim3 dimBlock2(16, 16);
	dim3 dimGrid2(bx, by);

	float* D_buf;
	hipMalloc(&D_buf, sizeof(float) * (bx * by + 1) );
	float* D_res = D_buf + (bx*by);

	// Step 1: reduce 2D from image to a single vector, taking sum of squares

	reduce2D<<< dimGrid2, dimBlock2, shared_mem2>>>(D_data, D_buf, pitch, width, height);
	checkCuda(hipDeviceSynchronize(), "dotProduct2D reduce2D");

	// Step 2: reduce 1D: add up elements in vector
	if (bx * by > 512)
		reduce1D<512><<< 1, 512, sizeof(float)*512>>>(D_buf, D_res, bx*by);
	else if (bx * by > 128)
		reduce1D<128><<< 1, 128, sizeof(float)*128>>>(D_buf, D_res, bx*by);
	else if (bx * by > 32)
		reduce1D<32><<< 1, 32, sizeof(float)*32*2>>>(D_buf, D_res, bx*by);
	else if (bx * by > 8)
		reduce1D<8><<< 1, 8, sizeof(float)*8*2>>>(D_buf, D_res, bx*by);
	else
		reduce1D<1><<< 1, 1, sizeof(float)*1*2>>>(D_buf, D_res, bx*by);

	float x;
	hipMemcpy(&x, D_res, 4, hipMemcpyDeviceToHost);

	checkCuda(hipDeviceSynchronize(), "dotProduct2D");

	hipFree(D_buf);

	return x;
}

bool checkCuda(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		ASTRA_ERROR("%s: CUDA error %d: %s.", msg, err, hipGetErrorString(err));
		return false;
	} else {
		return true;
	}
}

}
