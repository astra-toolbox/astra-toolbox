#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2021, imec Vision Lab, University of Antwerp
           2014-2021, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA {

bool copyVolumeToDevice(const float* in_data, unsigned int in_pitch,
		const SDimensions& dims,
		float* outD_data, unsigned int out_pitch)
{
	size_t width = dims.iVolWidth;
	size_t height = dims.iVolHeight;
	// TODO: memory order
	hipError_t err;
	err = hipMemcpy2D(outD_data, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice);
	ASTRA_CUDA_ASSERT(err);
	assert(err == hipSuccess);
	return true;
}

bool copyVolumeFromDevice(float* out_data, unsigned int out_pitch,
		const SDimensions& dims,
		float* inD_data, unsigned int in_pitch)
{
	size_t width = dims.iVolWidth;
	size_t height = dims.iVolHeight;
	// TODO: memory order
	hipError_t err = hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost);
	ASTRA_CUDA_ASSERT(err);
	return true;
}


bool copySinogramFromDevice(float* out_data, unsigned int out_pitch,
		const SDimensions& dims,
		float* inD_data, unsigned int in_pitch)
{
	size_t width = dims.iProjDets;
	size_t height = dims.iProjAngles;
	// TODO: memory order
	hipError_t err = hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost);
	ASTRA_CUDA_ASSERT(err);
	return true;
}

bool copySinogramToDevice(const float* in_data, unsigned int in_pitch,
		const SDimensions& dims,
		float* outD_data, unsigned int out_pitch)
{
	size_t width = dims.iProjDets;
	size_t height = dims.iProjAngles;
	// TODO: memory order
	hipError_t err;
	err = hipMemcpy2D(outD_data, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice);
	ASTRA_CUDA_ASSERT(err);
	return true;
}


bool allocateVolume(float*& ptr, unsigned int width, unsigned int height, unsigned int& pitch)
{
	size_t p;
	hipError_t ret = hipMallocPitch((void**)&ptr, &p, sizeof(float)*width, height);
	if (ret != hipSuccess) {
		reportCudaError(ret);
		ASTRA_ERROR("Failed to allocate %dx%d GPU buffer", width, height);
		return false;
	}

	assert(p % sizeof(float) == 0);

	pitch = p / sizeof(float);

	return true;
}

void zeroVolume(float* data, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipError_t err;
	err = hipMemset2D(data, sizeof(float)*pitch, 0, sizeof(float)*width, height);
	ASTRA_CUDA_ASSERT(err);
}

bool allocateVolumeData(float*& D_ptr, unsigned int& pitch, const SDimensions& dims)
{
	return allocateVolume(D_ptr, dims.iVolWidth, dims.iVolHeight, pitch);
}

bool allocateProjectionData(float*& D_ptr, unsigned int& pitch, const SDimensions& dims)
{
	return allocateVolume(D_ptr, dims.iProjDets, dims.iProjAngles, pitch);
}

void zeroVolumeData(float* D_ptr, unsigned int pitch, const SDimensions& dims)
{
	zeroVolume(D_ptr, pitch, dims.iVolWidth, dims.iVolHeight);
}

void zeroProjectionData(float* D_ptr, unsigned int pitch, const SDimensions& dims)
{
	zeroVolume(D_ptr, pitch, dims.iProjDets, dims.iProjAngles);
}

void duplicateVolumeData(float* D_dst, float* D_src, unsigned int pitch, const SDimensions& dims)
{
	hipMemcpy2D(D_dst, sizeof(float)*pitch, D_src, sizeof(float)*pitch, sizeof(float)*dims.iVolWidth, dims.iVolHeight, hipMemcpyDeviceToDevice);
}

void duplicateProjectionData(float* D_dst, float* D_src, unsigned int pitch, const SDimensions& dims)
{
	hipMemcpy2D(D_dst, sizeof(float)*pitch, D_src, sizeof(float)*pitch, sizeof(float)*dims.iProjDets, dims.iProjAngles, hipMemcpyDeviceToDevice);
}

template <unsigned int blockSize>
__global__ void reduce1D(float *g_idata, float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;

	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		volatile float* smem = sdata;
		if (blockSize >= 64) smem[tid] += smem[tid + 32];
		if (blockSize >= 32) smem[tid] += smem[tid + 16];
		if (blockSize >= 16) smem[tid] += smem[tid + 8];
		if (blockSize >= 8) smem[tid] += smem[tid + 4];
		if (blockSize >= 4) smem[tid] += smem[tid + 2];
		if (blockSize >= 2) smem[tid] += smem[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2D(float *g_idata, float *g_odata,
                         unsigned int pitch,
                         unsigned int nx, unsigned int ny)
{
	extern __shared__ float sdata[];
	const unsigned int tidx = threadIdx.x;
	const unsigned int tidy = threadIdx.y;
	const unsigned int tid = tidy * 16 + tidx;

	unsigned int x = blockIdx.x*16 + tidx;
	unsigned int y = blockIdx.y*16 + tidy;

	sdata[tid] = 0;

	if (x < nx) {

		while (y < ny) {
			sdata[tid] += (g_idata[pitch*y+x] * g_idata[pitch*y+x]);
			y += 16 * gridDim.y;
		}

	}

	__syncthreads();

	if (tid < 128)
		sdata[tid] += sdata[tid + 128];
	__syncthreads();

	if (tid < 64)
		sdata[tid] += sdata[tid + 64];
	__syncthreads();

	if (tid < 32) { // 32 is warp size
		volatile float* smem = sdata;
		smem[tid] += smem[tid + 32];
		smem[tid] += smem[tid + 16];
		smem[tid] += smem[tid + 8];
		smem[tid] += smem[tid + 4];
		smem[tid] += smem[tid + 2];
		smem[tid] += smem[tid + 1];
	} 

	if (tid == 0)
		g_odata[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
}

float dotProduct2D(float* D_data, unsigned int pitch,
                   unsigned int width, unsigned int height)
{
	unsigned int bx = (width + 15) / 16;
	unsigned int by = (height + 127) / 128;
	unsigned int shared_mem2 = sizeof(float) * 16 * 16;

	dim3 dimBlock2(16, 16);
	dim3 dimGrid2(bx, by);

	float* D_buf;
	hipMalloc(&D_buf, sizeof(float) * (bx * by + 1) );
	float* D_res = D_buf + (bx*by);

	// Step 1: reduce 2D from image to a single vector, taking sum of squares

	reduce2D<<< dimGrid2, dimBlock2, shared_mem2>>>(D_data, D_buf, pitch, width, height);
	cudaTextForceKernelsCompletion();

	// Step 2: reduce 1D: add up elements in vector
	if (bx * by > 512)
		reduce1D<512><<< 1, 512, sizeof(float)*512>>>(D_buf, D_res, bx*by);
	else if (bx * by > 128)
		reduce1D<128><<< 1, 128, sizeof(float)*128>>>(D_buf, D_res, bx*by);
	else if (bx * by > 32)
		reduce1D<32><<< 1, 32, sizeof(float)*32*2>>>(D_buf, D_res, bx*by);
	else if (bx * by > 8)
		reduce1D<8><<< 1, 8, sizeof(float)*8*2>>>(D_buf, D_res, bx*by);
	else
		reduce1D<1><<< 1, 1, sizeof(float)*1*2>>>(D_buf, D_res, bx*by);

	float x;
	hipMemcpy(&x, D_res, 4, hipMemcpyDeviceToHost);

	cudaTextForceKernelsCompletion();

	hipFree(D_buf);

	return x;
}


bool cudaTextForceKernelsCompletion()
{
	hipError_t returnedCudaError = hipDeviceSynchronize();

	if(returnedCudaError != hipSuccess) {
		ASTRA_ERROR("Failed to force completion of cuda kernels: %d: %s.", returnedCudaError, hipGetErrorString(returnedCudaError));
		return false;
	}

	return true;
}

void reportCudaError(hipError_t err)
{
	if(err != hipSuccess)
		ASTRA_ERROR("CUDA error %d: %s.", err, hipGetErrorString(err));
}


}
