#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_FanVolumeTexture;


namespace astraCUDA {

static const unsigned g_MaxAngles = 2560;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];


// optimization parameters
static const unsigned int g_anglesPerBlock = 16;
static const unsigned int g_detBlockSize = 32;
static const unsigned int g_blockSlices = 64;

static bool bindVolumeDataTexture(float* data, hipArray*& dataArray, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	dataArray = 0;
	hipMallocArray(&dataArray, &channelDesc, width, height);
	hipMemcpy2DToArray(dataArray, 0, 0, data, pitch*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	gT_FanVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_FanVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_FanVolumeTexture.filterMode = hipFilterModeLinear;
	gT_FanVolumeTexture.normalized = false;

	// TODO: For very small sizes (roughly <=512x128) with few angles (<=180)
	// not using an array is more efficient.
	//hipBindTexture2D(0, gT_FanVolumeTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);
	hipBindTextureToArray(gT_FanVolumeTexture, dataArray, channelDesc);

	// TODO: error value?

	return true;
}

// projection for angles that are roughly horizontal
// (detector roughly vertical)
__global__ void FanFPhorizontal(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	float* projData = (float*)D_projData;
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;
	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy > fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: y = alpha * x + beta
		const float alpha = (fSrcY - fDetY) / (fSrcX - fDetX);
		const float beta = fSrcY - alpha * fSrcX;
	
		const float fDistCorr = sqrt(alpha*alpha+1.0f) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fY = -alpha * (startSlice - 0.5f*dims.iVolWidth + 0.5f) - beta + 0.5f*dims.iVolHeight - 0.5f + 0.5f;
		float fX = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolWidth)
			endSlice = dims.iVolWidth;

		float fV = 0.0f;
		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D(gT_FanVolumeTexture, fX, fY);
			fY -= alpha;
			fX += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}


// projection for angles that are roughly vertical
// (detector roughly horizontal)
__global__ void FanFPvertical(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;

	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	float* projData = (float*)D_projData;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy <= fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet /*- gC_angle_offset[angle]*/;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: x = alpha * y + beta
		const float alpha = (fSrcX - fDetX) / (fSrcY - fDetY);
		const float beta = fSrcX - alpha * fSrcY;
	
		const float fDistCorr = sqrt(alpha*alpha+1) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fX = -alpha * (startSlice - 0.5f*dims.iVolHeight + 0.5f) + beta + 0.5f*dims.iVolWidth - 0.5f + 0.5f;
		float fY = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolHeight)
			endSlice = dims.iVolHeight;

		float fV = 0.0f;

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D(gT_FanVolumeTexture, fX, fY);
			fX -= alpha;
			fY += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}

bool FanFP_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipArray* D_dataArray;
	bindVolumeDataTexture(D_volumeData, D_dataArray, volumePitch, dims.iVolWidth, dims.iVolHeight);

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	dim3 dimBlock(g_detBlockSize, g_anglesPerBlock); // region size, angles
	const unsigned int g_blockSliceSize = g_detBlockSize;

	std::list<hipStream_t> streams;


	unsigned int blockStart = 0;
	unsigned int blockEnd = dims.iProjAngles;

	dim3 dimGrid((blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock,
	             (dims.iProjDets+g_blockSliceSize-1)/g_blockSliceSize); // angle blocks, regions
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	streams.push_back(stream1);
	for (unsigned int i = 0; i < dims.iVolWidth; i += g_blockSlices)
		FanFPhorizontal<<<dimGrid, dimBlock, 0, stream1>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);

	hipStream_t stream2;
	hipStreamCreate(&stream2);
	streams.push_back(stream2);
	for (unsigned int i = 0; i < dims.iVolHeight; i += g_blockSlices)
		FanFPvertical<<<dimGrid, dimBlock, 0, stream2>>>(D_projData, projPitch, i, blockStart, blockEnd, dims, outputScale);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipFreeArray(D_dataArray);

	return true;
}

bool FanFP(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanFP_internal(D_volumeData, volumePitch,
		                         D_projData + iAngle * projPitch, projPitch,
		                         subdims, angles + iAngle,
		                         outputScale);
		if (!ret)
			return false;
	}
	return true;
}

}
